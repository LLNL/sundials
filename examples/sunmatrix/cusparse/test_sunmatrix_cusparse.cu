#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2023, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * SUNMATRIX_CUSPARSE unit tests.
 * -----------------------------------------------------------------
 */

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#include <nvector/nvector_cuda.h>
#include <nvector/nvector_serial.h>
#include <sundials/sundials_math.h>
#include <sundials/sundials_matrix.h>
#include <sundials/sundials_types.h>
#include <sunmatrix/sunmatrix_cusparse.h>
#include <sunmatrix/sunmatrix_sparse.h>
#include "test_sunmatrix.h"
#include "dreadrb.h"

enum { IDENTITY, RANDOM, RBFILE };

/* Implementation specific test of SUNMatrix_cuSparse_SetKernelExecPolicy */
int Test_SetKernelExecPolicy(SUNMatrix A, int myid);

class ATestExecPolicy : public SUNCudaExecPolicy
{
public:
  ATestExecPolicy() : stream_(0) {}

  virtual size_t gridSize(size_t numWorkElements = 0, size_t blockDim = 0) const
  {
    return 1;
  }

  virtual size_t blockSize(size_t numWorkElements = 0, size_t gridDim = 0) const
  {
    return 1;
  }

  virtual const hipStream_t* stream() const
  {
    return &stream_;
  }

  virtual SUNCudaExecPolicy* clone() const
  {
    return static_cast<SUNCudaExecPolicy*>(new ATestExecPolicy());
  }

private:
  const hipStream_t stream_;
};

static SUNContext sunctx;

 /* ----------------------------------------------------------------------
  * Main SUNMatrix Testing Routine
  * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails=0;                    /* counter for test failures  */
  sunindextype M, N;                       /* overall matrix dims        */
  sunindextype blkrows, blkcols;           /* block matrix dims          */
  int          nblocks;                    /* number of matrix blocks    */
  int          block_nnz_max;              /* max number of nnz in block */
  int          mattype;                    /* matrix storage type        */
  N_Vector     x, y, d_x, d_y;             /* test vectors               */
  realtype*    vecdata;                    /* pointers to vector data    */
  SUNMatrix    A, B, C, D, dA, dB, dI;     /* test matrices              */
  realtype*    matdata;                    /* pointer to matrix data     */
  int          print_timing, square;
  int          matrix_to_use;
  sunindextype i, j;
  FILE*        matrixfp;
  char*        filename;
  hipsparseStatus_t cusp_status;
  hipsparseHandle_t cusp_handle;

  if (SUNContext_Create(NULL, &sunctx)) {
    printf("ERROR: SUNContext_Create failed\n");
    return(-1);
  }

  /* initialize some input variables */
  blkrows = 0;
  blkcols = 0;
  nblocks = 0;
  square  = 0;

  /* check input */
  if (argc < 7) {
    printf("ERROR: SIX (6) inputs required: matrix (filename|random|identity), matrix rows, matrix cols, number of blocks, matrix type (CSR/BCSR), print timing (0/1)\n");
    return(-1);
  }

  /* determine what test matrix to use */
  if (!strcmp(argv[1], "random")) {
    matrix_to_use = RANDOM;
  } else if (!strcmp(argv[1], "identity")) {
    matrix_to_use = IDENTITY;
  } else {
    matrix_to_use = RBFILE;
    filename = argv[1];
  }

  /* if we are not reading from a file, verify that the dimension args are legal */
  if (matrix_to_use != RBFILE) {
    blkrows = (sunindextype) atol(argv[2]);
    if (blkrows <= 0) {
      printf("ERROR: number of rows must be a positive integer\n");
      return(-1);
    }

    blkcols = (sunindextype) atol(argv[3]);
    if (blkcols <= 0) {
      printf("ERROR: number of cols must be a positive integer\n");
      return(-1);
    }

    square = (blkrows == blkcols) ? 1 : 0;
  }

  nblocks = (sunindextype) atol(argv[4]);
  if (nblocks < 1) {
    printf("ERROR: number of blocks must be a positive integer\n");
    return(-1);
  }

  if (!strcmp(argv[5], "CSR")) {
    mattype = SUNMAT_CUSPARSE_CSR;
    if (nblocks != 1) {
        printf("ERROR: the CSR format only supports 1 block\n");
        return(-1);
    }
  } else if (!strcmp(argv[5], "BCSR")) {
    mattype = SUNMAT_CUSPARSE_BCSR;
    if (matrix_to_use == RBFILE) {
        printf("ERROR: cannot read BCSR format from a file\n");
    }
    if (!square) {
        printf("ERROR: the BCSR format only supports square block matrices\n");
        return(-1);
    }
  } else {
    printf("ERROR: matrix type must be CSR or BCSR\n");
    return(-1);
  }

  print_timing = atoi(argv[6]);
  SetTiming(print_timing);

  /* Initialize cuSPARSE */
  cusp_status = hipsparseCreate(&cusp_handle);
  if (cusp_status != HIPSPARSE_STATUS_SUCCESS) {
    printf("ERROR: could not create cuSPARSE handle\n");
    return(-1);
  }

  /* Initialize vectors and matrices to NULL */
  x  = NULL;
  y  = NULL;
  A  = NULL;
  B  = NULL;
  C  = NULL;
  D  = NULL;
  dA = NULL;
  dB = NULL;
  dI = NULL;

  if (matrix_to_use == RANDOM) {
    M = blkrows * nblocks;
    N = blkcols * nblocks;
    block_nnz_max = blkrows*blkcols / 2;

    /* Create sparsity pattern for a block. */
    sunindextype *cols = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
    sunindextype *rows = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
    for (i=0; i<block_nnz_max; i++) {
        cols[i] = rand() % blkcols;
        rows[i] = rand() % blkrows;
    }

    /* Fill matrix with uniform random data in [0,1/N] */
    D = SUNDenseMatrix(M, N, sunctx);
    for (i=0; i<nblocks; i++) {
        for (j=0; j<block_nnz_max; j++) {
          sunindextype col = cols[j] + blkcols*i;
          sunindextype row = rows[j] + blkrows*i;
          matdata = SUNDenseMatrix_Column(D,col);
          matdata[row] = (realtype) rand() / (realtype) RAND_MAX / N;
        }
    }
    if (SUNMatScaleAddI(RCONST(1.0), D)) {
      printf("ERROR: SUNMatScaleAddI failed for dense matrix D\n");
      return(-1);
    }

    /* Fill matrix with uniform random data in [0,1/N] */
    C = SUNDenseMatrix(M, N, sunctx);
    for (i=0; i<nblocks; i++) {
        for (j=0; j<block_nnz_max; j++) {
          sunindextype col = cols[j] + blkcols*i;
          sunindextype row = rows[j] + blkrows*i;
          matdata = SUNDenseMatrix_Column(C,col);
          matdata[row] = (realtype) rand() / (realtype) RAND_MAX / N;
        }
    }
    if (SUNMatScaleAddI(RCONST(1.0), C)) {
      printf("ERROR: SUNMatScaleAddI failed for dense matrix C\n");
      return(-1);
    }

    free(cols);
    free(rows);

    /* Create sparse matrices from dense */
    A = SUNSparseFromDenseMatrix(C, ZERO, CSR_MAT);
    if (A == NULL) {
      printf("ERROR: SUNSparseFromDenseMatrix returned NULL for A\n");
      return(-1);
    }
    B = SUNSparseFromDenseMatrix(D, ZERO, CSR_MAT);
    if (B == NULL) {
      printf("ERROR: SUNSparseFromDenseMatrix returned NULL B\n");
      return(-1);
    }
  } else if (matrix_to_use == IDENTITY) {
    M = blkrows * nblocks;
    N = blkcols * nblocks;

    D = SUNDenseMatrix(M, N, sunctx);
    SUNMatScaleAddI(RCONST(0.0), D);
    if (SUNMatScaleAddI(RCONST(0.0), D)) {
      printf("ERROR: SUNMatScaleAddI failed for dense matrix D\n");
      return(-1);
    }

    C = SUNDenseMatrix(M, N, sunctx);
    if (SUNMatScaleAddI(RCONST(0.0), C)) {
      printf("ERROR: SUNMatScaleAddI failed for dense matrix C\n");
      return(-1);
    }

    /* Create sparse matrices from dense */
    A = SUNSparseFromDenseMatrix(C, ZERO, CSR_MAT);
    if (A == NULL) {
      printf("ERROR: SUNSparseFromDenseMatrix returned NULL for A\n");
      return(-1);
    }
    B = SUNSparseFromDenseMatrix(D, ZERO, CSR_MAT);
    if (B == NULL) {
      printf("ERROR: SUNSparseFromDenseMatrix returned NULL B\n");
      return(-1);
    }
  } else {
    SUNMatrix cscA;

    matrixfp = fopen(filename, "r");
    dreadrb_dist(0, matrixfp, &cscA, sunctx);
    fclose(matrixfp);

    if (SUNSparseMatrix_ToCSR(cscA, &A)) {
      printf("ERROR: cannot convert matrix that was read to CSR\n");
      return(-1);
    }
    SUNMatDestroy(cscA);

    if (SUNMatScaleAddI(RCONST(1.0), A)) {
      printf("ERROR: SUNMatScaleAddI failed on matrix that read\n");
      return(-1);
    }

    blkrows = SUNSparseMatrix_Rows(A);
    blkcols = SUNSparseMatrix_Columns(A);
    square = (blkrows == blkcols) ? 1 : 0;
    nblocks = 1;
    M = blkrows * nblocks;
    N = blkcols * nblocks;

    B = SUNMatClone(A);
    if (B == NULL || (SUNMatCopy(A, B) != 0)) {
      printf("ERROR: failed to SUNMatClone and SUNMatCopy\n");
      return(-1);
    }
  }

  printf("cuSPARSE SUNMatrix test: size %ld by %ld, nblocks %ld, block size %ld by %ld, format = %i\n\n",
  (long int) M, (long int) N, (long int) nblocks, (long int) blkrows, (long int) blkcols, mattype);

  if (mattype == SUNMAT_CUSPARSE_CSR) {
    /* Create matrices that will be on the device */
    dA = SUNMatrix_cuSparse_NewCSR(SM_ROWS_S(A), SM_COLUMNS_S(A), SM_NNZ_S(A), cusp_handle, sunctx);
    if (dA == NULL) {
      printf("ERROR: SUNMatrix_cuSparse_NewCSR returned NULL for dA\n");
      return(-1);
    }
    dB = SUNMatrix_cuSparse_NewCSR(SM_ROWS_S(B), SM_COLUMNS_S(B), SM_NNZ_S(B), cusp_handle, sunctx);
    if (dB == NULL) {
      printf("ERROR: SUNMatrix_cuSparse_NewCSR returned NULL for dB\n");
      return(-1);
    }
  } else if (mattype == SUNMAT_CUSPARSE_BCSR) {
    sunindextype block_nnz;

    /* Calculate actual number of nonzeros per block */
    block_nnz = SUNSparseMatrix_NNZ(A) / nblocks;

    /* Create matrices that will be on the device */
    dA = SUNMatrix_cuSparse_NewBlockCSR(nblocks, blkrows, blkrows, block_nnz, cusp_handle, sunctx);
    if (dA == NULL) {
      printf("ERROR: SUNMatrix_cuSparse_NewCSR returned NULL for dA\n");
      return(-1);
    }
    dB = SUNMatrix_cuSparse_NewBlockCSR(nblocks, blkrows, blkrows, block_nnz, cusp_handle, sunctx);
    if (dB == NULL) {
      printf("ERROR: SUNMatrix_cuSparse_NewCSR returned NULL for dB\n");
      return(-1);
    }
  } else {
    printf("ERROR: unknown mattype\n");
    return(-1);
  }

  /* Copy data to device */
  fails = SUNMatrix_cuSparse_CopyToDevice(dA, SM_DATA_S(A), SM_INDEXPTRS_S(A), SM_INDEXVALS_S(A));
  if (fails != 0) {
    printf("ERROR: could not copy A to the device\n");
    return(-1);
  }
  fails = SUNMatrix_cuSparse_CopyToDevice(dB, SM_DATA_S(B), SM_INDEXPTRS_S(B), SM_INDEXVALS_S(B));
  if (fails != 0) {
    printf("ERROR: could not copy B to the device\n");
    return(-1);
  }

  /* Create/fill I matrix */
  dI = NULL;
  if (square) {
    dI = SUNMatClone_cuSparse(dA);
    if (dI == NULL) {
      printf("ERROR: SUNMatClone_cuSparse returned NULL\n");
      return(-1);
    }
    if (SUNMatCopy_cuSparse(dA, dI)) {
      printf("ERROR: SUNMatCopy_cuSparse failed\n");
      return(-1);
    }
    if (SUNMatScaleAddI_cuSparse(ZERO, dI)) {
      printf("ERROR: SUNMatScaleAddI_cuSparse failed\n");
      return(-1);
    }
  }

  /* Create vectors */
  d_x = N_VNew_Cuda(N, sunctx);
  d_y = N_VNew_Cuda(M, sunctx);
  if (d_x == NULL || d_y == NULL) {
    printf("ERROR: N_VNew_Cuda returned NULL\n");
    return(-1);
  }
  x = N_VMake_Serial(N, N_VGetHostArrayPointer_Cuda(d_x), sunctx);
  y = N_VMake_Serial(M, N_VGetHostArrayPointer_Cuda(d_y), sunctx);
  if (x == NULL || y == NULL) {
    printf("ERROR: N_VMake_Serial returned NULL\n");
    return(-1);
  }

  /* Zero the vectors on the host */
  N_VConst(ZERO, x);
  N_VConst(ZERO, y);

  /* Fill vector on the host */
  vecdata = N_VGetArrayPointer(x);
  for(i=0; i<N; i++)
    vecdata[i] = (realtype) rand() / (realtype) RAND_MAX;

  /* Compute reference y on the host */
  if (SUNMatMatvec(A, x, y)) {
    printf("FAIL: SUNSparseMatrix matvec failure \n \n");
    SUNMatDestroy(A);  SUNMatDestroy(B);
    SUNMatDestroy(C);  SUNMatDestroy(D);
    SUNMatDestroy(dA); SUNMatDestroy(dB);
    N_VDestroy(x);  N_VDestroy(y);
    N_VDestroy(d_x); N_VDestroy(d_y);
    if (square) {
        SUNMatDestroy(dI);
    }
    return(1);
  }

  /* Copy vectors to the device */
  N_VCopyToDevice_Cuda(d_x);
  N_VCopyToDevice_Cuda(d_y);

  printf("Setup complete\n");
  printf("Beginning tests\n\n");

  /* SUNMatrix Tests */
  fails += Test_SUNMatGetID(dA, SUNMATRIX_CUSPARSE, 0);
  fails += Test_SUNMatClone(dA, 0);
  fails += Test_SUNMatCopy(dA, 0);
  fails += Test_SUNMatZero(dA, 0);
  fails += Test_SUNMatScaleAdd(dA, dI, 0);
  if (square) fails += Test_SUNMatScaleAddI(dA, dI, 0);
  fails += Test_SUNMatMatvec(dA, d_x, d_y, 0);
  if (square) fails += Test_SetKernelExecPolicy(dI, 0);

  /* Print result */
  if (fails) {
    SUNMatrix_cuSparse_CopyFromDevice(dA, SM_DATA_S(A), NULL, NULL);
    SUNMatrix_cuSparse_CopyFromDevice(dB, SM_DATA_S(B), NULL, NULL);
    printf("\nA =\n");
    SUNSparseMatrix_Print(A,stdout);
    printf("\nB =\n");
    SUNSparseMatrix_Print(B,stdout);
    N_VCopyFromDevice_Cuda(d_x);
    N_VCopyFromDevice_Cuda(d_y);
    printf("\nx\n");
    N_VPrint_Cuda(d_x);
    printf("\ny = Ax (reference)\n");
    N_VPrint_Cuda(d_y);
  } else {
    printf("SUCCESS: SUNMatrix module passed all tests \n \n");
  }

  printf("Beginning teardown\n");

  /* Free vectors and matrices */
  N_VDestroy(x);
  N_VDestroy(y);
  N_VDestroy(d_x);
  N_VDestroy(d_y);
  SUNMatDestroy(A);
  SUNMatDestroy(B);
  SUNMatDestroy(C);
  SUNMatDestroy(D);
  SUNMatDestroy(dA);
  SUNMatDestroy(dB);
  if (square) {
    SUNMatDestroy(dI);
  }

  hipsparseDestroy(cusp_handle);
  SUNContext_Free(&sunctx);

  printf("Teardown complete\n");

  return(fails);
 }

 /* ----------------------------------------------------------------------
  * Test the SUNMatrix_cuSparse_SetKernelExecPolicy function.
  * --------------------------------------------------------------------*/
int Test_SetKernelExecPolicy(SUNMatrix I, int myid)
{
  int print_all_ranks = 0;
  realtype  tol = 100*UNIT_ROUNDOFF;
  SUNMatrix B = SUNMatClone(I);

  /* check cloned matrix */
  if (B == NULL) {
    TEST_STATUS(">>> FAILED test -- SetKernelExecPolicy \n", myid);
    TEST_STATUS("    After SUNMatClone, B == NULL \n \n", myid);
    return(1);
  }

  /* copy data */
  if (SUNMatCopy(I, B)) {
    TEST_STATUS(">>> FAILED test -- SetKernelExecPolicy \n", myid);
    TEST_STATUS("    SUNMatCopy returned nonzero \n \n", myid);
    SUNMatDestroy(B);
    return(1);
  }

  /* set kernel exec policy */
  ATestExecPolicy exec_policy;
  SUNMatrix_cuSparse_SetKernelExecPolicy(B, &exec_policy);

  /* try out an operation */
  if (SUNMatScaleAddI(RCONST(-1.0), B)) {
    TEST_STATUS(">>> FAILED test -- SetKernelExecPolicy \n", myid);
    TEST_STATUS("    SUNMatScaleAddI returned nonzero \n \n", myid);
    SUNMatDestroy(B);
    return(1);
  }

  /* check matrix */
  if (check_matrix_entry(B, ZERO, tol)) {
    TEST_STATUS(">>> FAILED test -- SetKernelExecPolicy \n", myid);
    TEST_STATUS("    check_matrix_entry returned nonzero \n \n", myid);
    SUNMatDestroy(B);
    return(1);
  }

  TEST_STATUS("    PASSED test -- SetKernelExecPolicy \n", myid);

  SUNMatDestroy(B);

  return 0;
}

 /* ----------------------------------------------------------------------
  * Check matrix
  * --------------------------------------------------------------------*/
 int check_matrix(SUNMatrix dA, SUNMatrix dB, realtype tol)
 {
   int failure = 0;
   SUNMatrix A, B;
   realtype *Adata, *Bdata;
   sunindextype *Aindexptrs, *Bindexptrs;
   sunindextype *Aindexvals, *Bindexvals;
   sunindextype i, ANP, Annz, Bnnz;

   /* copy matrix data to host for the checks */
   A = SUNSparseMatrix(SUNMatrix_cuSparse_Rows(dA), SUNMatrix_cuSparse_Columns(dA),
                       SUNMatrix_cuSparse_NNZ(dA), CSR_MAT, sunctx);
   B = SUNSparseMatrix(SUNMatrix_cuSparse_Rows(dB), SUNMatrix_cuSparse_Columns(dB),
                       SUNMatrix_cuSparse_NNZ(dB), CSR_MAT, sunctx);

   failure = SUNMatrix_cuSparse_CopyFromDevice(dA, SM_DATA_S(A),
                                               SM_INDEXPTRS_S(A),
                                               SM_INDEXVALS_S(A));
   failure = SUNMatrix_cuSparse_CopyFromDevice(dB, SM_DATA_S(B),
                                               SM_INDEXPTRS_S(B),
                                               SM_INDEXVALS_S(B));
   hipDeviceSynchronize();

   /* get matrix pointers */
   Adata = SUNSparseMatrix_Data(A);
   Aindexptrs = SUNSparseMatrix_IndexPointers(A);
   Aindexvals = SUNSparseMatrix_IndexValues(A);
   ANP = SUNSparseMatrix_NP(A);
   Annz = SUNSparseMatrix_NNZ(A);

   Bdata = SUNSparseMatrix_Data(B);
   Bindexptrs = SUNSparseMatrix_IndexPointers(B);
   Bindexvals = SUNSparseMatrix_IndexValues(B);
   Bnnz = SUNSparseMatrix_NNZ(B);

   /* matrices must have same sparsetype, shape and actual data lengths */
   if (SUNMatGetID(dA) != SUNMatGetID(dB)) {
     printf(">>> ERROR: check_matrix: Different storage types (%d vs %d)\n",
            SUNMatGetID(dA), SUNMatGetID(dB));
     SUNMatDestroy(dA); SUNMatDestroy(dB);
     return(1);
   }
   if (SUNMatrix_cuSparse_SparseType(A) != SUNMatrix_cuSparse_SparseType(B)) {
     printf(">>> ERROR: check_matrix: Different storage types (%d vs %d)\n",
            SUNMatrix_cuSparse_SparseType(A), SUNMatrix_cuSparse_SparseType(B));
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }
   if (SUNMatrix_cuSparse_Rows(dA) != SUNMatrix_cuSparse_Rows(dB)) {
     printf(">>> ERROR: check_matrix: Different numbers of rows (%ld vs %ld)\n",
            (long int) SUNMatrix_cuSparse_Rows(dA), (long int) SUNMatrix_cuSparse_Rows(dB));
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }
   if (SUNMatrix_cuSparse_Columns(dA) != SUNMatrix_cuSparse_Columns(dB)) {
     printf(">>> ERROR: check_matrix: Different numbers of columns (%ld vs %ld)\n",
            (long int) SUNMatrix_cuSparse_Columns(dA),
            (long int) SUNMatrix_cuSparse_Columns(dB));
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }
   if (Annz != Bnnz) {
     printf(">>> ERROR: check_matrix: Different numbers of nonzeros (%ld vs %ld)\n",
            (long int) Annz, (long int) Bnnz);
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }

   /* compare sparsity patterns */
   for (i=0; i<ANP; i++)
     failure += (Aindexptrs[i] != Bindexptrs[i]);
   if (failure > ZERO) {
     printf(">>> ERROR: check_matrix: Different indexptrs \n");
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }
   for (i=0; i<Annz; i++)
     failure += (Aindexvals[i] != Bindexvals[i]);
   if (failure > ZERO) {
     printf(">>> ERROR: check_matrix: Different indexvals \n");
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }

   /* compare matrix values */
   for(i=0; i<Annz; i++)
     failure += SUNRCompareTol(Adata[i], Bdata[i], tol);
   if (failure > ZERO) {
     printf(">>> ERROR: check_matrix: Different entries \n");
     SUNMatDestroy(A); SUNMatDestroy(B);
     return(1);
   }

   SUNMatDestroy(A); SUNMatDestroy(B);

   return(0);
 }

 int check_matrix_entry(SUNMatrix dA, realtype val, realtype tol)
 {
   int failure = 0;
   realtype *Adata;
   sunindextype i;

   /* copy matrix data to host for the checks */
   Adata = (realtype*) malloc(SUNMatrix_cuSparse_NNZ(dA)*sizeof(realtype));
   failure = SUNMatrix_cuSparse_CopyFromDevice(dA, Adata, NULL, NULL);
   hipDeviceSynchronize();

   /* compare data */
   for(i=0; i < SUNMatrix_cuSparse_NNZ(dA); i++) {
     failure += SUNRCompareTol(Adata[i], val, tol);
   }

   free(Adata);

   if (failure > ZERO)
     return(1);
   else
     return(0);
 }

 int check_vector(N_Vector expected, N_Vector computed, realtype tol)
 {
   int failure = 0;
   realtype *xdata, *ydata;
   sunindextype xldata, yldata;
   sunindextype i;

   /* get vector data */
   xdata = N_VGetHostArrayPointer_Cuda(expected);
   ydata = N_VGetHostArrayPointer_Cuda(computed);

   /* copy data to host */
   N_VCopyFromDevice_Cuda(expected);
   N_VCopyFromDevice_Cuda(computed);
   hipDeviceSynchronize();

   /* check data lengths */
   xldata = N_VGetLength_Cuda(expected);
   yldata = N_VGetLength_Cuda(computed);

   if (xldata != yldata) {
     printf(">>> ERROR: check_vector: Different data array lengths \n");
     return(1);
   }

   /* check vector data */
   for(i=0; i < xldata; i++){
     failure += SUNRCompareTol(xdata[i], ydata[i], tol);
   }

   if (failure > ZERO)
     return(1);
   else
     return(0);
 }

 booleantype has_data(SUNMatrix A)
 {
   realtype *Adata = SUNMatrix_cuSparse_Data(A);
   if (Adata == NULL)
     return SUNFALSE;
   else
     return SUNTRUE;
 }

 booleantype is_square(SUNMatrix A)
 {
   if (SUNMatrix_cuSparse_Rows(A) == SUNMatrix_cuSparse_Columns(A))
     return SUNTRUE;
   else
     return SUNFALSE;
 }

void sync_device(SUNMatrix A)
{
  hipDeviceSynchronize();
}
