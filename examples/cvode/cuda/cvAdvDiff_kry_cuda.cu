#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Acknowledgements: This example is based on cvAdvDiff_bnd
 *                   example by Scott D. Cohen, Alan C.
 *                   Hindmarsh and Radu Serban @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * Example problem:
 *
 * The following is a simple example problem with a banded Jacobian,
 * with the program for its solution by CVODE.
 * The problem is the semi-discrete form of the advection-diffusion
 * equation in 2-D:
 *   du/dt = d^2 u / dx^2 + .5 du/dx + d^2 u / dy^2
 * on the rectangle 0 <= x <= 2, 0 <= y <= 1, and the time
 * interval 0 <= t <= 1. Homogeneous Dirichlet boundary conditions
 * are posed, and the initial condition is
 *   u(x,y,t=0) = x(2-x)y(1-y)exp(5xy).
 * The PDE is discretized on a uniform MX+2 by MY+2 grid with
 * central differencing, and with boundary values eliminated,
 * leaving an ODE system of size NEQ = MX*MY.
 * This program solves the problem with the BDF method, Newton
 * iteration with the CVBAND band linear solver, and a user-supplied
 * Jacobian routine.
 * It uses scalar relative and absolute tolerances.
 * Output is printed at t = .1, .2, ..., 1.
 * Run statistics (optional outputs) are printed at the end.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <cvode/cvode.h>               /* prototypes for CVODE fcts., consts. */
#include <sunlinsol/sunlinsol_spgmr.h> /* access to SPGMR SUNLinearSolver     */
#include <sundials/sundials_types.h>   /* definition of type realtype */
#include <sundials/sundials_math.h>    /* definition of ABS and EXP   */

#include <nvector/nvector_cuda.h>

/* Real Constants */

#define ATOL  RCONST(1.0e-5) /* scalar absolute tolerance */
#define T0    RCONST(0.0)    /* initial time              */
#define T1    RCONST(0.1)    /* first output time         */
#define DTOUT RCONST(0.1)    /* output time increment     */
#define NOUT  10             /* number of output times    */

#define ZERO RCONST(0.0)
#define HALF RCONST(0.5)
#define ONE  RCONST(1.0)
#define TWO  RCONST(2.0)
#define FIVE RCONST(5.0)

#if defined(SUNDIALS_EXTENDED_PRECISION)
#define GSYM "Lg"
#define ESYM "Le"
#define FSYM "Lf"
#else
#define GSYM "g"
#define ESYM "e"
#define FSYM "f"
#endif

#if defined(SUNDIALS_INT64_T)
#define DSYM "ld"
#else
#define DSYM "d"
#endif


/*
 * CUDA kernels
 */

__global__ void fKernel(const realtype *u, realtype *udot,
                               sunindextype MX, sunindextype MY,
                               realtype hordc, realtype horac, realtype verdc)
{
  realtype uij, udn, uup, ult, urt, hdiff, hadv, vdiff;
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < MX*MY) {
    i = tid/MY;
    j = tid%MY;

    uij = u[tid];
    udn = (j ==    0) ? ZERO : u[tid - 1];
    uup = (j == MY-1) ? ZERO : u[tid + 1];
    ult = (i ==    0) ? ZERO : u[tid - MY];
    urt = (i == MX-1) ? ZERO : u[tid + MY];

    /* Set diffusion and advection terms and load into udot */

    hdiff = hordc*(ult - TWO*uij + urt);
    hadv  = horac*(urt - ult);
    vdiff = verdc*(uup - TWO*uij + udn);
    udot[tid] = hdiff + hadv + vdiff;
  }

}

__global__ void jtvKernel(const realtype *vdata, realtype *Jvdata,
                          sunindextype MX, sunindextype MY,
                          realtype hordc, realtype horac, realtype verdc)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < MX*MY) {

    i = tid/MY;
    j = tid%MY;


    /* set the tid-th element of Jv */

    Jvdata[tid] = -TWO*(verdc+hordc) * vdata[tid];
    if (i !=    0) Jvdata[tid] += (hordc - horac) * vdata[tid-MY];
    if (i != MX-1) Jvdata[tid] += (hordc + horac) * vdata[tid+MY];
    if (j !=    0) Jvdata[tid] += verdc * vdata[tid-1];
    if (j != MY-1) Jvdata[tid] += verdc * vdata[tid+1];

  }

}

/* Type : _UserData (contains model and discretization parameters) */
struct _UserData {
  sunindextype MX, MY, NEQ;
  realtype dx, dy, XMAX, YMAX;
  realtype hdcoef, hacoef, vdcoef;
};

typedef _UserData *UserData;

/* Problem setup and initialization functions */
static UserData SetUserData(int argc, char** argv);
static void SetIC(N_Vector u, UserData data);

/* Functions Called by the Solver */
static int f(realtype t, N_Vector u, N_Vector udot, void *user_data);
static int jtv(N_Vector v, N_Vector Jv, realtype t,
               N_Vector u, N_Vector fu,
               void *user_data, N_Vector tmp);

/* Private Helper Functions */
static void PrintHeader(realtype reltol, realtype abstol, realtype umax, UserData data);
static void PrintOutput(realtype t, realtype umax, long int nst);
static void PrintFinalStats(void *cvode_mem);

/* Private function to check function return values */
static int check_retval(void *returnvalue, const char *funcname, int opt);


/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */

int main(int argc, char** argv)
{
  realtype reltol, abstol, t, tout, umax;
  N_Vector u;
  UserData data;
  SUNLinearSolver LS;
  void *cvode_mem;
  int iout, retval;
  long int nst;
  hipStream_t stream;
  hipError_t cuerr;

  u = NULL;
  data = NULL;
  LS = NULL;
  cvode_mem = NULL;

  /* optional: create a cudaStream to use with the CUDA NVector
     (otherwise the default stream is used) */
  cuerr = hipStreamCreate(&stream);
  if(cuerr != hipSuccess) { printf("Error: hipStreamCreate() failed\n"); return(1); }

  /* Set model parameters */
  data = SetUserData(argc, argv);
  if(check_retval((void *)data, "malloc", 2)) return(1);

  reltol = ZERO;  /* Set the tolerances */
  abstol = ATOL;

  /* Create a CUDA vector with initial values */
  u = N_VNew_Cuda(data->NEQ);  /* Allocate u vector */
  if(check_retval((void*)u, "N_VNew_Cuda", 0)) return(1);

  /* Use a non-default cuda stream for kernel execution */
  N_VSetCudaStream_Cuda(u, &stream);

  SetIC(u, data);  /* Initialize u vector */

  /* Call CVodeCreate to create the solver memory and specify the 
   * Backward Differentiation Formula */
  cvode_mem = CVodeCreate(CV_BDF);
  if(check_retval((void *)cvode_mem, "CVodeCreate", 0)) return(1);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in u'=f(t,u), the initial time T0, and
   * the initial dependent variable vector u. */
  retval = CVodeInit(cvode_mem, f, T0, u);
  if(check_retval(&retval, "CVodeInit", 1)) return(1);

  /* Call CVodeSStolerances to specify the scalar relative tolerance
   * and scalar absolute tolerance */
  retval = CVodeSStolerances(cvode_mem, reltol, abstol);
  if (check_retval(&retval, "CVodeSStolerances", 1)) return(1);

  /* Set the pointer to user-defined data */
  retval = CVodeSetUserData(cvode_mem, data);
  if(check_retval(&retval, "CVodeSetUserData", 1)) return(1);

  /* Create SPGMR solver structure without preconditioning
   * and the maximum Krylov dimension maxl */
  LS = SUNLinSol_SPGMR(u, PREC_NONE, 0);
  if(check_retval(&retval, "SUNLinSol_SPGMR", 1)) return(1);

  /* Set CVode linear solver to LS */
  retval = CVodeSetLinearSolver(cvode_mem, LS, NULL);
  if(check_retval(&retval, "CVodeSetLinearSolver", 1)) return(1);

  /* Set the Jacobian-times-vector function */
  retval = CVodeSetJacTimes(cvode_mem, NULL, jtv);
  if(check_retval(&retval, "CVodeSetJacTimesVecFn", 1)) return(1);

  /* In loop over output points: call CVode, print results, test for errors */

  umax = N_VMaxNorm(u);
  PrintHeader(reltol, abstol, umax, data);
  for(iout=1, tout=T1; iout <= NOUT; iout++, tout += DTOUT) {
    retval = CVode(cvode_mem, tout, u, &t, CV_NORMAL);
    if(check_retval(&retval, "CVode", 1)) break;
    umax = N_VMaxNorm(u);
    retval = CVodeGetNumSteps(cvode_mem, &nst);
    check_retval(&retval, "CVodeGetNumSteps", 1);
    PrintOutput(t, umax, nst);
  }

  PrintFinalStats(cvode_mem);  /* Print some final statistics   */

  N_VDestroy(u);          /* Free the u vector */
  CVodeFree(&cvode_mem);  /* Free the integrator memory */
  SUNLinSolFree(LS);      /* Free linear solver memory */
  free(data);             /* Free the user data */
  
  cuerr = hipStreamDestroy(stream); /* Free and cleanup the CUDA stream */
  if(cuerr != hipSuccess) { printf("Error: hipStreamDestroy() failed\n"); return(1); }

  return(0);
}

/*
 *-------------------------------------------
 * Problem setup and initialization functions
 *-------------------------------------------
 */

/* Set model and discretization parameters */

UserData SetUserData(int argc, char *argv[])
{
  const sunindextype MX = 10;
  const sunindextype MY = 5;
  const realtype XMAX = RCONST(2.0);    /* domain boundaries         */
  const realtype YMAX = RCONST(1.0);

  /* Allocate user data structure */
  UserData ud = (UserData) malloc(sizeof *ud);
  if(check_retval((void*) ud, "AllocUserData", 2)) return(NULL);

  ud->MX  = MX;
  ud->MY  = MY;
  ud->NEQ = MX*MY;
  ud->XMAX = XMAX;
  ud->YMAX = YMAX;
  ud->dx = XMAX/(MX+1);  /* Set grid coefficients in data */
  ud->dy = YMAX/(MY+1);
  ud->hdcoef = ONE/(ud->dx*ud->dx);
  ud->hacoef = HALF/(TWO*ud->dx);
  ud->vdcoef = ONE/(ud->dy*ud->dy);

  return ud;
}

/* Set initial conditions in u vector */

static void SetIC(N_Vector u, UserData data)
{
  /* Extract needed constants from data */

  const realtype dx = data->dx;
  const realtype dy = data->dy;
  const realtype xmax = data->XMAX;
  const realtype ymax = data->YMAX;
  const sunindextype MY = data->MY;
  const sunindextype NEQ = data->NEQ;

  /* Extract pointer to solution vector data on the host */
  realtype *udata = N_VGetHostArrayPointer_Cuda(u);

  sunindextype i, j, tid;
  realtype x, y;


  /* Load initial profile into u vector */

  for (tid=0; tid < NEQ; tid++) {
    i = tid / MY;
    j = tid % MY;

    x = (i+1)*dx;
    y = (j+1)*dy;

    udata[tid] = x*(xmax - x)*y*(ymax - y)*SUNRexp(FIVE*x*y);
  }
  N_VCopyToDevice_Cuda(u);
}


/*
 *-------------------------------
 * Functions called by the solver
 *-------------------------------
 */

/* f routine. Compute f(t,u). */

static int f(realtype t, N_Vector u, N_Vector udot, void *user_data)
{
  UserData data = (UserData) user_data;

  /* Extract needed constants from data */
  const sunindextype MX  = data->MX;
  const sunindextype MY  = data->MY;
  const realtype hordc   = data->hdcoef;
  const realtype horac   = data->hacoef;
  const realtype verdc   = data->vdcoef;

  /* Extract pointers to vector data */
  const realtype *udata = N_VGetDeviceArrayPointer_Cuda(u);
  realtype *dudata      = N_VGetDeviceArrayPointer_Cuda(udot);

  unsigned block = 256;
  unsigned grid = (MX*MY + block - 1) / block;

  fKernel<<<grid,block>>>(udata, dudata, MX, MY, hordc, horac, verdc);

  return(0);
}


/* Jacobian-times-vector routine. */

static int jtv(N_Vector v, N_Vector Jv, realtype t,
               N_Vector u, N_Vector fu,
               void *user_data, N_Vector tmp)
{
  UserData data = (UserData) user_data;

  /* Extract needed constants from data */
  const sunindextype MX  = data->MX;
  const sunindextype MY  = data->MY;
  const realtype hordc   = data->hdcoef;
  const realtype horac   = data->hacoef;
  const realtype verdc   = data->vdcoef;

  /* Extract pointers to vector data */
  const realtype *vdata = N_VGetDeviceArrayPointer_Cuda(v);
  realtype *Jvdata      = N_VGetDeviceArrayPointer_Cuda(Jv);

  unsigned block = 256;
  unsigned grid = (MX*MY + block - 1) / block;

  N_VConst(ZERO, Jv);

  jtvKernel<<<grid,block>>>(vdata, Jvdata, MX, MY, hordc, horac, verdc);

  return(0);
}

/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */

/* Print first lines of output (problem description) */

static void PrintHeader(realtype reltol, realtype abstol, realtype umax,
                        UserData data)
{
  printf("\n2-D Advection-Diffusion Equation\n");
  printf("Mesh dimensions = %" DSYM " X %" DSYM "\n", data->MX, data->MY);
  printf("Total system size = %" DSYM "\n", data->NEQ);
  printf("Tolerance parameters: reltol = %" GSYM "   abstol = %" GSYM "\n\n",
         reltol, abstol);
  printf("At t = %" GSYM "      max.norm(u) =%14.6" ESYM " \n", T0, umax);
  return;
}

/* Print current value */

static void PrintOutput(realtype t, realtype umax, long int nst)
{
  printf("At t = %4.2" FSYM "   max.norm(u) =%14.6" ESYM "   nst = %4ld\n", t, umax, nst);
  return;
}

/* Get and print some final statistics */

static void PrintFinalStats(void *cvode_mem)
{
  long lenrw, leniw ;
  long lenrwLS, leniwLS;
  long int nst, nfe, nsetups, nni, ncfn, netf;
  long int nli, npe, nps, ncfl, nfeLS;
  int retval;

  retval = CVodeGetWorkSpace(cvode_mem, &lenrw, &leniw);
  check_retval(&retval, "CVodeGetWorkSpace", 1);
  retval = CVodeGetNumSteps(cvode_mem, &nst);
  check_retval(&retval, "CVodeGetNumSteps", 1);
  retval = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_retval(&retval, "CVodeGetNumRhsEvals", 1);
  retval = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_retval(&retval, "CVodeGetNumLinSolvSetups", 1);
  retval = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_retval(&retval, "CVodeGetNumErrTestFails", 1);
  retval = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_retval(&retval, "CVodeGetNumNonlinSolvIters", 1);
  retval = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_retval(&retval, "CVodeGetNumNonlinSolvConvFails", 1);

  retval = CVodeGetLinWorkSpace(cvode_mem, &lenrwLS, &leniwLS);
  check_retval(&retval, "CVodeGetLinWorkSpace", 1);
  retval = CVodeGetNumLinIters(cvode_mem, &nli);
  check_retval(&retval, "CVodeGetNumLinIters", 1);
  retval = CVodeGetNumPrecEvals(cvode_mem, &npe);
  check_retval(&retval, "CVodeGetNumPrecEvals", 1);
  retval = CVodeGetNumPrecSolves(cvode_mem, &nps);
  check_retval(&retval, "CVodeGetNumPrecSolves", 1);
  retval = CVodeGetNumLinConvFails(cvode_mem, &ncfl);
  check_retval(&retval, "CVodeGetNumLinConvFails", 1);
  retval = CVodeGetNumLinRhsEvals(cvode_mem, &nfeLS);
  check_retval(&retval, "CVodeGetNumLinRhsEvals", 1);

  printf("\nFinal Statistics.. \n\n");
  printf("lenrw   = %5ld     leniw   = %5ld\n"  , lenrw, leniw);
  printf("lenrwLS = %5ld     leniwLS = %5ld\n"  , lenrwLS, leniwLS);
  printf("nst     = %5ld\n"                     , nst);
  printf("nfe     = %5ld     nfeLS   = %5ld\n"  , nfe, nfeLS);
  printf("nni     = %5ld     nli     = %5ld\n"  , nni, nli);
  printf("nsetups = %5ld     netf    = %5ld\n"  , nsetups, netf);
  printf("npe     = %5ld     nps     = %5ld\n"  , npe, nps);
  printf("ncfn    = %5ld     ncfl    = %5ld\n\n", ncfn, ncfl);

  return;
}

/* Check function return value...
     opt == 0 means SUNDIALS function allocates memory so check if
              returned NULL pointer
     opt == 1 means SUNDIALS function returns an integer value so check if
              retval >= 0
     opt == 2 means function allocates memory so check if returned
              NULL pointer */

static int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */

  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if retval < 0 */

  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n",
              funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */

  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}
