#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2023, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * Example problem:
 *
 * The following is a simple example problem, with the program for
 * its solution by CVODE. The problem is the semi-discrete
 * form of the advection-diffusion equation in 1-D:
 *   du/dt = d^2 u / dx^2 + .5 du/dx
 * on the interval 0 <= x <= 2, and the time interval 0 <= t <= 5.
 * Homogeneous Dirichlet boundary conditions are posed, and the
 * initial condition is the following:
 *   u(x,t=0) = x(2-x)exp(2x) .
 * The PDE is discretized on a uniform grid of size MX+2 with
 * central differencing, and with boundary values eliminated,
 * leaving an ODE system of size NEQ = MX.
 * This program solves the problem with the ADAMS integration method,
 * and with Newton iteration using diagonal approximate Jacobians.
 * It can use scalar (default) relative and absolute tolerances or a
 * vector of absolute tolerances (controlled by a runtime argument).
 * The constraint u_i >= 0 is posed for all components.
 * Output is printed at t = .5, 1.0, ..., 5.
 * Run statistics (optional outputs) are printed at the end.
 *
 * ./cvAdvDiff_diag_cuda [0 (scalar atol) | 1 (vector atol)]
 *                       [0 (unfused) | 1 (fused)]
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <cvode/cvode.h>                  /* prototypes for CVODE fcts., consts.  */
#include <cvode/cvode_diag.h>             /* prototypes for CVODE diagonal solver */
#include <nvector/nvector_cuda.h>         /* access to cuda N_Vector              */
#include <sundials/sundials_types.h>      /* definition of type realtype          */

/* Problem Constants */

#define ZERO  RCONST(0.0)

#define XMAX  RCONST(2.0)    /* domain boundary           */
#define MX    10             /* mesh dimension            */
#define NEQ   MX             /* number of equations       */
#define ATOL  RCONST(1e-10)  /* scalar absolute tolerance */
#define T0    ZERO           /* initial time              */
#define T1    RCONST(0.5)    /* first output time         */
#define DTOUT RCONST(0.5)    /* output time increment     */
#define NOUT  10             /* number of output times    */

/* Type : UserData
   contains mesh spacing and problem parameters. */

typedef struct {
  realtype dx;
  realtype hdcoef;
  realtype hacoef;
} *UserData;

/* Private Helper Functions */

static void SetIC(N_Vector u, realtype dx);

static void PrintIntro(int toltype, int usefused);

static void PrintData(realtype t, realtype umax, long int nst);

static void PrintFinalStats(void *cvode_mem);

/* Functions Called by the Solver */

static int f(realtype t, N_Vector u, N_Vector udot, void *user_data);

/* Private function to check function return values */

static int check_retval(void *returnvalue, const char *funcname, int opt);

/***************************** Main Program ******************************/

int main(int argc, char *argv[])
{
  SUNContext sunctx;
  realtype dx, reltol, abstol, t, tout, umax;
  N_Vector u;
  UserData data;
  void *cvode_mem;
  int iout, retval, toltype, usefused;
  long int nst;

  u = NULL;
  data = NULL;
  cvode_mem = NULL;
  toltype = 0;
  usefused = 0;

  /* Create the SUNDIALS context */
  retval = SUNContext_Create(NULL, &sunctx);
  if(check_retval(&retval, "SUNContext_Create", 1)) return(1);

  if (argc >= 2) {
    /* use vector or scalar atol? */
    toltype = atoi(argv[1]);
    /* use fused operations? */
    if (argc == 3)
      usefused = atoi(argv[2]);
  }

  data = (UserData) malloc(sizeof *data);  /* Allocate data memory */
  if(check_retval((void *)data, "malloc", 2)) return 1;

  u = N_VNew_Cuda(NEQ, sunctx);  /* Allocate u vector */
  if(check_retval((void *)u, "N_VNew", 0)) return 1;

  reltol = ZERO;  /* Set the tolerances */
  abstol = ATOL;

  dx = data->dx = XMAX/((realtype)(MX+1));  /* Set grid coefficients in data */
  data->hdcoef = RCONST(1.0)/(dx*dx);
  data->hacoef = RCONST(0.5)/(RCONST(2.0)*dx);

  SetIC(u, dx);  /* Initialize u vector */

  /* Call CVodeCreate to create the solver memory and specify the
   * Adams-Moulton LMM */
  cvode_mem = CVodeCreate(CV_ADAMS, sunctx);
  if(check_retval((void *)cvode_mem, "CVodeCreate", 0)) return 1;

  retval = CVodeSetUserData(cvode_mem, data);
  if(check_retval(&retval, "CVodeSetUserData", 1)) return 1;

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in u'=f(t,u), the inital time T0, and
   * the initial dependent variable vector u. */
  retval = CVodeInit(cvode_mem, f, T0, u);
  if(check_retval(&retval, "CVodeInit", 1)) return(1);

  /* Call CVodeSStolerances to specify the scalar relative tolerance
   * and scalar absolute tolerances */

  if (toltype == 0) {
    retval = CVodeSStolerances(cvode_mem, reltol, abstol);
    if (check_retval(&retval, "CVodeSStolerances", 1)) return(1);
  } else {
    N_Vector vabstol = N_VClone_Cuda(u);
    if (check_retval(&vabstol, "N_VClone_Cuda", 0)) return(1);
    N_VConst(abstol, vabstol);
    retval = CVodeSVtolerances(cvode_mem, reltol, vabstol);
    if (check_retval(&retval, "CVodeSVtolerances", 1)) return(1);
    N_VDestroy(vabstol);
  }

  /* Call CVDiag to create and attach CVODE-specific diagonal linear solver */
  retval = CVDiag(cvode_mem);
  if(check_retval(&retval, "CVDiag", 1)) return(1);

  /* Tell CVode to use fused kernels if they are available. */
  retval = CVodeSetUseIntegratorFusedKernels(cvode_mem, usefused);
  check_retval(&retval, "CVodeSetUseIntegratorFusedKernels", 1);

  PrintIntro(toltype, usefused);

  umax = N_VMaxNorm(u);

  t = T0;
  PrintData(t, umax, 0);

  /* In loop over output points, call CVode, print results, test for error */

  for (iout=1, tout=T1; iout <= NOUT; iout++, tout += DTOUT) {
    retval = CVode(cvode_mem, tout, u, &t, CV_NORMAL);
    if(check_retval(&retval, "CVode", 1)) break;
    umax = N_VMaxNorm(u);
    retval = CVodeGetNumSteps(cvode_mem, &nst);
    check_retval(&retval, "CVodeGetNumSteps", 1);
    PrintData(t, umax, nst);
  }

  PrintFinalStats(cvode_mem);  /* Print some final statistics */

  N_VDestroy(u);                 /* Free the u vector */
  CVodeFree(&cvode_mem);         /* Free the integrator memory */
  free(data);                    /* Free user data */
  SUNContext_Free(&sunctx);

  return(0);
}

/************************ Private Helper Functions ***********************/

/* Set initial conditions in u vector */

static void SetIC(N_Vector u, realtype dx)
{
  int i;
  sunindextype N;
  realtype x;
  realtype *udata;

  /* Set pointer to data array and get local length of u. */
  udata = N_VGetHostArrayPointer_Cuda(u);
  N = N_VGetLength(u);

  /* Load initial profile into u vector */
  for (i=1; i<=N; i++) {
    x = i*dx;
    udata[i-1] = x*(XMAX - x)*exp(RCONST(2.0)*x);
  }
  N_VCopyToDevice_Cuda(u);
}

/* Print problem introduction */

static void PrintIntro(int toltype, int usefused)
{
  printf("\n 1-D advection-diffusion equation, mesh size =%3d \n", MX);
  printf("\n Diagonal linear solver CVDiag \n");
  if (usefused)
    printf(" Using fused CVODE kernels \n");
  if (toltype == 0)
    printf(" Using scalar ATOL\n");
  else
    printf(" Using vector ATOL\n");
  printf("\n");

  return;
}

/* Print data */

static void PrintData(realtype t, realtype umax, long int nst)
{

#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("At t = %4.2Lf  max.norm(u) =%14.6Le  nst =%4ld \n", t, umax, nst);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("At t = %4.2f  max.norm(u) =%14.6e  nst =%4ld \n", t, umax, nst);
#else
  printf("At t = %4.2f  max.norm(u) =%14.6e  nst =%4ld \n", t, umax, nst);
#endif

  return;
}

/* Print some final statistics located in the iopt array */

static void PrintFinalStats(void *cvode_mem)
{
  long int nst, nfe, nni, ncfn, netf;
  int retval;

  retval = CVodeGetNumSteps(cvode_mem, &nst);
  check_retval(&retval, "CVodeGetNumSteps", 1);
  retval = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_retval(&retval, "CVodeGetNumRhsEvals", 1);
  retval = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_retval(&retval, "CVodeGetNumErrTestFails", 1);
  retval = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_retval(&retval, "CVodeGetNumNonlinSolvIters", 1);
  retval = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_retval(&retval, "CVodeGetNumNonlinSolvConvFails", 1);

  printf("\nFinal Statistics: \n\n");
  printf("nst = %-6ld  nfe  = %-6ld  ", nst, nfe);
  printf("nni = %-6ld  ncfn = %-6ld  netf = %ld\n \n", nni, ncfn, netf);
}

 /***************** Function Called by the Solver ***********************/

 /* f routine. Compute f(t,u). */

__global__
static void f_kernel(sunindextype N,
                     realtype hordc, realtype horac,
                     const realtype* u, realtype* udot)
{
  sunindextype i = blockDim.x*blockIdx.x + threadIdx.x;
  realtype ui, ult, urt, hdiff, hadv;

  if (i < N) {
    /* Extract u at x_i and two neighboring points */
    ui = u[i];
    ult = (i == 0) ? ZERO : u[i-1];
    urt = (i == N-1) ? ZERO : u[i+1];

    /* Set diffusion and advection terms and load into udot */
    hdiff = hordc*(ult - RCONST(2.0)*ui + urt);
    hadv = horac*(urt - ult);
    udot[i] = hdiff + hadv;
  }
}

static int f(realtype t, N_Vector u, N_Vector udot, void *user_data)
{
  realtype hordc, horac;
  realtype *udata, *dudata;
  sunindextype N;
  size_t grid, block;
  UserData data;
  hipError_t cuerr;

  udata = N_VGetDeviceArrayPointer_Cuda(u);
  dudata = N_VGetDeviceArrayPointer_Cuda(udot);

  /* Extract needed problem constants from data */
  data = (UserData) user_data;
  hordc = data->hdcoef;
  horac = data->hacoef;

  /* Extract parameters for parallel computation. */
  N = N_VGetLength(u); /* Number of elements of u. */

  block = 64;
  grid  = (block + N - 1)/block;
  f_kernel<<<grid, block>>>(N, hordc, horac, udata, dudata);

  hipDeviceSynchronize();
  cuerr = hipGetLastError();
  if (cuerr != hipSuccess) {
    fprintf(stderr, "ERROR in f: f_kernel --> %s\n", hipGetErrorString(cuerr));
    return(-1);
  }

  return(0);
}

 /* Check function return value...
      opt == 0 means SUNDIALS function allocates memory so check if
               returned NULL pointer
      opt == 1 means SUNDIALS function returns an integer value so check if
               retval < 0
      opt == 2 means function allocates memory so check if returned
               NULL pointer */

static int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n", funcname);
    return(1); }

  /* Check if retval < 0 */
  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n", funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n", funcname);
    return(1); }

  return(0);
}