#include "hip/hip_runtime.h"
/* ------------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * ------------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2023, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * ------------------------------------------------------------------
 * The following is a simple example problem based off of
 * cvRoberts_klu.c. We simulate a scenario where a set of independent
 * ODEs are grouped together to form a larger system. For simplicity,
 * each set of ODEs is the same problem. The problem is from chemical
 * kinetics, and consists of the following three rate equations:
 *    dy1/dt = -.04*y1 + 1.e4*y2*y3
 *    dy2/dt = .04*y1 - 1.e4*y2*y3 - 3.e7*(y2)^2
 *    dy3/dt = 3.e7*(y2)^2
 * on the interval from t = 0.0 to t = 4.e10, with initial
 * conditions: y1 = 1.0, y2 = y3 = 0. The problem is stiff.
 * This program solves the problem with the BDF method, Newton
 * iteration, a user-supplied Jacobian routine, and since the grouping
 * of the independent systems results in a block diagonal linear
 * system, with the cuSOLVER sparse batched QR linear solver. It uses
 * a scalar relative tolerance and a vector absolute tolerance. Output
 * is printed in decades from t = .4 to t = 4.e10. Run statistics
 * (optional outputs) are printed at the end.
 *
 * The program takes one optional argument, the number of groups
 * of independent ODE systems:
 *
 *    ./cvRoberts_block_cusolversp_batchqr [number of groups]
 *
 * This problem is comparable to the cvRoberts_block_klu.c example.
 * ------------------------------------------------------------------*/

#include <stdio.h>

#include <cvode/cvode.h>                              /* prototypes for CVODE fcts., consts.           */
#include <nvector/nvector_cuda.h>                     /* access to cuda N_Vector                       */
#include <sunmatrix/sunmatrix_cusparse.h>             /* access to cusparse SUNMatrix                  */
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>   /* access to cuSolverSp batch QR SUNLinearSolver */
#include <sundials/sundials_types.h>                  /* defs. of realtype, int                        */

/* Problem Constants */

#define GROUPSIZE 3            /* number of equations per group */
#define Y1    RCONST(1.0)      /* initial y components */
#define Y2    RCONST(0.0)
#define Y3    RCONST(0.0)
#define RTOL  RCONST(1.0e-4)   /* scalar relative tolerance            */
#define ATOL1 RCONST(1.0e-8)   /* vector absolute tolerance components */
#define ATOL2 RCONST(1.0e-14)
#define ATOL3 RCONST(1.0e-6)
#define T0    RCONST(0.0)      /* initial time           */
#define T1    RCONST(0.4)      /* first output time      */
#define TMULT RCONST(10.0)     /* output time factor     */
#define NOUT  12               /* number of output times */

#define ZERO  RCONST(0.0)

/* Functions Called by the Solver */

static int f(realtype t, N_Vector y, N_Vector ydot, void *user_data);

__global__
static void f_kernel(realtype t, realtype* y, realtype* ydot,
                     int neq, int ngroups);

static int Jac(realtype t, N_Vector y, N_Vector fy, SUNMatrix J,
               void *user_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3);

__global__
static void j_kernel(int ngroups, int nnzper, realtype* ydata, realtype *Jdata);

/* Private function to initialize the Jacobian sparsity pattern */
static int JacInit(SUNMatrix J);

/* Private function to output results */

static void PrintOutput(realtype t, realtype y1, realtype y2, realtype y3);

/* Private function to print final statistics */

static void PrintFinalStats(void *cvode_mem, SUNLinearSolver LS);

/* Private function to check function return values */

static int check_retval(void *returnvalue, const char *funcname, int opt);

/* user data structure */
typedef struct {
  int ngroups;
  int neq;
} UserData;

/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */

int main(int argc, char *argv[])
{
  SUNContext sunctx;
  realtype reltol, t, tout;
  realtype *ydata, *abstol_data;
  N_Vector y, abstol;
  SUNMatrix A;
  SUNLinearSolver LS;
  void *cvode_mem;
  int retval, iout;
  int neq, ngroups, groupj;
  UserData udata;
  hipsparseHandle_t cusp_handle;
  hipsolverSpHandle_t cusol_handle;

  y = abstol = NULL;
  A = NULL;
  LS = NULL;
  cvode_mem = NULL;

  /* Parse command line arguments */
  if (argc > 1) {
    ngroups = atoi(argv[1]);
  } else {
    ngroups = 100;
  }
  neq = ngroups * GROUPSIZE;

  udata.ngroups = ngroups;
  udata.neq = neq;

  /* Initialize cuSOLVER and cuSPARSE handles */
  hipsparseCreate(&cusp_handle);
  hipsolverSpCreate(&cusol_handle);

  /* Create the SUNDIALS context */
  retval = SUNContext_Create(NULL, &sunctx);
  if(check_retval(&retval, "SUNContext_Create", 1)) return(1);

  /* Create CUDA vector of length neq for I.C. and abstol */
  y = N_VNew_Cuda(neq, sunctx);
  if (check_retval((void *)y, "N_VNew_Cuda", 0)) return(1);
  abstol = N_VNew_Cuda(neq, sunctx);
  if (check_retval((void *)abstol, "N_VNew_Cuda", 0)) return(1);

  ydata = N_VGetHostArrayPointer_Cuda(y);
  abstol_data = N_VGetHostArrayPointer_Cuda(abstol);

  /* Initialize y */
  for (groupj = 0; groupj < neq; groupj += GROUPSIZE) {
    ydata[groupj]   = Y1;
    ydata[groupj+1] = Y2;
    ydata[groupj+2] = Y3;
  }
  N_VCopyToDevice_Cuda(y);

  /* Set the scalar relative tolerance */
  reltol = RTOL;

  /* Set the vector absolute tolerance */
  for (groupj = 0; groupj < neq; groupj += GROUPSIZE) {
    abstol_data[groupj]   = ATOL1;
    abstol_data[groupj+1] = ATOL2;
    abstol_data[groupj+2] = ATOL3;
  }
  N_VCopyToDevice_Cuda(abstol);

  /* Call CVodeCreate to create the solver memory and specify the
   * Backward Differentiation Formula */
  cvode_mem = CVodeCreate(CV_BDF, sunctx);
  if (check_retval((void *)cvode_mem, "CVodeCreate", 0)) return(1);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in y'=f(t,y), the inital time T0, and
   * the initial dependent variable vector y. */
  retval = CVodeInit(cvode_mem, f, T0, y);
  if (check_retval(&retval, "CVodeInit", 1)) return(1);

  /* Call CVodeSetUserData to attach the user data structure */
  retval = CVodeSetUserData(cvode_mem, &udata);
  if (check_retval(&retval, "CVodeSetUserData", 1)) return(1);

  /* Call CVodeSVtolerances to specify the scalar relative tolerance
   * and vector absolute tolerances */
  retval = CVodeSVtolerances(cvode_mem, reltol, abstol);
  if (check_retval(&retval, "CVodeSVtolerances", 1)) return(1);

  /* Create sparse SUNMatrix for use in linear solves */
  A = SUNMatrix_cuSparse_NewBlockCSR(ngroups, GROUPSIZE, GROUPSIZE, GROUPSIZE*GROUPSIZE, cusp_handle, sunctx);
  if(check_retval((void *)A, "SUNMatrix_cuSparse_NewBlockCSR", 0)) return(1);

  /* Set the sparsity pattern to be fixed so that the row pointers
   * and column indicies are not zeroed out by SUNMatZero */
  retval = SUNMatrix_cuSparse_SetFixedPattern(A, 1);

  /* Initialiize the Jacobian with its fixed sparsity pattern */
  JacInit(A);

  /* Create the SUNLinearSolver object for use by CVode */
  LS = SUNLinSol_cuSolverSp_batchQR(y, A, cusol_handle, sunctx);
  if(check_retval((void *)LS, "SUNLinSol_cuSolverSp_batchQR", 0)) return(1);

  /* Call CVodeSetLinearSolver to attach the matrix and linear solver to CVode */
  retval = CVodeSetLinearSolver(cvode_mem, LS, A);
  if(check_retval(&retval, "CVodeSetLinearSolver", 1)) return(1);

  /* Set the user-supplied Jacobian routine Jac */
  retval = CVodeSetJacFn(cvode_mem, Jac);
  if(check_retval(&retval, "CVodeSetJacFn", 1)) return(1);

  /* In loop, call CVode, print results, and test for error.
     Break out of loop when NOUT preset output times have been reached.  */
  printf(" \nGroup of independent 3-species kinetics problems\n\n");
  printf("number of groups = %d\n\n", ngroups);

  iout = 0;  tout = T1;
  while(1) {
    retval = CVode(cvode_mem, tout, y, &t, CV_NORMAL);

    N_VCopyFromDevice_Cuda(y);
    for (groupj = 0; groupj < ngroups; groupj += 10) {
      printf("group %d: ", groupj);
      PrintOutput(t, ydata[GROUPSIZE*groupj],
                  ydata[1+GROUPSIZE*groupj],
                  ydata[2+GROUPSIZE*groupj]);
    }

    if (check_retval(&retval, "CVode", 1)) break;
    if (retval == CV_SUCCESS) {
      iout++;
      tout *= TMULT;
    }

    if (iout == NOUT) break;
  }

  /* Print some final statistics */
  PrintFinalStats(cvode_mem, LS);

  /* Free y and abstol vectors */
  N_VDestroy(y);
  N_VDestroy(abstol);

  /* Free integrator memory */
  CVodeFree(&cvode_mem);

  /* Free the linear solver memory */
  SUNLinSolFree(LS);

  /* Free the matrix memory */
  SUNMatDestroy(A);

  SUNContext_Free(&sunctx);

  /* Destroy the cuSOLVER and cuSPARSE handles */
  hipsparseDestroy(cusp_handle);
  hipsolverSpDestroy(cusol_handle);

  return(0);
}


/*
 *-------------------------------
 * Functions called by the solver
 *-------------------------------
 */

/* Right hand side function. This just launches the CUDA kernel
   to do the actual computation. At the very least, doing this
   saves moving the vector data in y and ydot to/from the device
   every evaluation of f. */
static int f(realtype t, N_Vector y, N_Vector ydot, void *user_data)
{
  UserData *udata;
  realtype *ydata, *ydotdata;

  udata = (UserData*) user_data;
  ydata = N_VGetDeviceArrayPointer_Cuda(y);
  ydotdata = N_VGetDeviceArrayPointer_Cuda(ydot);

  unsigned block_size = 32;
  unsigned grid_size = (udata->neq + block_size - 1) / block_size;
  f_kernel<<<grid_size, block_size>>>(t, ydata, ydotdata, udata->neq, udata->ngroups);

  hipDeviceSynchronize();
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess) {
    fprintf(stderr,
            ">>> ERROR in f: hipGetLastError returned %s\n",
            hipGetErrorName(cuerr));
    return(-1);
  }

  return(0);
}

/* Right hand side function evalutation kernel. */
__global__
static void f_kernel(realtype t, realtype* ydata, realtype* ydotdata,
                     int neq, int ngroups)
{
  realtype y1, y2, y3, yd1, yd3;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int groupj = i*GROUPSIZE;

  if (i < neq) {
    y1 = ydata[groupj]; y2 = ydata[groupj+1]; y3 = ydata[groupj+2];

    yd1 = ydotdata[groupj]   = RCONST(-0.04)*y1 + RCONST(1.0e4)*y2*y3;
    yd3 = ydotdata[groupj+2] = RCONST(3.0e7)*y2*y2;
    ydotdata[groupj+1] = -yd1 - yd3;
  }
}


/*
 * Jacobian initialization routine. This sets the sparisty pattern of
 * the blocks of the Jacobian J(t,y) = df/dy. This is performed on the CPU,
 * and only occurs at the beginning of the simulation.
 */

static int JacInit(SUNMatrix J)
{
  int rowptrs[4], colvals[9];

  /* Zero out the Jacobian */
  SUNMatZero(J);

  /* there are 3 entries per row */
  rowptrs[0] = 0;
  rowptrs[1] = 3;
  rowptrs[2] = 6;
  rowptrs[3] = 9;

  /* first row of block */
  colvals[0] = 0;
  colvals[1] = 1;
  colvals[2] = 2;

  /* second row of block */
  colvals[3] = 0;
  colvals[4] = 1;
  colvals[5] = 2;

  /* third row of block */
  colvals[6] = 0;
  colvals[7] = 1;
  colvals[8] = 2;

  /* copy rowptrs, colvals to the device */
  SUNMatrix_cuSparse_CopyToDevice(J, NULL, rowptrs, colvals);
  hipDeviceSynchronize();

  return(0);
}

/*
 * Jacobian routine. Compute J(t,y) = df/dy.
 * This is done on the GPU.
 */

static int Jac(realtype t, N_Vector y, N_Vector fy, SUNMatrix J,
               void *user_data, N_Vector tmp1, N_Vector tmp2, N_Vector tmp3)
{
  UserData *udata = (UserData*) user_data;
  int nnzper;
  realtype *Jdata, *ydata;
  unsigned block_size, grid_size;

  nnzper  = GROUPSIZE * GROUPSIZE;
  Jdata   = SUNMatrix_cuSparse_Data(J);
  ydata   = N_VGetDeviceArrayPointer_Cuda(y);

  block_size = 32;
  grid_size = (udata->neq + block_size - 1) / block_size;
  j_kernel<<<grid_size, block_size>>>(udata->ngroups, nnzper, ydata, Jdata);

  hipDeviceSynchronize();
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess) {
    fprintf(stderr,
            ">>> ERROR in Jac: hipGetLastError returned %s\n",
            hipGetErrorName(cuerr));
    return(-1);
  }

  return(0);
}

/* Jacobian evaluation GPU kernel */
__global__
static void j_kernel(int ngroups, int nnzper, realtype* ydata, realtype *Jdata)
{
  int groupj;
  realtype y2, y3;

  for (groupj = blockIdx.x*blockDim.x + threadIdx.x;
       groupj < ngroups;
       groupj += blockDim.x * gridDim.x)
  {
    /* get y values */
    y2 = ydata[GROUPSIZE*groupj + 1];
    y3 = ydata[GROUPSIZE*groupj + 2];

    /* first row of block */
    Jdata[nnzper*groupj]       = RCONST(-0.04);
    Jdata[nnzper*groupj + 1]   = RCONST(1.0e4)*y3;
    Jdata[nnzper*groupj + 2]   = RCONST(1.0e4)*y2;

    /* second row of block */
    Jdata[nnzper*groupj + 3]   = RCONST(0.04);
    Jdata[nnzper*groupj + 4]   = (RCONST(-1.0e4)*y3) - (RCONST(6.0e7)*y2);
    Jdata[nnzper*groupj + 5]   = RCONST(-1.0e4)*y2;

    /* third row of block */
    Jdata[nnzper*groupj + 6]   = ZERO;
    Jdata[nnzper*groupj + 7]   = RCONST(6.0e7)*y2;
    Jdata[nnzper*groupj + 8]   = ZERO;
  }
}

/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */

static void PrintOutput(realtype t, realtype y1, realtype y2, realtype y3)
{
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("At t = %0.4Le      y =%14.6Le  %14.6Le  %14.6Le\n", t, y1, y2, y3);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
#else
  printf("At t = %0.4e      y =%14.6e  %14.6e  %14.6e\n", t, y1, y2, y3);
#endif

  return;
}

/*
 * Get and print some final statistics
 */

static void PrintFinalStats(void *cvode_mem, SUNLinearSolver LS)
{
  long int nst, nfe, nsetups, nje, nni, ncfn, netf, nge;
  int retval;

  retval = CVodeGetNumSteps(cvode_mem, &nst);
  check_retval(&retval, "CVodeGetNumSteps", 1);
  retval = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_retval(&retval, "CVodeGetNumRhsEvals", 1);
  retval = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_retval(&retval, "CVodeGetNumLinSolvSetups", 1);
  retval = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_retval(&retval, "CVodeGetNumErrTestFails", 1);
  retval = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_retval(&retval, "CVodeGetNumNonlinSolvIters", 1);
  retval = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_retval(&retval, "CVodeGetNumNonlinSolvConvFails", 1);

  retval = CVodeGetNumJacEvals(cvode_mem, &nje);
  check_retval(&retval, "CVodeGetNumJacEvals", 1);

  retval = CVodeGetNumGEvals(cvode_mem, &nge);
  check_retval(&retval, "CVodeGetNumGEvals", 1);

  printf("\nFinal Statistics:\n");
  printf("nst = %-6ld nfe  = %-6ld nsetups = %-6ld nje = %ld\n",
         nst, nfe, nsetups, nje);
  printf("nni = %-6ld ncfn = %-6ld netf = %-6ld    nge = %ld\n",
         nni, ncfn, netf, nge);
}

/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns an integer value so check if
 *            retval < 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

static int check_retval(void *returnvalue, const char *funcname, int opt)
{
  int *retval;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && returnvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if retval < 0 */
  else if (opt == 1) {
    retval = (int *) returnvalue;
    if (*retval < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with retval = %d\n\n",
              funcname, *retval);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && returnvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}
