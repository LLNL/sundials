#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------------
 * Programmer(s): Shelby Lockhart @ UIUC/LLNL
 * -----------------------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2023, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------------------
 * Example problem:
 *
 * The following test implements a variation of the expectation-maximization
 * problem for mixture densities from [1] with performance and convergence
 * results presented in [2]. Here, we consider a mixture density composed of
 * three univariate normal densities with a mixture density given by
 *
 *   P(x) = \sum_{i=1}^{3} alpha_i * Z_i(x|mu_i, sigma_i)
 *
 * where
 *
 *   Z_i(x|mu_i, sigma_i) =
 *        1 / (sqrt(2*pi)*sigma_i) * e^{-(x-mu_i)^2 / (2*sigma_i^2)}
 *
 * Mixture proportions {alpha_i}_{i=1}^{3} are non-negative and sum to 1. The
 * mixture proportions and variances are assumed to be known and the means
 * {mu_i}_{i=1}^3 are estimated from a set of unlabeled samples {x_k}_{k=1}^N,
 * or samples of unknown origin. Determining the unknown means distribution
 * parameters is given by the following function for 1 <= i <= 3
 *
 *   G(mu_i) = mu_i =
 *      [ \sum_{k=1}^N x_k * (alpha_i * Z_i(x_k|mu_i, sigma_i)) / (P(x_k)) ] /
 *      [ \sum_{k=1}^N (alpha_i * Z_i(x_k|mu_i, sigma_i)) / (P(x_k)) ]
 *
 * with current mean estimations being applied alongside the known mixture
 * proportions and variances as the original test case,
 *
 *  alpha_1 = 0.3   alpha_2 = 0.3   alpha_3 = 0.4
 *
 * and
 *
 *  sigma_1 = sigma_2 = sigma_3 = 1.0.
 *
 * We generate 100,000 samples for the mean distribution set
 *
 *  mu_1 = 0   mu_2 = 0.5   mu_3 = 1.0
 *
 * corresponding to a poorly separated mixture and used the same AA parameter of
 * m=3 as in [1]. We estimate a single set of mean distribution parameters
 * redundantly for every entry in a global vector u.
 *
 * 1. H.F. Walker and P. Ni, "Anderson acceleration for fixed-point iterations",
 *    SIAM Journal on Numerical Analysis, 49 (2011), pp. 1715-1735.
 *
 * 2. S. Lockhart, D.J. Gardner, C.S. Woodward, S. Thomas and L.N. Olson,
 *    "Performance of Low Synchronization Orthogonliazation Methods in Anderson
 *    Accelerated Fixed Point Solvers." arXiv preprint arXiv:2110.09667 (2021).
 *
 * Several command line options are available to change the problem parameters
 * and KINSOL settings. Use the retval --help for more information.
 * ---------------------------------------------------------------------------*/

#include <sundials/sundials_logger.h>
// Header file containing UserData and function declarations
#include "kin_em_mpicuda.hpp"

// -----------------------------------------------------------------------------
// Cuda Kernels
// -----------------------------------------------------------------------------

__global__
void PxKernel(realtype *mu, realtype *Px, realtype *x,
              realtype a1, realtype a2, realtype a3, realtype scale,
              sunindextype N)
{
  // Calculate all P(x_k) for each x value
  realtype val1, val2, val3;

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < N) {
    val1 = x[tid] - mu[0];
    val2 = x[tid] - mu[1];
    val3 = x[tid] - mu[2];

    Px[tid] = a1 * scale * exp( -(val1 * val1)/TWO );
    Px[tid] += a2 * scale * exp( -(val2 * val2)/TWO );
    Px[tid] += a3 * scale * exp( -(val3 * val3)/TWO );
  }
}

__global__
void EMKernel(realtype *mu, realtype *mu_top, realtype *mu_bottom,
              realtype *x, realtype *Px,
              realtype a1, realtype a2, realtype a3, realtype scale,
              sunindextype N)
{
  realtype val1, val2, val3;
  realtype frac1, frac2, frac3;

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < N) {
    val1 = x[tid] - mu[0];
    val2 = x[tid] - mu[1];
    val3 = x[tid] - mu[2];

    frac1 = a1 * scale * exp( -(val1 * val1)/TWO ) / Px[tid];
    frac2 = a2 * scale * exp( -(val2 * val2)/TWO ) / Px[tid];
    frac3 = a3 * scale * exp( -(val3 * val3)/TWO ) / Px[tid];

    atomicAdd(mu_top,     x[tid] * frac1);
    atomicAdd(mu_top + 1, x[tid] * frac2);
    atomicAdd(mu_top + 2, x[tid] * frac3);

    atomicAdd(mu_bottom,     frac1);
    atomicAdd(mu_bottom + 1, frac2);
    atomicAdd(mu_bottom + 2, frac3);
  }
}

__global__
void EMKernelFin(realtype *mu, realtype *mu_top, realtype *mu_bottom,
                 sunindextype localn)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < localn) {
    mu[3*tid]   = mu_top[0] / mu_bottom[0];
    mu[3*tid+1] = mu_top[1] / mu_bottom[1];
    mu[3*tid+2] = mu_top[2] / mu_bottom[2];
  }
}

// -----------------------------------------------------------------------------
// Main Program
// -----------------------------------------------------------------------------

int main(int argc, char* argv[])
{
  // Initialize MPI
  int retval = MPI_Init(&argc, &argv);
  if (check_retval(&retval, "MPI_Init", 1)) return 1;

  // Add scope so objects are destroyed before MPI_Finalize
  {
    // User data structure
    UserData *udata = NULL;

    // Timing variables
    double t1 = 0.0;
    double t2 = 0.0;

    // MPI communicator and process ID
    MPI_Comm comm_w = MPI_COMM_WORLD;
    int myid;

    retval = MPI_Comm_rank(comm_w, &myid);
    if (check_retval(&retval, "MPI_Comm_rank", 1)) return 1;

    // Set output process flag
    bool outproc = (myid == 0);

    // SUNDIALS context
    sundials::Context sunctx(&comm_w);

    // ------------------------------------------
    // Setup UserData and parallel decomposition
    // ------------------------------------------

    // Allocate and initialize user data structure with default values. The
    // defaults may be overwritten by command line inputs in ReadInputs below.
    udata = new UserData;
    retval = InitUserData(udata);
    if (check_retval(&retval, "InitUserData", 1)) return 1;

    // Parse command line inputs
    retval = ReadInputs(&argc, &argv, udata, outproc);
    if (retval != 0) return 1;

    // Output problem setup/options
    if (outproc)
    {
      retval = PrintUserData(udata);
      if (check_retval(&retval, "PrintUserData", 1)) return 1;
    }

    // ---------------
    // Setup debugging
    // ---------------
    SUNLogger logger = NULL;
    if (udata->debug)
    {
      char fname[MXSTR];
      snprintf(fname, MXSTR, "kinsol_output_%06d.txt", myid);

      /* This requires that SUNDIALS was configured with the CMake options
         SUNDIALS_LOGGING_LEVEL=n where n is one of:
            1 --> log only errors,
            2 --> log errors + warnings,
            3 --> log errors + warnings + informational output
            4 --> all of the above plus debugging output
            5 --> all of the above and even more
          SUNDIALS will only log up to the max level n, but a lesser level can
          be configured at runtime by only providing output files for the
          desired levels. We will enable all logging here: */
      retval = SUNLogger_Create((void*)&comm_w, -1, &logger); /* output on all ranks */
      if (check_retval(&retval, "SUNLogger_Create", 1)) return 1;
      retval = SUNLogger_SetDebugFilename(logger, fname);
      if (check_retval(&retval, "SUNLogger_SetDebugFilename", 1)) return 1;
      retval = SUNContext_SetLogger(sunctx, logger);
      if (check_retval(&retval, "SUNContext_SetLogger", 1)) return 1;
    }

    // --------------------------
    // Create MPI + Cuda vectors
    // --------------------------

    // Create vector for solution
    N_Vector ulocal = N_VNew_Cuda(3 * udata->nodes_loc, sunctx);
    if (check_retval((void *) ulocal, "N_VNew_Cuda", 0)) return 1;

    N_Vector u = N_VMake_MPIPlusX(udata->comm, ulocal, sunctx);
    if (check_retval((void *) u, "N_VMake_MPIPlusX", 0)) return 1;

    // Create vector for scaling initial value
    N_Vector scale = N_VClone(u);
    if (check_retval((void *) scale, "N_VClone", 0)) return 1;
    N_VConst(ONE, scale);

    // Set initial condition
    retval = SetStartGuess(u, udata);
    if (check_retval(&retval, "RandomVec", 1)) return 1;

    // Create vector true mu values
    udata->mu_true = N_VClone(u);
    if (check_retval((void *) (udata->mu_true), "N_VClone", 0)) return 1;

    // Create temporary vector for residual and error output
    udata->vtemp = N_VClone(u);
    if (check_retval((void *) (udata->vtemp), "N_VClone", 0)) return 1;

    // Create temporary vector for mu calculation
    udata->mu_bottom = N_VNew_Cuda(3, sunctx);
    if (check_retval((void *) (udata->mu_bottom), "N_VNewCuda", 0)) return 1;

    udata->mu_top = N_VNew_Cuda(3, sunctx);
    if (check_retval((void *) (udata->mu_top), "N_VNewCuda", 0)) return 1;

    // Create vector for samples
    udata->samples_local = N_VNew_Cuda(udata->num_samples, sunctx);
    if (check_retval((void *) udata->samples_local, "N_VNew_Cuda", 0)) return 1;

    // Clone samples for temporary vector
    udata->px = N_VClone(udata->samples_local);
    if (check_retval((void *) (udata->px), "N_VClone", 0)) return 1;

    // --------------
    // Setup Mus
    // --------------

    retval = SetMus(udata);
    if (check_retval(&retval, "SetMus", 1)) return 1;

    // --------------
    // Setup Samples
    // --------------

    retval = SetupSamples(udata);
    if (check_retval(&retval, "SetupSamples", 1)) return 1;

    // --------------
    // Setup KINSOL
    // --------------

    // Initialize KINSOL memory
    void* kin_mem = KINCreate(sunctx);
    if (check_retval((void *) kin_mem, "KINCreate", 0)) return 1;

    // Set number of prior residuals used in Anderson Accleration
    retval = KINSetMAA(kin_mem, udata->maa);
    if (check_retval(&retval, "KINSetMAA", 0)) return 1;

    // Set orthogonlization routine used in Anderson Accleration
    retval = KINSetOrthAA(kin_mem, udata->orthaa);
    if (check_retval(&retval, "KINSetOrthAA", 0)) return 1;

    // Set Fixed Point Function
    retval = KINInit(kin_mem, FPFunction, u);
    if (check_retval(&retval, "KINInit", 1)) return 1;

    // Specify tolerances
    retval = KINSetFuncNormTol(kin_mem, udata->rtol);
    if (check_retval(&retval, "KINSetFuncNormTol", 1)) return 1;

    // Set maximum number of iterations
    retval = KINSetNumMaxIters(kin_mem, udata->maxits);
    if (check_retval(&retval, "KINSetMaxNumIters", 1)) return 1;

    // Set Anderson Acceleration damping parameter
    retval = KINSetDampingAA(kin_mem, udata->damping);
    if (check_retval(&retval, "KINSetDampingAA", 1)) return 1;

    // Attach user data
    retval = KINSetUserData(kin_mem, (void *) udata);
    if (check_retval(&retval, "KINSetUserData", 1)) return 1;

    // ----------------------------
    // Call KINSol to solve problem
    // ----------------------------

    // No scaling used
    N_VConst(ONE, scale);

    if (udata->output > 1)
    {
      retval = OpenOutput(udata);
      if (check_retval(&retval, "OpenOutput", 1)) return 1;
    }

    // Start timer
    t1 = MPI_Wtime();

    // Call main solver
    retval = KINSol(kin_mem,        // KINSol memory block
                  u,              // inital guess on input; solution vector
                  KIN_FP,         // global strategy choice
                  scale,          // scaling vector, for the variable u
                  scale);         // scaling vector for function values fval
    if (check_retval(&retval, "KINSol", 1)) return(1);

    // Stop timer
    t2 = MPI_Wtime();

    // Update timer
    udata->totaltime = t2 - t1;

    // -----------------------
    // Get solver statistics
    // -----------------------

    if (udata->output > 0 && outproc)
    {
      cout << "Final statistics:" << endl;
      retval = OutputStats(kin_mem, udata);
      if (check_retval(&retval, "OutputStats", 1)) return 1;
    }
    if (udata->output > 1)
    {
      retval = CloseOutput(udata);
      if (check_retval(&retval, "CloseOutput", 1)) return 1;
    }

    // ------------------------------
    // Print timing
    // ------------------------------

    if (udata->timing)
    {
      retval = OutputTiming(udata);
      if (check_retval(&retval, "OutputTiming", 1)) return 1;
    }

    // ------------------------------
    // Free memory
    // ------------------------------

    if (udata->debug) SUNLogger_Destroy(&logger);
    KINFree(&kin_mem);         // Free solver memory
    N_VDestroy(u);             // Free vectors
    N_VDestroy(scale);
    FreeUserData(udata);       // Free user data
    delete udata;
  }

  // Finalize MPI
  retval = MPI_Finalize();

  return 0;
}

// -----------------------------------------------------------------------------
// Functions called by the solver
// -----------------------------------------------------------------------------

// Fixed point function to compute G(u)
static int FPFunction(N_Vector u, N_Vector f, void *user_data)
{
  int retval;

  // Access problem data
  UserData *udata = (UserData *) user_data;

  // Start timer
  double t1 = MPI_Wtime();

  // Call EM Algorithm
  retval = EM(u, f, user_data);
  if (check_retval(&retval, "EM", 1)) return -1;

  // Stop timer
  double t2 = MPI_Wtime();

  // Update timer
  udata->fevaltime += t2 - t1;

  // Calculate and output residual and error history
  if (udata->output > 1)
  {
    retval = WriteOutput(u, f, udata);
    if (check_retval(&retval, "WriteOutput", 1)) return 1;
  }

  // Return success
  return 0;
}

// Setup mean distribution samples
static int SetupSamples(UserData *udata)
{
  sunindextype i, j, start, end;
  realtype mean, val;

  // Access problem data
  realtype *samples_local = N_VGetHostArrayPointer_Cuda(udata->samples_local);
  if (check_retval((void *) samples_local, "N_VGetHostArrayPointer_Cuda", 0)) return 1;

  realtype *mu_host = N_VGetHostArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(udata->mu_true));
  if (check_retval((void *) mu_host, "N_VGetHostArrayPointer_Cuda", 0)) return 1;

  realtype std_dev = ONE;

  for (i = 0; i < 3; i++) {
    // Set number of samples with this mean
    if (i == 0 || i == 1) {
      end = 3 * (udata->num_samples / 10);
      start = i * end;
      end += start;
    }
    else {
      end = 4 * (udata->num_samples / 10);
      start = 2 * (3 * (udata->num_samples / 10));
      end += start;
    }

    // Setup distribution parameters
    mean = mu_host[i];
    std::default_random_engine generator;
    std::normal_distribution<realtype> distribution(mean, std_dev);

    // Get samples
    for (j = start; j < end; j++) {
      val = distribution(generator);
      samples_local[j] = val;
    }
  }

  N_VCopyToDevice_Cuda(udata->samples_local);

  // Return success
  return 0;
}

// Fill the vector u with random data
static int SetMus(UserData *udata)
{
  sunindextype i;

  realtype *mu_host = N_VGetHostArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(udata->mu_true));
  if (check_retval((void *) mu_host, "N_VGetHostArrayPointer_Cuda", 0)) return 1;

  // Fill vectors with uniform random data in [-1,1]
  for (i = 0; i < udata->nodes_loc; i++)
  {
    mu_host[3*i]   = ZERO;
    mu_host[3*i+1] = HALF;
    mu_host[3*i+2] = ONE;
  }

  N_VCopyToDevice_Cuda(N_VGetLocalVector_MPIPlusX(udata->mu_true));

  // Return success
  return 0;
}

static int SetStartGuess(N_Vector u, UserData* udata)
{
  realtype *u_host = N_VGetHostArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(u));
  if (check_retval((void *) u_host, "N_VGetHostArrayPointer_Cuda", 0)) return 1;

  for (sunindextype i = 0; i < udata->nodes_loc; i++)
  {
    u_host[3 * i]     = RCONST(0.25);
    u_host[3 * i + 1] = RCONST(3.0);
    u_host[3 * i + 2] = RCONST(0.75);
  }

  N_VCopyToDevice_Cuda(N_VGetLocalVector_MPIPlusX(u));

  // Return success
  return 0;
}


static int EM(N_Vector u, N_Vector f, void *user_data)
{
  // Access problem data
  UserData *udata = (UserData *) user_data;

  // Set grid and block sizes for kernel launch
  unsigned block = 256;
  unsigned grid1 = (udata->num_samples + block - 1) / block;
  unsigned grid2 = (udata->nodes_loc + block - 1) / block;

  // ---------
  // PX KERNEL
  // ---------

  // Scale value for functions
  realtype scale = ONE / sqrt(TWO * PI);

  // Get input device pointers
  realtype *u_dev  = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(u));
  realtype *x_dev  = N_VGetDeviceArrayPointer_Cuda(udata->samples_local);

  // Get output device pointer
  realtype *Px_dev = N_VGetDeviceArrayPointer_Cuda(udata->px);

  // Compute Px
  PxKernel<<<grid1, block>>>(u_dev, Px_dev, x_dev,
                             udata->alpha1, udata->alpha2, udata->alpha3, scale,
                             udata->num_samples);

  // ---------
  // EM KERNEL
  // ---------

  // Get output device pointers
  realtype *mu_bottom_dev = N_VGetDeviceArrayPointer_Cuda(udata->mu_bottom);
  realtype *mu_top_dev    = N_VGetDeviceArrayPointer_Cuda(udata->mu_top);

  // Initilaize output vectors to zero (for sum reduction)
  N_VConst(ZERO, udata->mu_bottom);
  N_VConst(ZERO, udata->mu_top);

  EMKernel<<<grid1, block>>>(u_dev, mu_top_dev, mu_bottom_dev, x_dev, Px_dev,
                             udata->alpha1, udata->alpha2, udata->alpha3, scale,
                             udata->num_samples);

  // ------------------
  // EM FINALIZE KERNEL
  // ------------------

  realtype *f_dev = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(f));

  EMKernelFin<<<grid2, block>>>(f_dev, mu_top_dev, mu_bottom_dev,
                                udata->nodes_loc);

  // Return success
  return 0;
}

// -----------------------------------------------------------------------------
// UserData and input functions
// -----------------------------------------------------------------------------

// Initialize memory allocated within Userdata
static int InitUserData(UserData *udata)
{
  int retval;

  // Sigmas
  udata->sigma = ONE;

  // Alphas - mixture proportions
  udata->alpha1 = PTTHREE;
  udata->alpha2 = PTTHREE;
  udata->alpha3 = PTFOUR;

  // MPI variables
  udata->comm = MPI_COMM_WORLD;

  // Get the number of processes
  retval = MPI_Comm_size(udata->comm, &(udata->nprocs_w));
  if (retval != MPI_SUCCESS)
  {
    cerr << "Error in MPI_Comm_size = " << retval << endl;
    return -1;
  }

  // Get my rank
  retval = MPI_Comm_rank(udata->comm, &(udata->myid));
  if (retval != MPI_SUCCESS)
  {
    cerr << "Error in MPI_Comm_rank = " << retval << endl;
    return -1;
  }

  // Local number of nodes
  udata->nodes_loc = 1;

  // Global total number of nodes
  udata->nodes = udata->nodes_loc * udata->nprocs_w;

  // Integrator settings
  udata->rtol        = RCONST(1.e-8);   // relative tolerance
  udata->maa         = 3;               // 3 vectors in Anderson Acceleration space
  udata->damping     = ONE;             // no damping for Anderson Acceleration
  udata->orthaa      = 0;               // use MGS for Anderson Acceleration
  udata->maxits      = 200;             // max number of fixed point iterations

  // Vectors
  udata->samples_local = NULL;
  udata->px            = NULL;
  udata->mu_bottom     = NULL;
  udata->mu_top        = NULL;
  udata->mu_true       = NULL;

  // Number samples
  udata->num_samples = 100000;

  // Output variables
  udata->output = 1;   // 0 = no output, 1 = stats output, 2 = output to disk
  udata->vtemp  = NULL;

  // Timing variables
  udata->timing       = false;
  udata->totaltime    = 0.0;
  udata->fevaltime    = 0.0;

  udata->debug = false;

  // Return success
  return 0;
}

// Free memory allocated within Userdata
static int FreeUserData(UserData *udata)
{

  // Free samples vectors
  if (udata->samples_local)
  {
    N_VDestroy(udata->samples_local);
    udata->samples_local = NULL;
  }

  // Free temporary vectors
  if (udata->px)
  {
    N_VDestroy(udata->px);
    udata->px = NULL;
  }
  if (udata->mu_bottom)
  {
    N_VDestroy(udata->mu_bottom);
    udata->mu_bottom = NULL;
  }
  if (udata->mu_top)
  {
    N_VDestroy(udata->mu_top);
    udata->mu_top = NULL;
  }
  if (udata->mu_true)
  {
    N_VDestroy(udata->mu_true);
    udata->mu_true = NULL;
  }

  // Free error vector
  if (udata->vtemp)
  {
    N_VDestroy(udata->vtemp);
    udata->vtemp = NULL;
  }

  // Free MPI communicator
  udata->comm = MPI_COMM_NULL;

  // Return success
  return 0;
}

// Read command line inputs
static int ReadInputs(int *argc, char ***argv, UserData *udata, bool outproc)
{
  // Check for input args
  int arg_idx = 1;

  while (arg_idx < (*argc))
  {
    string arg = (*argv)[arg_idx++];

    // Mesh points
    if (arg == "--nodes_loc")
    {
      udata->nodes_loc = stoi((*argv)[arg_idx++]);
    }
    // Fixed Point settings
    else if (arg == "--rtol")
    {
      udata->rtol = stod((*argv)[arg_idx++]);
    }
    else if (arg == "--maa")
    {
      udata->maa = stoi((*argv)[arg_idx++]);
    }
    else if (arg == "--damping")
    {
      udata->damping = stod((*argv)[arg_idx++]);
    }
    else if (arg == "--orthaa")
    {
      udata->orthaa = stoi((*argv)[arg_idx++]);
    }
    else if (arg == "--maxits")
    {
      udata->maxits = stoi((*argv)[arg_idx++]);
    }
    // Output settings
    else if (arg == "--output")
    {
      udata->output = stoi((*argv)[arg_idx++]);
    }
    else if (arg == "--timing")
    {
      udata->timing = true;
    }
    else if (arg == "--debug")
    {
      if (SUNDIALS_LOGGING_LEVEL > 4) {
        udata->debug = true;
      } else {
        cerr << "ERROR: SUNDIALS_LOGGING_LEVEL set too low for debug output" << endl;
      }
    }
    // Help
    else if (arg == "--help")
    {
      if (outproc) InputHelp();
      return -1;
    }
    // Unknown input
    else
    {
      if (outproc)
      {
        cerr << "ERROR: Invalid input " << arg << endl;
        InputHelp();
      }
      return -1;
    }
  }

  // Recompute local number of nodes
  udata->nodes = udata->nodes_loc * udata->nprocs_w;

  // Return success
  return 0;
}

// -----------------------------------------------------------------------------
// Output and utility functions
// -----------------------------------------------------------------------------

// Compute the solution error
static int SolutionError(N_Vector u_true, N_Vector u, N_Vector err,
                         UserData *udata)
{
  // Put true solution in error vector
  SetMus(udata);

  // Compute absolute error
  N_VLinearSum(ONE, u_true, -ONE, u, err);
  N_VAbs(err, err);

  return 0;
}

// Print command line options
static void InputHelp()
{
  cout << endl;
  cout << "Command line options:" << endl;
  cout << "  --nodes                 : global number of values in vector" << endl;
  cout << "  --rtol <rtol>           : relative tolerance" << endl;
  cout << "  --maa                   : size of Anderson Acceleration subspace" << endl;
  cout << "  --damping               : damping for Anderson Acceleration" << endl;
  cout << "  --orthaa                : orthogonalization routined used in Anderson Acceleration" << endl;
  cout << "  --maxits <iterations>   : max fixed point iterations" << endl;
  cout << "  --output                : output nonlinear solver statistics" << endl;
  cout << "  --timing                : print timing data" << endl;
  cout << "  --help                  : print this message and exit" << endl;
}

// Print user data
static int PrintUserData(UserData *udata)
{
  cout << endl;
  cout << "Expectation-Maximizaton Alg. for Mixture Densities Terms:" << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << "  nprocs             = " << udata->nprocs_w                << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << "  nodes              = " << udata->nodes                   << endl;
  cout << "  nodes_loc (proc 0) = " << udata->nodes_loc               << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << "  sigma              = {"
             << udata->sigma << ", " << udata->sigma << ", "
             << udata->sigma << "}" << endl;
  cout << "  alpha              = {"
             << udata->alpha1 << ", " << udata->alpha2 << ", "
             << udata->alpha3 << "}" << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << "  rtol               = " << udata->rtol                        << endl;
  cout << "  maa                = " << udata->maa                         << endl;
  cout << "  damping            = " << udata->damping                     << endl;
  cout << "  orthaa             = " << udata->orthaa                      << endl;
  cout << "  maxits             = " << udata->maxits                      << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << "  output             = " << udata->output                      << endl;
  cout << " ------------------------------------------------------- " << endl;
  cout << endl;

  return 0;
}

// Print nonlinear solver statistics
static int OutputStats(void *kinsol_mem, UserData* udata)
{
  int retval;

  // Get solver stats
  long int nfe, nni;
  retval = KINGetNumNonlinSolvIters(kinsol_mem, &nni);
  if (check_retval(&retval, "KINGetNumNonlinSolvIters", 1)) return(1);
  retval = KINGetNumFuncEvals(kinsol_mem, &nfe);
  if (check_retval(&retval, "KINGetNumFuncEvals", 1)) return(1);

  cout << setprecision(6);

  cout << "  Func evals       = " << nfe     << endl;
  cout << "  NLS iters        = " << nni     << endl;
  cout << endl;

  return 0;
}

static int OutputTiming(UserData *udata)
{
  bool outproc = (udata->myid == 0);

  if (outproc)
  {
    cout << scientific;
    cout << setprecision(6);
  }

  double maxtime = 0.0;

  MPI_Reduce(&(udata->totaltime), &maxtime, 1, MPI_DOUBLE, MPI_MAX, 0,
             udata->comm);
  if (outproc)
  {
    cout << "  Total time                = " << maxtime << " sec" << endl;
  }

  MPI_Reduce(&(udata->fevaltime), &maxtime, 1, MPI_DOUBLE, MPI_MAX, 0,
             udata->comm);
  if (outproc)
  {
    cout << "  Function evaluation time  = " << maxtime << " sec" << endl;
  }

  return 0;
}

// Open residual and error output
static int OpenOutput(UserData *udata)
{
  bool outproc = (udata->myid == 0);

  if (outproc)
  {
    stringstream fname;

    // Open output stream for residual
    fname.str("");
    fname.clear();
    fname << "EM_res_m" << udata->maa << "_orth" << udata->orthaa
          << "_len" << udata->nodes_loc << ".txt";
    udata->rout.open(fname.str());

    udata->rout << scientific;
    udata->rout << setprecision(numeric_limits<realtype>::digits10);

    // Open output stream for error
    fname.str("");
    fname.clear();
    fname << "EM_err_m" << udata->maa << "_orth" << udata->orthaa
          << "_len" << udata->nodes_loc << ".txt";
    udata->eout.open(fname.str());

    udata->eout << scientific;
    udata->eout << setprecision(numeric_limits<realtype>::digits10);
  }

  return 0;
}

// Write residual and error out to file
static int WriteOutput(N_Vector u, N_Vector f, UserData *udata)
{
  int retval;
  bool outproc = (udata->myid == 0);

  // r = \|G(u) - u\|_inf
  N_VLinearSum(ONE, f, -ONE, u, udata->vtemp);
  realtype res = N_VMaxNorm(udata->vtemp);

  // e = \|u_exact - u\|_inf
  retval = SolutionError(udata->mu_true, u, udata->vtemp, udata);
  if (check_retval(&retval, "SolutionError", 1)) return 1;
  realtype err = N_VMaxNorm(udata->vtemp);

  if (outproc)
  {
    // Output residual
    udata->rout << res;
    udata->rout << endl;

    // Output error
    udata->eout << err;
    udata->eout << endl;
  }

  return 0;
}

// Close residual and error output files
static int CloseOutput(UserData *udata)
{
  bool outproc = (udata->myid == 0);

  if (outproc)
  {
    // Close output streams
    udata->rout.close();
    udata->eout.close();
  }

  return 0;
}

// Check function return value
static int check_retval(void *flagvalue, const string funcname, int opt)
{
  // Check if the function returned a NULL pointer
  if (opt == 0)
  {
    if (flagvalue == NULL)
    {
      cerr << endl << "ERROR: " << funcname << " returned NULL pointer" << endl
           << endl;
      return 1;
    }
  }
  // Check the function return value
  else if (opt == 1 || opt == 2)
  {
    int errflag = *((int *) flagvalue);
    if  ((opt == 1 && errflag < 0) || (opt == 2 && errflag != 0))
    {
      cerr << endl << "ERROR: " << funcname << " returned = "
           << errflag << endl << endl;
      return 1;
    }
  }
  else
  {
    cerr << endl << "ERROR: check_retval called with an invalid option value"
         << endl;
    return 1;
  }

  return 0;
}

//---- end of file ----
