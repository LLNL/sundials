#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR Raja module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <nvector/nvector_mpiplusx.h>
#include <nvector/nvector_raja.h>
#include <nvector/raja/Vector.hpp>
#include <sundials/sundials_math.h>
#include "test_nvector.h"

#include <mpi.h>

using namespace sunrajavec;

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;           /* counter for test failures */
  int          globfails = 0;       /* counter for test failures */
  int          retval;              /* function return value     */
  sunindextype local_length;        /* local vector length       */
  sunindextype global_length;       /* global vector length      */
  N_Vector     U, V, X;             /* local test vectors        */
  N_Vector     plusU, plusV, plusX; /* MPIPlusX test vectors     */
  N_Vector     plusY, plusZ;        /* MPIPlusX test vectors     */
  int          print_timing;        /* turn timing on/off        */
  MPI_Comm     comm;                /* MPI Communicator          */
  int          nprocs, myid;        /* Number of procs, proc id  */

  /* Get processor number and total number of processes */
  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &nprocs);
  MPI_Comm_rank(comm, &myid);

  /* check inputs */
  if (argc < 3) {
    if (myid == 0)
      printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    MPI_Abort(comm, -1);
  }

  local_length = (sunindextype) atol(argv[1]);
  if (local_length < 1) {
    if (myid == 0)
      printf("ERROR: local vector length must be a positive integer \n");
    MPI_Abort(comm, -1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, myid);

  /* global length */
  global_length = nprocs*local_length;

  if (myid == 0) {
    printf("Testing the RAJA N_Vector \n");
    printf("Vector global length %ld \n", (long int) global_length);
    printf("MPI processes %d \n", nprocs);
  }

  /* Create new vectors */
  X = N_VNew_Raja(local_length);
  if (X == NULL) {
    if (myid == 0) printf("FAIL: Unable to create a new RAJA vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* Create the MPI+X vector */
  plusX = N_VMake_MPIPlusX(comm, X);
  if (plusX == NULL) {
    N_VDestroy(X);
    if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* Check vector ID */
  fails += Test_N_VGetVectorID(plusX, SUNDIALS_NVEC_MPIPLUSX, myid);

  /* Check vector length */
  fails += Test_N_VGetLength(plusX, myid);

  /* Check vector communicator */
  fails += Test_N_VGetCommunicator(plusX, &comm, myid);

  /* Test clone functions */
  fails += Test_N_VCloneEmpty(plusX, myid);
  fails += Test_N_VClone(plusX, local_length, myid);
  fails += Test_N_VCloneEmptyVectorArray(5, plusX, myid);
  fails += Test_N_VCloneVectorArray(5, plusX, local_length, myid);

  /* Clone additional vectors for testing */
  plusY = N_VClone(plusX);
  if (plusY == NULL) {
    N_VDestroy(X);
    N_VDestroy(plusX);
    if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
    MPI_Abort(comm, 1);
  }

  plusZ = N_VClone(plusX);
  if (plusZ == NULL) {
    N_VDestroy(X);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* Standard vector operation tests */
  if (myid == 0) printf("\nTesting standard vector operations:\n\n");

  fails += Test_N_VConst(plusX, local_length, myid);
  fails += Test_N_VLinearSum(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VProd(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VDiv(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VScale(plusX, plusZ, local_length, myid);
  fails += Test_N_VAbs(plusX, plusZ, local_length, myid);
  fails += Test_N_VInv(plusX, plusZ, local_length, myid);
  fails += Test_N_VAddConst(plusX, plusZ, local_length, myid);
  fails += Test_N_VDotProd(plusX, plusY, local_length, myid);
  fails += Test_N_VMaxNorm(plusX, local_length, myid);
  fails += Test_N_VWrmsNorm(plusX, plusY, local_length, myid);
  fails += Test_N_VWrmsNormMask(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VMin(plusX, local_length, myid);
  fails += Test_N_VWL2Norm(plusX, plusY, local_length, myid);
  fails += Test_N_VL1Norm(plusX, local_length, myid);
  fails += Test_N_VCompare(plusX, plusZ, local_length, myid);
  fails += Test_N_VInvTest(plusX, plusZ, local_length, myid);
  fails += Test_N_VConstrMask(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VMinQuotient(plusX, plusY, local_length, myid);

  /* Fused and vector array operations tests (disabled) */
  if (myid == 0) printf("\nTesting fused and vector array operations (disabled):\n\n");

  /* create vector and disable all fused and vector array operations */
  U = N_VNew_Raja(local_length);
  retval = N_VEnableFusedOps_Raja(U, SUNFALSE);
  if (U == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    N_VDestroy(plusZ);
    if (myid == 0) printf("FAIL: Unable to create a new RAJA vector \n\n");
    MPI_Abort(comm, 1);
  }

  plusU = N_VMake_MPIPlusX(comm, U);
  if (U == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(U);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    N_VDestroy(plusZ);
    if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* fused operations */
  fails += Test_N_VLinearCombination(plusU, local_length, myid);
  fails += Test_N_VScaleAddMulti(plusU, local_length, myid);
  fails += Test_N_VDotProdMulti(plusU, local_length, myid);

  /* vector array operations */
  fails += Test_N_VLinearSumVectorArray(plusU, local_length, myid);
  fails += Test_N_VScaleVectorArray(plusU, local_length, myid);
  fails += Test_N_VConstVectorArray(plusU, local_length, myid);
  fails += Test_N_VWrmsNormVectorArray(plusU, local_length, myid);
  fails += Test_N_VWrmsNormMaskVectorArray(plusU, local_length, myid);
  fails += Test_N_VScaleAddMultiVectorArray(plusU, local_length, myid);
  fails += Test_N_VLinearCombinationVectorArray(plusU, local_length, myid);

  /* Fused and vector array operations tests (enabled) */
  if (myid == 0) printf("\nTesting fused and vector array operations (enabled):\n\n");

  /* create vector and enable all fused and vector array operations */
  V = N_VNew_Raja(local_length);
  retval = N_VEnableFusedOps_Raja(V, SUNTRUE);
  if (V == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(U);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    N_VDestroy(plusZ);
    N_VDestroy(plusU);
    if (myid == 0) printf("FAIL: Unable to create a new RAJA vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* create the MPIPlusX vector */
  plusV = N_VMake_MPIPlusX(comm, V);
  if (V == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(U);
    N_VDestroy(V);
    N_VDestroy(plusU);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    N_VDestroy(plusZ);
    if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
    MPI_Abort(comm, 1);
  }

  /* fused operations */
  fails += Test_N_VLinearCombination(plusV, local_length, myid);
  fails += Test_N_VScaleAddMulti(plusV, local_length, myid);
  fails += Test_N_VDotProdMulti(plusV, local_length, myid);

  /* vector array operations */
  fails += Test_N_VLinearSumVectorArray(plusV, local_length, myid);
  fails += Test_N_VScaleVectorArray(plusV, local_length, myid);
  fails += Test_N_VConstVectorArray(plusV, local_length, myid);
  fails += Test_N_VWrmsNormVectorArray(plusV, local_length, myid);
  fails += Test_N_VWrmsNormMaskVectorArray(plusV, local_length, myid);
  fails += Test_N_VScaleAddMultiVectorArray(plusV, local_length, myid);
  fails += Test_N_VLinearCombinationVectorArray(plusV, local_length, myid);

  /* local reduction operations */
  printf("\nTesting local reduction operations:\n\n");

  fails += Test_N_VDotProdLocal(plusX, plusY, local_length, myid);
  fails += Test_N_VMaxNormLocal(plusX, local_length, myid);
  fails += Test_N_VMinLocal(plusX, local_length, myid);
  fails += Test_N_VL1NormLocal(plusX, local_length, myid);
  fails += Test_N_VWSqrSumLocal(plusX, plusY, local_length, myid);
  fails += Test_N_VWSqrSumMaskLocal(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VInvTestLocal(plusX, plusZ, local_length, myid);
  fails += Test_N_VConstrMaskLocal(plusX, plusY, plusZ, local_length, myid);
  fails += Test_N_VMinQuotientLocal(plusX, plusY, local_length, myid);

  /* Free vectors */
  N_VDestroy(X);
  N_VDestroy(U);
  N_VDestroy(V);
  N_VDestroy(plusX);
  N_VDestroy(plusY);
  N_VDestroy(plusZ);
  N_VDestroy(plusU);
  N_VDestroy(plusV);

  /* Print result */
  if (fails) {
    printf("FAIL: NVector module failed %i tests, Proc %d \n\n", fails, myid);
  } else {
    if (myid == 0)
      printf("SUCCESS: NVector module passed all tests \n\n");
  }

  /* check if any other process failed */
  (void) MPI_Allreduce(&fails, &globfails, 1, MPI_INT, MPI_MAX, comm);

  MPI_Finalize();

  return(globfails);
}

/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector plusX, sunindextype local_length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;
  N_Vector     X;

  X = N_VGetLocalVector_MPIPlusX(plusX);
  N_VCopyFromDevice_Raja(X);
  Xdata = N_VGetHostArrayPointer_Raja(X);

  /* check vector data */
  for (i = 0; i < local_length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector plusX)
{
  return (N_VGetLocalVector_MPIPlusX(plusX)->content == NULL) ? SUNFALSE : SUNTRUE;
}

void set_element(N_Vector plusX, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(plusX, i, i, val);
}

void set_element_range(N_Vector plusX, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;
  N_Vector     X;

  X = N_VGetLocalVector_MPIPlusX(plusX);

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Raja(X);
  xd = N_VGetHostArrayPointer_Raja(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Raja(X);
}

realtype get_element(N_Vector plusX, sunindextype i)
{
  N_Vector X = N_VGetLocalVector_MPIPlusX(plusX);

  /* get i-th element of data array */
  N_VCopyFromDevice_Raja(X);
  return (N_VGetHostArrayPointer_Raja(X))[i];
}

double max_time(N_Vector plusX, double time)
{
  MPI_Comm *comm;
  double maxt;

  comm = (MPI_Comm*) N_VGetCommunicator(plusX);

  /* get max time across all MPI ranks */
  (void) MPI_Reduce(&time, &maxt, 1, MPI_DOUBLE, MPI_MAX, 0, *comm);
  return(maxt);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}
