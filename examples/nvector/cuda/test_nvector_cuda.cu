/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR CUDA module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_math.h>
#include <sundials/sundials_types.h>
#include <nvector/nvector_cuda.h>
#include <nvector/cuda/ThreadPartitioning.hpp>
#include "test_nvector.h"


using namespace suncudavec;

/* private custom allocator functions */
static void* sunalloc(size_t);
static void sunfree(void* ptr);

/* CUDA vector specific tests */
static int Test_N_VMake_Cuda(N_Vector X, sunindextype length, int myid);
static int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype length, int myid);

/* CUDA vector can use unmanaged or managed memory */
enum mem_type { UNMANAGED, MANAGED, CUSTOM };

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;         /* counter for test failures */
  int          retval;            /* function return value     */
  sunindextype length;            /* vector length             */
  N_Vector     U, V, X, Y, Z;     /* test vectors              */
  int          print_timing;      /* turn timing on/off        */
  int          i;

  /* check input and set vector length */
  if (argc < 3){
    printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    return(-1);
  }

  length = (sunindextype) atol(argv[1]);
  if (length <= 0) {
    printf("ERROR: length of vector must be a positive integer \n");
    return(-1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, 0);

  /* test with unmanaged and managed memory */
  for (i=UNMANAGED; i<=CUSTOM; ++i) {
    if (i==UNMANAGED) {
      printf("Testing CUDA N_Vector \n");
    } else if (i==MANAGED) {
      printf("\nTesting CUDA N_Vector with managed memory \n");
    } else {
      printf("\nTesting CUDA N_Vector with custom allocator \n");
    }
    printf("Vector length %ld \n\n", (long int) length);

    /* Create new vectors */
    if (i == UNMANAGED)    X = N_VNew_Cuda(length);
    else if (i == MANAGED) X = N_VNewManaged_Cuda(length);
    else                   X = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
    if (X == NULL) {
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* Check vector ID */
    fails += Test_N_VGetVectorID(X, SUNDIALS_NVEC_CUDA, 0);

    /* Check vector length */
    fails += Test_N_VGetLength(X, 0);

    /* Check vector communicator */
    fails += Test_N_VGetCommunicator(X, NULL, 0);

    /* Test clone functions */
    fails += Test_N_VCloneEmpty(X, 0);
    fails += Test_N_VClone(X, length, 0);
    fails += Test_N_VCloneEmptyVectorArray(5, X, 0);
    fails += Test_N_VCloneVectorArray(5, X, length, 0);

    /* Clone additional vectors for testing */
    Y = N_VClone(X);
    if (Y == NULL) {
      N_VDestroy(X);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    Z = N_VClone(X);
    if (Z == NULL) {
      N_VDestroy(X);
      N_VDestroy(Y);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* Standard vector operation tests */
    printf("\nTesting standard vector operations:\n\n");

    fails += Test_N_VConst(X, length, 0);
    fails += Test_N_VLinearSum(X, Y, Z, length, 0);
    fails += Test_N_VProd(X, Y, Z, length, 0);
    fails += Test_N_VDiv(X, Y, Z, length, 0);
    fails += Test_N_VScale(X, Z, length, 0);
    fails += Test_N_VAbs(X, Z, length, 0);
    fails += Test_N_VInv(X, Z, length, 0);
    fails += Test_N_VAddConst(X, Z, length, 0);
    fails += Test_N_VDotProd(X, Y, length, 0);
    fails += Test_N_VMaxNorm(X, length, 0);
    fails += Test_N_VWrmsNorm(X, Y, length, 0);
    fails += Test_N_VWrmsNormMask(X, Y, Z, length, 0);
    fails += Test_N_VMin(X, length, 0);
    fails += Test_N_VWL2Norm(X, Y, length, 0);
    fails += Test_N_VL1Norm(X, length, 0);
    fails += Test_N_VCompare(X, Z, length, 0);
    fails += Test_N_VInvTest(X, Z, length, 0);
    fails += Test_N_VConstrMask(X, Y, Z, length, 0);
    fails += Test_N_VMinQuotient(X, Y, length, 0);

    /* Fused and vector array operations tests (disabled) */
    printf("\nTesting fused and vector array operations (disabled):\n\n");

    /* create vector and disable all fused and vector array operations */
    if (i == UNMANAGED)    U = N_VNew_Cuda(length);
    else if (i == MANAGED) U = N_VNewManaged_Cuda(length);
    else                   U = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
    if (U == NULL) {
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }
    retval = N_VEnableFusedOps_Cuda(U, SUNFALSE);
    if (retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(U, length, 0);
    fails += Test_N_VScaleAddMulti(U, length, 0);
    fails += Test_N_VDotProdMulti(U, length, 0);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(U, length, 0);
    fails += Test_N_VScaleVectorArray(U, length, 0);
    fails += Test_N_VConstVectorArray(U, length, 0);
    fails += Test_N_VWrmsNormVectorArray(U, length, 0);
    fails += Test_N_VWrmsNormMaskVectorArray(U, length, 0);
    fails += Test_N_VScaleAddMultiVectorArray(U, length, 0);
    fails += Test_N_VLinearCombinationVectorArray(U, length, 0);

    /* Fused and vector array operations tests (enabled) */
    printf("\nTesting fused and vector array operations (enabled):\n\n");

    /* create vector and enable all fused and vector array operations */
    if (i == UNMANAGED)    V = N_VNew_Cuda(length);
    else if (i == MANAGED) V = N_VNewManaged_Cuda(length);
    else                   V = N_VMakeWithManagedAllocator_Cuda(length, sunalloc, sunfree);
    retval = N_VEnableFusedOps_Cuda(V, SUNTRUE);
    if (V == NULL) {
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }
    if (retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      N_VDestroy(U);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(V, length, 0);
    fails += Test_N_VScaleAddMulti(V, length, 0);
    fails += Test_N_VDotProdMulti(V, length, 0);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(V, length, 0);
    fails += Test_N_VScaleVectorArray(V, length, 0);
    fails += Test_N_VConstVectorArray(V, length, 0);
    fails += Test_N_VWrmsNormVectorArray(V, length, 0);
    fails += Test_N_VWrmsNormMaskVectorArray(V, length, 0);
    fails += Test_N_VScaleAddMultiVectorArray(V, length, 0);
    fails += Test_N_VLinearCombinationVectorArray(V, length, 0);

    /* local reduction operations */
    printf("\nTesting local reduction operations:\n\n");

    fails += Test_N_VDotProdLocal(X, Y, length, 0);
    fails += Test_N_VMaxNormLocal(X, length, 0);
    fails += Test_N_VMinLocal(X, length, 0);
    fails += Test_N_VL1NormLocal(X, length, 0);
    fails += Test_N_VWSqrSumLocal(X, Y, length, 0);
    fails += Test_N_VWSqrSumMaskLocal(X, Y, Z, length, 0);
    fails += Test_N_VInvTestLocal(X, Z, length, 0);
    fails += Test_N_VConstrMaskLocal(X, Y, Z, length, 0);
    fails += Test_N_VMinQuotientLocal(X, Y, length, 0);

    /* CUDA specific tests */
    printf("\nTesting cuda vector specific operations:\n\n");
    if (i==UNMANAGED) {
      fails += Test_N_VMake_Cuda(X, length, 0);
    } else if (i==MANAGED) {
      fails += Test_N_VMakeManaged_Cuda(X, length, 0);
    }

    /* Free vectors */
    N_VDestroy(X);
    N_VDestroy(Y);
    N_VDestroy(Z);
    N_VDestroy(U);
    N_VDestroy(V);
  }

  /* Print result */
  if (fails) {
    printf("\n\nFAIL: NVector module failed %i tests \n\n", fails);
  } else {
    printf("\n\nSUCCESS: NVector module passed all tests \n\n");
  }

  return(fails);
}


/* ----------------------------------------------------------------------
 * CUDA specific tests
 * --------------------------------------------------------------------*/

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMake_Cuda function. Requires N_VConst
 * to check data.
 */
int Test_N_VMake_Cuda(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *h_data, *d_data;
  N_Vector Y;

  N_VConst(NEG_HALF, X);
  N_VCopyFromDevice_Cuda(X);

  h_data = N_VGetHostArrayPointer_Cuda(X);
  d_data = N_VGetDeviceArrayPointer_Cuda(X);

  /* Case 1: h_data and d_data are not null */
  Y = N_VMake_Cuda(length, h_data, d_data);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  if (N_VGetHostArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector host data == NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (N_VGetDeviceArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector device data -= NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);

  if (failure) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 1 \n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMake_Cuda(length, NULL, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 2 \n");
  }

  N_VDestroy(Y);

  return(failure);
}

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMakeManaged_Cuda function. Requires
 * N_VConst to check data. X must be using managed memory.
 */
int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *vdata;
  N_Vector Y;

  if(!N_VIsManagedMemory_Cuda(X)) {
    printf(">>> FAILED test -- N_VIsManagedMemory_Cuda, Proc %d \n", myid);
    return(1);
  }

  N_VConst(NEG_HALF, X);
  vdata = N_VGetHostArrayPointer_Cuda(X);

  /* Case 1: data is not null */
  Y = N_VMakeManaged_Cuda(length, vdata);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);

  /* Case 2: data is null */
  Y = N_VMakeManaged_Cuda(length, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Cuda Case 2 \n");
  }

  N_VDestroy(Y);

  return(failure);
}


/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector X, sunindextype length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;

  N_VCopyFromDevice_Cuda(X);
  Xdata = N_VGetHostArrayPointer_Cuda(X);

  /* check vector data */
  for (i = 0; i < length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector X)
{
  /* check if vector content is non-null */
  return (X->content == NULL ? SUNFALSE : SUNTRUE);
}

void set_element(N_Vector X, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(X, i, i, val);
}

void set_element_range(N_Vector X, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Cuda(X);
  xd = N_VGetHostArrayPointer_Cuda(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Cuda(X);
}

realtype get_element(N_Vector X, sunindextype i)
{
  /* get i-th element of data array */
  N_VCopyFromDevice_Cuda(X);
  return (N_VGetHostArrayPointer_Cuda(X))[i];
}

double max_time(N_Vector X, double time)
{
  /* not running in parallel, just return input time */
  return(time);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}

void* sunalloc(size_t mem_size)
{
  void* ptr;
  hipError_t err;
  err = hipMallocManaged(&ptr, mem_size);
  if (err != hipSuccess) {
    printf("Error in sunalloc\n");
    ptr = NULL;
  }
  return ptr;
}

void sunfree(void* ptr)
{
  hipFree(ptr);
}
