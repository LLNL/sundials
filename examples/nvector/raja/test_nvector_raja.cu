/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR Raja module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <nvector/nvector_raja.h>
#include <sundials/sundials_math.h>
#include "test_nvector.h"

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;         /* counter for test failures */
  int          retval;            /* function return value     */
  sunindextype length;            /* vector length             */
  N_Vector     U, V, X, Y, Z;     /* test vectors              */
  int          print_timing;      /* turn timing on/off        */


  /* check input and set vector length */
  if (argc < 3){
    printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    return(-1);
  }

  length = (sunindextype) atol(argv[1]);
  if (length <= 0) {
    printf("ERROR: length of vector must be a positive integer \n");
    return(-1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, 0);

  printf("Testing RAJA N_Vector \n");
  printf("Vector length %ld \n\n", (long int) length);

  /* Create new vectors */
  X = N_VNew_Raja(length);
  if (X == NULL) {
    printf("FAIL: Unable to create a new vector \n\n");
    return(1);
  }

  /* Check vector ID */
  fails += Test_N_VGetVectorID(X, SUNDIALS_NVEC_RAJA, 0);

  /* Check vector length */
  fails += Test_N_VGetLength(X, 0);

  /* Check vector communicator */
  fails += Test_N_VGetCommunicator(X, NULL, 0);

  /* Test clone functions */
  fails += Test_N_VCloneEmpty(X, 0);
  fails += Test_N_VClone(X, length, 0);
  fails += Test_N_VCloneEmptyVectorArray(5, X, 0);
  fails += Test_N_VCloneVectorArray(5, X, length, 0);

  /* Clone additional vectors for testing */
  Y = N_VClone(X);
  if (Y == NULL) {
    N_VDestroy(X);
    printf("FAIL: Unable to create a new vector \n\n");
    return(1);
  }

  Z = N_VClone(X);
  if (Z == NULL) {
    N_VDestroy(X);
    N_VDestroy(Y);
    printf("FAIL: Unable to create a new vector \n\n");
    return(1);
  }

  /* Standard vector operation tests */
  printf("\nTesting standard vector operations:\n\n");

  fails += Test_N_VConst(X, length, 0);
  fails += Test_N_VLinearSum(X, Y, Z, length, 0);
  fails += Test_N_VProd(X, Y, Z, length, 0);
  fails += Test_N_VDiv(X, Y, Z, length, 0);
  fails += Test_N_VScale(X, Z, length, 0);
  fails += Test_N_VAbs(X, Z, length, 0);
  fails += Test_N_VInv(X, Z, length, 0);
  fails += Test_N_VAddConst(X, Z, length, 0);
  fails += Test_N_VDotProd(X, Y, length, 0);
  fails += Test_N_VMaxNorm(X, length, 0);
  fails += Test_N_VWrmsNorm(X, Y, length, 0);
  fails += Test_N_VWrmsNormMask(X, Y, Z, length, 0);
  fails += Test_N_VMin(X, length, 0);
  fails += Test_N_VWL2Norm(X, Y, length, 0);
  fails += Test_N_VL1Norm(X, length, 0);
  fails += Test_N_VCompare(X, Z, length, 0);
  fails += Test_N_VInvTest(X, Z, length, 0);
  fails += Test_N_VConstrMask(X, Y, Z, length, 0);
  fails += Test_N_VMinQuotient(X, Y, length, 0);

  /* Fused and vector array operations tests (disabled) */
  printf("\nTesting fused and vector array operations (disabled):\n\n");

  /* create vector and disable all fused and vector array operations */
  U = N_VNew_Raja(length);
  retval = N_VEnableFusedOps_Raja(U, SUNFALSE);
  if (U == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(Y);
    N_VDestroy(Z);
    printf("FAIL: Unable to create a new vector \n\n");
    return(1);
  }

  /* fused operations */
  fails += Test_N_VLinearCombination(U, length, 0);
  fails += Test_N_VScaleAddMulti(U, length, 0);
  fails += Test_N_VDotProdMulti(U, length, 0);

  /* vector array operations */
  fails += Test_N_VLinearSumVectorArray(U, length, 0);
  fails += Test_N_VScaleVectorArray(U, length, 0);
  fails += Test_N_VConstVectorArray(U, length, 0);
  fails += Test_N_VWrmsNormVectorArray(U, length, 0);
  fails += Test_N_VWrmsNormMaskVectorArray(U, length, 0);
  fails += Test_N_VScaleAddMultiVectorArray(U, length, 0);
  fails += Test_N_VLinearCombinationVectorArray(U, length, 0);

  /* Fused and vector array operations tests (enabled) */
  printf("\nTesting fused and vector array operations (enabled):\n\n");

  /* create vector and enable all fused and vector array operations */
  V = N_VNew_Raja(length);
  retval = N_VEnableFusedOps_Raja(V, SUNTRUE);
  if (V == NULL || retval != 0) {
    N_VDestroy(X);
    N_VDestroy(Y);
    N_VDestroy(Z);
    N_VDestroy(U);
    printf("FAIL: Unable to create a new vector \n\n");
    return(1);
  }

  /* fused operations */
  fails += Test_N_VLinearCombination(V, length, 0);
  fails += Test_N_VScaleAddMulti(V, length, 0);
  fails += Test_N_VDotProdMulti(V, length, 0);

  /* vector array operations */
  fails += Test_N_VLinearSumVectorArray(V, length, 0);
  fails += Test_N_VScaleVectorArray(V, length, 0);
  fails += Test_N_VConstVectorArray(V, length, 0);
  fails += Test_N_VWrmsNormVectorArray(V, length, 0);
  fails += Test_N_VWrmsNormMaskVectorArray(V, length, 0);
  fails += Test_N_VScaleAddMultiVectorArray(V, length, 0);
  fails += Test_N_VLinearCombinationVectorArray(V, length, 0);

  /* local reduction operations */
  printf("\nTesting local reduction operations:\n\n");

  fails += Test_N_VDotProdLocal(X, Y, length, 0);
  fails += Test_N_VMaxNormLocal(X, length, 0);
  fails += Test_N_VMinLocal(X, length, 0);
  fails += Test_N_VL1NormLocal(X, length, 0);
  fails += Test_N_VWSqrSumLocal(X, Y, length, 0);
  fails += Test_N_VWSqrSumMaskLocal(X, Y, Z, length, 0);
  fails += Test_N_VInvTestLocal(X, Z, length, 0);
  fails += Test_N_VConstrMaskLocal(X, Y, Z, length, 0);
  fails += Test_N_VMinQuotientLocal(X, Y, length, 0);

  /* Free vectors */
  N_VDestroy(X);
  N_VDestroy(Y);
  N_VDestroy(Z);
  N_VDestroy(U);
  N_VDestroy(V);

  /* Print result */
  if (fails) {
    printf("FAIL: NVector module failed %i tests \n\n", fails);
  } else {
    printf("SUCCESS: NVector module passed all tests \n\n");
  }

  return(fails);
}

/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector X, sunindextype local_length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;

  N_VCopyFromDevice_Raja(X);
  Xdata = N_VGetHostArrayPointer_Raja(X);

  /* check vector data */
  for (i = 0; i < local_length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector X)
{
  /* check if vector content is non-null */
  return (X->content == NULL ? SUNFALSE : SUNTRUE);
}

void set_element(N_Vector X, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(X, i, i, val);
}

void set_element_range(N_Vector X, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Raja(X);
  xd = N_VGetHostArrayPointer_Raja(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Raja(X);
}

realtype get_element(N_Vector X, sunindextype i)
{
  /* get i-th element of data array */
  N_VCopyFromDevice_Raja(X);
  return (N_VGetHostArrayPointer_Raja(X))[i];
}

double max_time(N_Vector X, double time)
{
  /* not running in parallel, just return input time */
  return(time);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}
