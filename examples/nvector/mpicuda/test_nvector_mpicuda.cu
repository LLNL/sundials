#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the MPIPlusX NVECTOR where
 * the X is the CUDA NVECTOR.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <nvector/cuda/Vector.hpp>
#include <nvector/nvector_cuda.h>
#include <nvector/nvector_mpiplusx.h>
#include <sundials/sundials_math.h>
#include "test_nvector.h"

#include <mpi.h>

/* CUDA vector can use unmanaged or managed memory */
enum mem_type { UNMANAGED, MANAGED };

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;           /* counter for test failures */
  int          globfails = 0;       /* counter for test failures */
  int          retval;              /* function return value     */
  sunindextype local_length;        /* local vector length       */
  sunindextype global_length;       /* global vector length      */
  N_Vector     U, V, X;             /* local test vectors        */
  N_Vector     plusU, plusV, plusX; /* MPIPlusX test vectors     */
  N_Vector     plusY, plusZ;        /* MPIPlusX test vectors     */
  int          print_timing;        /* turn timing on/off        */
  MPI_Comm     comm;                /* MPI Communicator          */
  int          nprocs, myid;        /* Number of procs, proc id  */
  int          i;

  /* Get processor number and total number of processes */
  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &nprocs);
  MPI_Comm_rank(comm, &myid);

  /* check inputs */
  if (argc < 3) {
    if (myid == 0)
      printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    MPI_Abort(comm, -1);
  }

  local_length = (sunindextype) atol(argv[1]);
  if (local_length < 1) {
    if (myid == 0)
      printf("ERROR: local vector length must be a positive integer \n");
    MPI_Abort(comm, -1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, myid);

  /* global length */
  global_length = nprocs*local_length;

  for (i=UNMANAGED; i<=MANAGED; ++i) {
    if (myid == 0) {
      if (i==UNMANAGED) {
        printf("Testing CUDA N_Vector \n");
      } else {
        printf("\nTesting CUDA N_Vector with managed memory \n");
      }
      printf("Vector global length %ld \n", (long int) global_length);
      printf("MPI processes %d \n", nprocs);
    }

    /* Create new local vectors */
    X = (i==UNMANAGED) ? N_VNew_Cuda(local_length) : N_VNewManaged_Cuda(local_length);
    if (X == NULL) {
      if (myid == 0) printf("FAIL: Unable to create a new CUDA vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* Create the MPI+X vector */
    plusX = N_VMake_MPIPlusX(comm, X);
    if (plusX == NULL) {
      N_VDestroy(X);
      if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* Check vector ID */
    fails += Test_N_VGetVectorID(plusX, SUNDIALS_NVEC_MPIPLUSX, myid);

    /* Check vector length */
    fails += Test_N_VGetLength(plusX, myid);

    /* Check vector communicator */
    fails += Test_N_VGetCommunicatorMPI(plusX, &comm, myid);

    /* Test clone functions */
    fails += Test_N_VCloneEmpty(plusX, myid);
    fails += Test_N_VClone(plusX, local_length, myid);
    fails += Test_N_VCloneEmptyVectorArray(5, plusX, myid);
    fails += Test_N_VCloneVectorArray(5, plusX, local_length, myid);

    /* Clone additional vectors for testing */
    plusY = N_VClone(plusX);
    if (plusY == NULL) {
      N_VDestroy(X);
      N_VDestroy(plusX);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    plusZ = N_VClone(plusX);
    if (plusZ == NULL) {
      N_VDestroy(X);
      N_VDestroy(plusX);
      N_VDestroy(plusY);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* Standard vector operation tests */
    if (myid == 0) printf("\nTesting standard vector operations:\n\n");

    fails += Test_N_VConst(plusX, local_length, myid);
    fails += Test_N_VLinearSum(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VProd(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VDiv(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VScale(plusX, plusZ, local_length, myid);
    fails += Test_N_VAbs(plusX, plusZ, local_length, myid);
    fails += Test_N_VInv(plusX, plusZ, local_length, myid);
    fails += Test_N_VAddConst(plusX, plusZ, local_length, myid);
    fails += Test_N_VDotProd(plusX, plusY, local_length, myid);
    fails += Test_N_VMaxNorm(plusX, local_length, myid);
    fails += Test_N_VWrmsNorm(plusX, plusY, local_length, myid);
    fails += Test_N_VWrmsNormMask(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VMin(plusX, local_length, myid);
    fails += Test_N_VWL2Norm(plusX, plusY, local_length, myid);
    fails += Test_N_VL1Norm(plusX, local_length, myid);
    fails += Test_N_VCompare(plusX, plusZ, local_length, myid);
    fails += Test_N_VInvTest(plusX, plusZ, local_length, myid);
    fails += Test_N_VConstrMask(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VMinQuotient(plusX, plusY, local_length, myid);

    /* Fused and vector array operations tests (disabled) */
    if (myid == 0) printf("\nTesting fused and vector array operations (disabled):\n\n");

    /* create vector and disable all fused and vector array operations */
    U = (i==UNMANAGED) ? N_VNew_Cuda(local_length) : N_VNewManaged_Cuda(local_length);
    retval = N_VEnableFusedOps_Cuda(U, SUNFALSE);
    if (U == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(plusX);
      N_VDestroy(plusY);
      N_VDestroy(plusZ);
      if (myid == 0) printf("FAIL: Unable to create a new CUDA vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* create the MPIPlusX vector */
    plusU = N_VMake_MPIPlusX(comm, U);
    if (U == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(U);
      N_VDestroy(plusX);
      N_VDestroy(plusY);
      N_VDestroy(plusZ);
      if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(plusU, local_length, myid);
    fails += Test_N_VScaleAddMulti(plusU, local_length, myid);
    fails += Test_N_VDotProdMulti(plusU, local_length, myid);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(plusU, local_length, myid);
    fails += Test_N_VScaleVectorArray(plusU, local_length, myid);
    fails += Test_N_VConstVectorArray(plusU, local_length, myid);
    fails += Test_N_VWrmsNormVectorArray(plusU, local_length, myid);
    fails += Test_N_VWrmsNormMaskVectorArray(plusU, local_length, myid);
    fails += Test_N_VScaleAddMultiVectorArray(plusU, local_length, myid);
    fails += Test_N_VLinearCombinationVectorArray(plusU, local_length, myid);

    /* Fused and vector array operations tests (enabled) */
    if (myid == 0) printf("\nTesting fused and vector array operations (enabled):\n\n");

    /* create vector and enable all fused and vector array operations */
    V = (i==UNMANAGED) ? N_VNew_Cuda(local_length) : N_VNewManaged_Cuda(local_length);
    retval = N_VEnableFusedOps_Cuda(V, SUNTRUE);
    if (V == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(U);
      N_VDestroy(plusX);
      N_VDestroy(plusY);
      N_VDestroy(plusZ);
      N_VDestroy(plusU);
      if (myid == 0) printf("FAIL: Unable to create a new CUDA vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* create the MPIPlusX vector */
    plusV = N_VMake_MPIPlusX(comm, V);
    if (V == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(U);
      N_VDestroy(V);
      N_VDestroy(plusU);
      N_VDestroy(plusX);
      N_VDestroy(plusY);
      N_VDestroy(plusZ);
      if (myid == 0) printf("FAIL: Unable to create a new MPIPlusX vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(plusV, local_length, myid);
    fails += Test_N_VScaleAddMulti(plusV, local_length, myid);
    fails += Test_N_VDotProdMulti(plusV, local_length, myid);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(plusV, local_length, myid);
    fails += Test_N_VScaleVectorArray(plusV, local_length, myid);
    fails += Test_N_VConstVectorArray(plusV, local_length, myid);
    fails += Test_N_VWrmsNormVectorArray(plusV, local_length, myid);
    fails += Test_N_VWrmsNormMaskVectorArray(plusV, local_length, myid);
    fails += Test_N_VScaleAddMultiVectorArray(plusV, local_length, myid);
    fails += Test_N_VLinearCombinationVectorArray(plusV, local_length, myid);

    /* local reduction operations */
    printf("\nTesting local reduction operations:\n\n");

    fails += Test_N_VDotProdLocal(plusX, plusY, local_length, myid);
    fails += Test_N_VMaxNormLocal(plusX, local_length, myid);
    fails += Test_N_VMinLocal(plusX, local_length, myid);
    fails += Test_N_VL1NormLocal(plusX, local_length, myid);
    fails += Test_N_VWSqrSumLocal(plusX, plusY, local_length, myid);
    fails += Test_N_VWSqrSumMaskLocal(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VInvTestLocal(plusX, plusZ, local_length, myid);
    fails += Test_N_VConstrMaskLocal(plusX, plusY, plusZ, local_length, myid);
    fails += Test_N_VMinQuotientLocal(plusX, plusY, local_length, myid);

    /* Free vectors */
    N_VDestroy(X);
    N_VDestroy(U);
    N_VDestroy(V);
    N_VDestroy(plusX);
    N_VDestroy(plusY);
    N_VDestroy(plusZ);
    N_VDestroy(plusU);
    N_VDestroy(plusV);
  }

  /* Print result */
  if (fails) {
    printf("FAIL: NVector module failed %i tests, Proc %d \n\n", fails, myid);
  } else {
    if (myid == 0)
      printf("SUCCESS: NVector module passed all tests \n\n");
  }

  /* check if any other process failed */
  (void) MPI_Allreduce(&fails, &globfails, 1, MPI_INT, MPI_MAX, comm);

  MPI_Finalize();

  return(globfails);
}

/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector plusX, sunindextype local_length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;
  N_Vector     X;

  X = N_VGetLocalVector_MPIPlusX(plusX);
  N_VCopyFromDevice_Cuda(X);
  Xdata = N_VGetHostArrayPointer_Cuda(X);

  /* check vector data */
  for (i = 0; i < local_length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector plusX)
{
  return (N_VGetLocalVector_MPIPlusX(plusX)->content == NULL) ? SUNFALSE : SUNTRUE;
}

void set_element(N_Vector plusX, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(plusX, i, i, val);
}

void set_element_range(N_Vector plusX, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;
  N_Vector     X;

  X = N_VGetLocalVector_MPIPlusX(plusX);

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Cuda(X);
  xd = N_VGetHostArrayPointer_Cuda(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Cuda(X);
}

realtype get_element(N_Vector plusX, sunindextype i)
{
  N_Vector X = N_VGetLocalVector_MPIPlusX(plusX);

  /* get i-th element of data array */
  N_VCopyFromDevice_Cuda(X);
  return (N_VGetHostArrayPointer_Cuda(X))[i];
}

double max_time(N_Vector plusX, double time)
{
  MPI_Comm *comm;
  double maxt;

  comm = (MPI_Comm*) N_VGetCommunicator(plusX);

  /* get max time across all MPI ranks */
  (void) MPI_Reduce(&time, &maxt, 1, MPI_DOUBLE, MPI_MAX, 0, *comm);
  return(maxt);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}
