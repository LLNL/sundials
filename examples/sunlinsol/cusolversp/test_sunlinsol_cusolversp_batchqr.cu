#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Cody J.Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2023, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the SUNLinSol cuSolverSp
 * module  implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>
#include <sunmatrix/sunmatrix_dense.h>
#include <sunmatrix/sunmatrix_sparse.h>
#include <sunmatrix/sunmatrix_cusparse.h>
#include <nvector/nvector_cuda.h>
#include <nvector/nvector_serial.h>
#include <sundials/sundials_math.h>
#include "test_sunlinsol.h"


/* ----------------------------------------------------------------------
 * SUNLinSol_KLU Linear Solver Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int             fails = 0;          /* counter for test failures     */
  sunindextype    N;                  /* matrix columns, rows          */
  int             block_size;         /* matrix block columns, rows    */
  int             block_nnz;          /* number of nonzeros in a block */
  int             block_nnz_max;      /* max nonzeros per block        */
  int             nblocks;            /* number of blocks              */
  SUNLinearSolver LS;                 /* linear solver object          */
  SUNMatrix       A, B, dA;           /* test matrices                 */
  N_Vector        x, b, d_x, d_xref, d_b;/* test vectors                  */
  realtype        *matdata, *xdata, *xrefdata;
  int             print_timing;
  sunindextype    i, j;
  SUNContext      sunctx;

  hipsparseStatus_t cusp_status;
  hipsolverStatus_t cusol_status;
  hipsparseHandle_t cusp_handle;
  hipsolverSpHandle_t cusol_handle;


  if (SUNContext_Create(NULL, &sunctx)) {
    printf("ERROR: SUNContext_Create failed\n");
    return(-1);
  }

  /* check input and set matrix dimensions */
  if (argc < 4){
    printf("ERROR: THREE (3) Inputs required: matrix block size, number of blocks, print timing \n");
    return(-1);
  }

  block_size = atol(argv[1]);
  if (block_size <= 0) {
    printf("ERROR: matrix size must be a positive integer \n");
    return(-1);
  }

  block_nnz_max = block_size*block_size / 4;

  nblocks = atol(argv[2]);
  if (nblocks <= 0) {
    printf("ERROR: number of blocks must be a positive integer \n");
    return(-1);
  }

  /* calculate the size of the overall martrix */
  N = block_size * nblocks;

  print_timing = atoi(argv[3]);
  SetTiming(print_timing);

  printf("\ncuSolverSp linear solver test: size %ld, block size %ld, number of blocks %ld\n\n",
    (long int) N, (long int) block_size, (long int) nblocks);

  /* Initialize cuSPARSE */
  cusp_status = hipsparseCreate(&cusp_handle);
  if (cusp_status != HIPSPARSE_STATUS_SUCCESS) {
    printf("ERROR: could not create cuSPARSE handle\n");
    return(-1);
  }

  /* Initialize cuSOLVER */
  cusol_status = hipsolverSpCreate(&cusol_handle);
  if (cusol_status != HIPSOLVER_STATUS_SUCCESS) {
    printf("ERROR: could not create cuSOLVER handle\n");
    return(-1);
  }

  /* Create matrices and vectors */
  B = SUNDenseMatrix(N, N, sunctx);
  d_x = N_VNew_Cuda(N, sunctx);
  d_xref = N_VNew_Cuda(N, sunctx);
  d_b = N_VNew_Cuda(N, sunctx);
  x = N_VMake_Serial(N, N_VGetHostArrayPointer_Cuda(d_x), sunctx);
  b = N_VMake_Serial(N, N_VGetHostArrayPointer_Cuda(d_b), sunctx);

  /* Zero the matrix */
  fails = SUNMatZero(B);

  /* Create sparsity pattern for a block. */
  sunindextype *cols = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
  sunindextype *rows = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
  for (i=0; i<block_nnz_max; i++) {
    cols[i] = rand() % block_size;
    rows[i] = rand() % block_size;
  }

  /* Fill matrix with uniform random data in [0,1/N] */
  for (i=0; i<nblocks; i++) {
    for (j=0; j<block_nnz_max; j++) {
      sunindextype col = cols[j] + block_size*i;
      sunindextype row = rows[j] + block_size*i;
      matdata = SUNDenseMatrix_Column(B,col);
      matdata[row] = (realtype) rand() / (realtype) RAND_MAX / N;
    }
  }

  /* Free temporary rows and cols variables */
  free(cols); free(rows);

  /* Add identity to B */
  fails = SUNMatScaleAddI(ONE, B);
  if (fails) {
    printf("FAIL: SUNMatScaleAddI failure\n");
    return(1);
  }

  /* Create sparse matrix from dense, and destroy B */
  A = SUNSparseFromDenseMatrix(B, ZERO, CSR_MAT);
  SUNMatDestroy(B);

  /* Calculate actual number of nonzeros per block */
  block_nnz = SUNSparseMatrix_NNZ(A) / nblocks;

  /* Create the device matrix */
  dA = SUNMatrix_cuSparse_NewBlockCSR(nblocks, block_size, block_size, block_nnz, cusp_handle, sunctx);
  if (dA == NULL) {
    printf("ERROR: could not create dA\n");
  }

  /* Copy data to device */
  fails = SUNMatrix_cuSparse_CopyToDevice(dA, SUNSparseMatrix_Data(A),
                                          SUNSparseMatrix_IndexPointers(A),
                                          SUNSparseMatrix_IndexValues(A));
  if (fails != 0) {
    printf("ERROR: could not copy A to the device\n");
    return(-1);
  }

  /* Fill x vector with uniform random data in [0,1] */
  xdata = N_VGetHostArrayPointer_Cuda(d_x);
  xrefdata = N_VGetHostArrayPointer_Cuda(d_xref);
  for (i=0; i<N; i++) {
    realtype tmp = (realtype) rand() / (realtype) RAND_MAX;
    xdata[i]    = tmp;
    xrefdata[i] = tmp;
  }
  N_VCopyToDevice_Cuda(d_x);
  N_VCopyToDevice_Cuda(d_xref);

  /* Synchronize before peforming dense operation on CPU */
  hipDeviceSynchronize();

  /* create right-hand side vector for linear solve */
  fails = SUNMatMatvec(A, x, b);
  if (fails) {
    printf("FAIL: SUNLinSol SUNMatMatvec failure\n");
    return(1);
  }
  N_VCopyToDevice_Cuda(d_b);

  /* Create cuSolverSp linear solver
   * The BatchedQR method allows you to solve many small subsystems in parallel.
   */
  LS = SUNLinSol_cuSolverSp_batchQR(d_x, dA, cusol_handle, sunctx);

  if (LS == NULL) {
    printf("FAIL: SUNLinSol_cuSolverSp_batchQR returned NULL\n");
    return(1);
  }

  /* Run Tests */
  fails += Test_SUNLinSolInitialize(LS, 0);
  fails += Test_SUNLinSolSetup(LS, dA, 0);
  fails += Test_SUNLinSolSolve(LS, dA, d_x, d_b, 1000*UNIT_ROUNDOFF, SUNTRUE, 0);

  fails += Test_SUNLinSolGetType(LS, SUNLINEARSOLVER_DIRECT, 0);
  fails += Test_SUNLinSolGetID(LS, SUNLINEARSOLVER_CUSOLVERSP_BATCHQR, 0);
  fails += Test_SUNLinSolLastFlag(LS, 0);
  fails += Test_SUNLinSolSpace(LS, 0);

  /* Print result */
  if (fails) {
    printf("FAIL: SUNLinSol module failed %i tests \n \n", fails);

    SUNMatrix_cuSparse_CopyFromDevice(dA, SUNSparseMatrix_Data(A), NULL, NULL);
    printf("\nA =\n");
    SUNSparseMatrix_Print(A,stdout);

    N_VCopyFromDevice_Cuda(d_xref);
    printf("x (reference)\n");
    N_VPrint_Cuda(d_xref);

    N_VCopyFromDevice_Cuda(d_x); /* copy solution from device */
    printf("x (computed)\n");
    N_VPrint_Cuda(d_x);

    N_VCopyFromDevice_Cuda(d_b);
    printf("\nb = Ax (reference)\n");
    N_VPrint_Cuda(d_b);
  } else {
    printf("SUCCESS: SUNLinSol module passed all tests \n \n");
  }

  /* Free solver, matrix and vectors */
  SUNLinSolFree(LS);
  SUNMatDestroy(A); SUNMatDestroy(dA);
  N_VDestroy(x); N_VDestroy(d_x); N_VDestroy(d_xref);
  N_VDestroy(b); N_VDestroy(d_b);

  /* Destroy the cuSOLVER and cuSPARSE handles */
  hipsparseDestroy(cusp_handle);
  hipsolverSpDestroy(cusol_handle);

  SUNContext_Free(&sunctx);

  return(fails);
}

/* ----------------------------------------------------------------------
 * Implementation-specific 'check' routines
 * --------------------------------------------------------------------*/
int check_vector(N_Vector X, N_Vector Y, realtype tol)
{
  int failure = 0;
  sunindextype i, local_length, maxloc;
  realtype *Xdata, *Ydata, maxerr;

  hipDeviceSynchronize();

  N_VCopyFromDevice_Cuda(X);
  N_VCopyFromDevice_Cuda(Y);

  Xdata = N_VGetHostArrayPointer_Cuda(X);
  Ydata = N_VGetHostArrayPointer_Cuda(Y);
  local_length = N_VGetLength(X);

  /* check vector data */
  for(i=0; i < local_length; i++)
    failure += SUNRCompareTol(Xdata[i], Ydata[i], tol);

  if (failure > ZERO) {
    maxerr = ZERO;
    maxloc = -1;
    for(i=0; i < local_length; i++) {
      if (SUNRabs(Xdata[i]-Ydata[i]) >  maxerr) {
        maxerr = SUNRabs(Xdata[i]-Ydata[i]);
        maxloc = i;
      }
    }
    printf("check err failure: maxerr = %g at loc %li (tol = %g)\n",
	   maxerr, (long int) maxloc, tol);
    return(1);
  }
  else
    return(0);
}

void sync_device()
{
  hipDeviceSynchronize();
}
