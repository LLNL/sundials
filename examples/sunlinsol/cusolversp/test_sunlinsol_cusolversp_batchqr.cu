#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Cody J.Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the SUNLinSol cuSolverSp
 * module  implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <sundials/sundials_types.h>
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>
#include <sunmatrix/sunmatrix_dense.h>
#include <sunmatrix/sunmatrix_sparse.h>
#include <nvector/nvector_cuda.h>
#include <sundials/sundials_math.h>
#include "test_sunlinsol.h"


/* ----------------------------------------------------------------------
 * SUNLinSol_KLU Linear Solver Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int             fails = 0;          /* counter for test failures     */
  sunindextype    N;                  /* matrix columns, rows          */
  int             block_size;         /* matrix block columns, rows    */
  int             block_nnz;          /* number of nonzeros in a block */
  int             block_nnz_max;      /* max nonzeros per block        */
  int             nblocks;            /* number of blocks              */
  SUNLinearSolver LS;                 /* linear solver object          */
  SUNMatrix       A, B;               /* test matrices                 */
  N_Vector        x, y, b;            /* test vectors                  */
  realtype        *matdata, *xdata;
  int             print_timing;
  sunindextype    i, j;

  /* check input and set matrix dimensions */
  if (argc < 4){
    printf("ERROR: THREE (3) Inputs required: matrix block size, number of blocks, print timing \n");
    return(-1);
  }

  block_size = atol(argv[1]);
  if (block_size <= 0) {
    printf("ERROR: matrix size must be a positive integer \n");
    return(-1);
  }

  block_nnz_max = block_size*block_size / 4;

  nblocks = atol(argv[2]);
  if (nblocks <= 0) {
    printf("ERROR: number of blocks must be a positive integer \n");
    return(-1);
  }

  /* calculate the size of the overall martrix */
  N = block_size * nblocks;

  print_timing = atoi(argv[3]);
  SetTiming(print_timing);

  printf("\ncuSolverSp linear solver test: size %ld, block size %ld, number of blocks %d\n\n",
    (long int) N, (long int) block_size, (long int) nblocks);

  /* Create matrices and vectors */
  B = SUNDenseMatrix(N, N);
  x = N_VNewManaged_Cuda(N);
  y = N_VNewManaged_Cuda(N);
  b = N_VNewManaged_Cuda(N);

  /* Zero the matrix */
  fails = SUNMatZero(B);

  /* Create sparsity pattern for a block. */
  sunindextype *cols = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
  sunindextype *rows = (sunindextype *) malloc(block_nnz_max*sizeof(sunindextype));
  for (i=0; i<block_nnz_max; i++) {
    cols[i] = rand() % block_size;
    rows[i] = rand() % block_size;
  }

  /* Fill matrix with uniform random data in [0,1/N] */
  for (i=0; i<nblocks; i++) {
    for (j=0; j<block_nnz_max; j++) {
      sunindextype col = cols[j] + block_size*i;
      sunindextype row = rows[j] + block_size*i;
      matdata = SUNDenseMatrix_Column(B,col);
      matdata[row] = (realtype) rand() / (realtype) RAND_MAX / N;
    }
  }

  /* Free temporary rows and cols variables */
  free(cols); free(rows);

  /* Add identity to B */
  fails = SUNMatScaleAddI(ONE, B);
  if (fails) {
    printf("FAIL: SUNMatScaleAddI failure\n");
    return(1);
  }

  /* Create sparse matrix from dense, and destroy B */
  A = SUNSparseFromDenseMatrix(B, ZERO, CSR_MAT);
  SUNMatDestroy(B);

  /* Calculate actual number of nonzeros per block */
  block_nnz = SUNSparseMatrix_NNZ(A) / nblocks;

  /* Fill x vector with uniform random data in [0,1] */
  xdata = N_VGetHostArrayPointer_Cuda(x);
  for (i=0; i<N; i++)
    xdata[i] = (realtype) rand() / (realtype) RAND_MAX;
  N_VCopyToDevice_Cuda(x);

  /* copy x into y to print in case of solver failure */
  N_VScale(ONE, x, y);

  /* create right-hand side vector for linear solve */
  fails = SUNMatMatvec(A, x, b);
  if (fails) {
    printf("FAIL: SUNLinSol SUNMatMatvec failure\n");
    return(1);
  }

  /* Create cuSolverSp linear solver
   * The BatchedQR method allows you to solve many small subsystems in parallel.
   */
  LS = SUNLinSol_cuSolverSp_batchQR(x,           /* the overall system vector */
                                    A,           /* the overall system matrix */
                                    nblocks,     /* number of subsystems */
                                    block_size,  /* size of a subsystem  */
                                    block_nnz);  /* number of nonzeros in a subsystem */

  if (LS == NULL) {
    printf("FAIL: SUNLinSol_cuSolverSp_batchQR returned NULL\n");
    return(1);
  }

  /* Run Tests */
  fails += Test_SUNLinSolInitialize(LS, 0);
  fails += Test_SUNLinSolSetup(LS, A, 0);
  fails += Test_SUNLinSolSolve(LS, A, x, b, 1000*UNIT_ROUNDOFF, 0);

  fails += Test_SUNLinSolGetType(LS, SUNLINEARSOLVER_DIRECT, 0);
  fails += Test_SUNLinSolGetID(LS, SUNLINEARSOLVER_CUSOLVERSP_BATCHQR, 0);
  fails += Test_SUNLinSolLastFlag(LS, 0);
  fails += Test_SUNLinSolSpace(LS, 0);

  /* Print result */
  if (fails) {
    printf("FAIL: SUNLinSol module failed %i tests \n \n", fails);
    printf("\nA =\n");
    SUNSparseMatrix_Print(A,stdout);
    printf("\nx (original) =\n");
    N_VPrint_Cuda(y);
    printf("\nb =\n");
    N_VPrint_Cuda(b);
    printf("\nx (computed) =\n");
    N_VPrint_Cuda(x);
  } else {
    printf("SUCCESS: SUNLinSol module passed all tests \n \n");
  }

  /* Free solver, matrix and vectors */
  SUNLinSolFree(LS);
  SUNMatDestroy(A);
  N_VDestroy(x);
  N_VDestroy(y);
  N_VDestroy(b);

  return(fails);
}

/* ----------------------------------------------------------------------
 * Implementation-specific 'check' routines
 * --------------------------------------------------------------------*/
int check_vector(N_Vector X, N_Vector Y, realtype tol)
{
  int failure = 0;
  sunindextype i, local_length, maxloc;
  realtype *Xdata, *Ydata, maxerr;

  hipDeviceSynchronize();

  Xdata = N_VGetHostArrayPointer_Cuda(X);
  Ydata = N_VGetHostArrayPointer_Cuda(Y);
  local_length = N_VGetLength(X);

  /* check vector data */
  for(i=0; i < local_length; i++)
    failure += FNEQ(Xdata[i], Ydata[i], tol);

  if (failure > ZERO) {
    maxerr = ZERO;
    maxloc = -1;
    for(i=0; i < local_length; i++) {
      if (SUNRabs(Xdata[i]-Ydata[i]) >  maxerr) {
        maxerr = SUNRabs(Xdata[i]-Ydata[i]);
        maxloc = i;
      }
    }
    printf("check err failure: maxerr = %g at loc %li (tol = %g)\n",
	   maxerr, (long int) maxloc, tol);
    return(1);
  }
  else
    return(0);
}
