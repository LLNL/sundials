#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Based on work by Allan Taylor, Alan Hindmarsh and
 *                Radu Serban @ LLNL
 * -----------------------------------------------------------------
 * Example problem for IDA: 2D heat equation, serial, GMRES.
 *
 * This example solves a discretized 2D heat equation problem.
 * This version uses the Krylov solver Spgmr.
 *
 * The DAE system solved is a spatial discretization of the PDE
 *          du/dt = d^2u/dx^2 + d^2u/dy^2
 * on the unit square. The boundary condition is u = 0 on all edges.
 * Initial conditions are given by u = 16 x (1 - x) y (1 - y). The
 * PDE is treated with central differences on a uniform M x M grid.
 * The values of u at the interior points satisfy ODEs, and
 * equations u = 0 at the boundaries are appended, to form a DAE
 * system of size N = M^2. Here M = 10.
 *
 * The system is solved with IDA using the Krylov linear solver
 * SPGMR. The preconditioner uses the diagonal elements of the
 * Jacobian only. Routines for preconditioning, required by
 * SPGMR, are supplied here. The constraints u >= 0 are posed
 * for all components. Output is taken at t = 0, .01, .02, .04,
 * ..., 10.24. Two cases are run -- with the Gram-Schmidt type
 * being Modified in the first case, and Classical in the second.
 * The second run uses IDAReInit.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <ida/ida.h>                   /* prototypes for IDA methods            */
#include <nvector/nvector_cuda.h>      /* access to CUDA N_Vector               */
#include <sunlinsol/sunlinsol_spgmr.h> /* access to spgmr SUNLinearSolver       */
#include <sundials/sundials_types.h>   /* definition of type realtype           */


/* Problem Constants */

#define NOUT  11
#define MGRID 10
#define NEQ   MGRID*MGRID
#define ZERO  RCONST(0.0)
#define ONE   RCONST(1.0)
#define TWO   RCONST(2.0)
#define FOUR  RCONST(4.0)

/* User data type */

struct _UserData {
  sunindextype mm;  /* number of grid points in one dimension */
  sunindextype neq; /* number of equations */
  realtype dx;
  realtype coeff;
  N_Vector pp;  /* vector of prec. diag. elements */
};

typedef _UserData *UserData;

/* Prototypes for functions called by IDA */

int resHeat(realtype tres, N_Vector uu, N_Vector up,
            N_Vector resval, void *user_data);

int PsetupHeat(realtype tt,
               N_Vector uu, N_Vector up, N_Vector rr,
               realtype c_j, void *prec_data);

int PsolveHeat(realtype tt,
               N_Vector uu, N_Vector up, N_Vector rr,
               N_Vector rvec, N_Vector zvec,
               realtype c_j, realtype delta, void *prec_data);

/* Prototypes for private functions */

static int SetInitialProfile(UserData data, N_Vector uu, N_Vector up,
                             N_Vector res);
static void PrintHeader(realtype rtol, realtype atol);
static void PrintOutput(void *mem, realtype t, N_Vector uu);
static int check_flag(void *flagvalue, const char *funcname, int opt);


/*
 *--------------------------------------------------------------------
 * CUDA Kernels
 *--------------------------------------------------------------------
 */

__global__
void resHeatKernel(const realtype *uu, const realtype *up, realtype *rr,
                   sunindextype mm, realtype coeff)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mm*mm) {
    i = tid % mm;
    j = tid / mm;

    if (j==0 || j==mm-1 || i==0 || i==mm-1) {
      /* Initialize rr to uu, to take care of boundary equations. */
      rr[tid] = uu[tid];
    } else {
      /* Loop over interior points; set res = up - (central difference). */
      realtype dif1 = uu[tid-1]  + uu[tid+1]  - TWO * uu[tid];
      realtype dif2 = uu[tid-mm] + uu[tid+mm] - TWO * uu[tid];
      rr[tid] = up[tid] - coeff * ( dif1 + dif2 );
    }
  }
}

__global__
void PsetupHeatKernel(realtype *ppv, sunindextype mm, realtype c_j, realtype coeff)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mm*mm) {
    i = tid % mm;
    j = tid / mm;

    if (j==0 || j==mm-1 || i==0 || i==mm-1) {
      /* Set ppv to one, to take care of boundary equations. */
      ppv[tid] = ONE;
    } else {
      /* Loop over interior points; ppv_i = 1/J_ii */
      ppv[tid] = ONE/(c_j + FOUR*coeff);
    }
  }
}

__global__
void setInitHeatKernel(realtype *up, sunindextype mm)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mm*mm) {
    i = tid % mm;
    j = tid / mm;

    if (j==0 || j==mm-1 || i==0 || i==mm-1) {
      up[tid] = ZERO;
    }
  }
}



/*
 *--------------------------------------------------------------------
 * MAIN PROGRAM
 *--------------------------------------------------------------------
 */

int main(int argc, char *argv[])
{
  void *mem;
  UserData data;
  N_Vector uu, up, constraints, res;
  int ier, iout;
  realtype rtol, atol, t0, t1, tout, tret;
  long int netf, ncfn, ncfl;
  SUNLinearSolver LS;
  SUNContext ctx;

  mem = NULL;
  data = NULL;
  uu = up = constraints = res = NULL;
  LS = NULL;

  /* Create the SUNDIALS context object for this simulation */

  ier = SUNContext_Create(NULL, &ctx);
  if (check_flag(&ier, "SUNContext_Create", 1)) return 1;

  /* Assign parameters in the user data structure. */

  data = (UserData) malloc(sizeof *data);
  data->pp = NULL;
  if(check_flag((void *)data, "malloc", 2)) return(1);

  data->mm  = MGRID;
  data->neq = data->mm * data->mm;
  data->dx = ONE/(data->mm-ONE);
  data->coeff = ONE/(data->dx * data->dx);

  /* Allocate N-vectors and the user data structure objects. */

  uu = N_VNew_Cuda(data->neq, ctx);
  if(check_flag((void *)uu, "N_VNew_Serial", 0)) return(1);

  up = N_VClone(uu);
  if(check_flag((void *)up, "N_VClone", 0)) return(1);

  res = N_VClone(uu);
  if(check_flag((void *)res, "N_VClone", 0)) return(1);

  constraints = N_VClone(uu);
  if(check_flag((void *)constraints, "N_VClone", 0)) return(1);

  data->pp = N_VClone(uu);
  if(check_flag((void *)data->pp, "N_VClone", 0)) return(1);

  /* Initialize uu, up. */

  SetInitialProfile(data, uu, up, res);

  /* Set constraints to all 1's for nonnegative solution values. */

  N_VConst(ONE, constraints);

  /* Assign various parameters. */

  t0   = ZERO;
  t1   = RCONST(0.01);
  rtol = ZERO;
  atol = RCONST(1.0e-3);

  /* Call IDACreate and IDAMalloc to initialize solution */

  mem = IDACreate(ctx);
  if(check_flag((void *)mem, "IDACreate", 0)) return(1);

  ier = IDASetUserData(mem, data);
  if(check_flag(&ier, "IDASetUserData", 1)) return(1);

  ier = IDASetConstraints(mem, constraints);
  if(check_flag(&ier, "IDASetConstraints", 1)) return(1);
  N_VDestroy(constraints);

  ier = IDAInit(mem, resHeat, t0, uu, up);
  if(check_flag(&ier, "IDAInit", 1)) return(1);

  ier = IDASStolerances(mem, rtol, atol);
  if(check_flag(&ier, "IDASStolerances", 1)) return(1);

  /* Create the linear solver SUNSPGMR with left preconditioning
     and the default Krylov dimension */
  LS = SUNLinSol_SPGMR(uu, SUN_PREC_LEFT, 0, ctx);
  if(check_flag((void *)LS, "SUNLinSol_SPGMR", 0)) return(1);

  /* IDA recommends allowing up to 5 restarts (default is 0) */
  ier = SUNLinSol_SPGMRSetMaxRestarts(LS, 5);
  if(check_flag(&ier, "SUNLinSol_SPGMRSetMaxRestarts", 1)) return(1);

  /* Attach the linear sovler */
  ier = IDASetLinearSolver(mem, LS, NULL);
  if(check_flag(&ier, "IDASetLinearSolver", 1)) return(1);

  /* Set the preconditioner solve and setup functions */
  ier = IDASetPreconditioner(mem, PsetupHeat, PsolveHeat);
  if(check_flag(&ier, "IDASetPreconditioner", 1)) return(1);

  /* Print output heading. */
  PrintHeader(rtol, atol);

  /*
   * -------------------------------------------------------------------------
   * CASE I
   * -------------------------------------------------------------------------
   */

  /* Print case number, output table heading, and initial line of table. */

  printf("\n\nCase 1: gsytpe = SUN_MODIFIED_GS\n");
  printf("\n   Output Summary (umax = max-norm of solution) \n\n");
  printf("  time     umax       k  nst  nni  nje   nre   nreLS    h      npe nps\n" );
  printf("----------------------------------------------------------------------\n");

  /* Loop over output times, call IDASolve, and print results. */

  for (tout = t1,iout = 1; iout <= NOUT ; iout++, tout *= TWO) {
    ier = IDASolve(mem, tout, &tret, uu, up, IDA_NORMAL);
    if(check_flag(&ier, "IDASolve", 1)) return(1);
    PrintOutput(mem, tret, uu);
  }

  /* Print remaining counters. */

  ier = IDAGetNumErrTestFails(mem, &netf);
  check_flag(&ier, "IDAGetNumErrTestFails", 1);

  ier = IDAGetNumNonlinSolvConvFails(mem, &ncfn);
  check_flag(&ier, "IDAGetNumNonlinSolvConvFails", 1);

  ier = IDAGetNumNonlinSolvConvFails(mem, &ncfl);
  check_flag(&ier, "IDAGetNumNonlinSolvConvFails", 1);

  printf("\nError test failures            = %ld\n", netf);
  printf("Nonlinear convergence failures = %ld\n", ncfn);
  printf("Linear convergence failures    = %ld\n", ncfl);

  /*
   * -------------------------------------------------------------------------
   * CASE II
   * -------------------------------------------------------------------------
   */

  /* Re-initialize uu, up. */

  SetInitialProfile(data, uu, up, res);

  /* Re-initialize IDA and SPGMR */

  ier = IDAReInit(mem, t0, uu, up);
  if(check_flag(&ier, "IDAReInit", 1)) return(1);

  ier = SUNLinSol_SPGMRSetGSType(LS, SUN_CLASSICAL_GS);
  if(check_flag(&ier, "SUNLinSol_SPGMRSetGSType",1)) return(1);

  /* Print case number, output table heading, and initial line of table. */

  printf("\n\nCase 2: gstype = SUN_CLASSICAL_GS\n");
  printf("\n   Output Summary (umax = max-norm of solution) \n\n");
  printf("  time     umax       k  nst  nni  nje   nre   nreLS    h      npe nps\n" );
  printf("----------------------------------------------------------------------\n");

  /* Loop over output times, call IDASolve, and print results. */

  for (tout = t1,iout = 1; iout <= NOUT ; iout++, tout *= TWO) {
    ier = IDASolve(mem, tout, &tret, uu, up, IDA_NORMAL);
    if(check_flag(&ier, "IDASolve", 1)) return(1);
    PrintOutput(mem, tret, uu);
  }

  /* Print remaining counters. */

  ier = IDAGetNumErrTestFails(mem, &netf);
  check_flag(&ier, "IDAGetNumErrTestFails", 1);

  ier = IDAGetNumNonlinSolvConvFails(mem, &ncfn);
  check_flag(&ier, "IDAGetNumNonlinSolvConvFails", 1);

  ier = IDAGetNumNonlinSolvConvFails(mem, &ncfl);
  check_flag(&ier, "IDAGetNumNonlinSolvConvFails", 1);

  printf("\nError test failures            = %ld\n", netf);
  printf("Nonlinear convergence failures = %ld\n", ncfn);
  printf("Linear convergence failures    = %ld\n", ncfl);

  /* Free Memory */

  IDAFree(&mem);
  SUNLinSolFree(LS);

  N_VDestroy(uu);
  N_VDestroy(up);
  N_VDestroy(res);

  N_VDestroy(data->pp);
  free(data);

  SUNContext_Free(&ctx);

  return(0);
}

/*
 *--------------------------------------------------------------------
 * FUNCTIONS CALLED BY IDA
 *--------------------------------------------------------------------
 */

/*
 * resHeat: heat equation system residual function (user-supplied)
 * This uses 5-point central differencing on the interior points, and
 * includes algebraic equations for the boundary values.
 * So for each interior point, the residual component has the form
 *    res_i = u'_i - (central difference)_i
 * while for each boundary point, it is res_i = u_i.
 */

int resHeat(realtype tt,
            N_Vector uu, N_Vector up, N_Vector rr,
            void *user_data)
{
  sunindextype mm;
  realtype coeff;
  UserData data;

  const realtype *uu_data = N_VGetDeviceArrayPointer_Cuda(uu);
  const realtype *up_data = N_VGetDeviceArrayPointer_Cuda(up);
  realtype *rr_data = N_VGetDeviceArrayPointer_Cuda(rr);

  data = (UserData) user_data;

  coeff = data->coeff;
  mm    = data->mm;

  unsigned block = 256;
  unsigned grid = (mm*mm + block - 1) / block;

  resHeatKernel<<<grid, block>>>(uu_data, up_data, rr_data, mm, coeff);

  return(0);
}

/*
 * PsetupHeat: setup for diagonal preconditioner for idaHeat2D_kry.
 *
 * The optional user-supplied functions PsetupHeat and
 * PsolveHeat together must define the left preconditoner
 * matrix P approximating the system Jacobian matrix
 *                   J = dF/du + cj*dF/du'
 * (where the DAE system is F(t,u,u') = 0), and solve the linear
 * systems P z = r.   This is done in this case by keeping only
 * the diagonal elements of the J matrix above, storing them as
 * inverses in a vector pp, when computed in PsetupHeat, for
 * subsequent use in PsolveHeat.
 *
 * In this instance, only cj and data (user data structure, with
 * pp etc.) are used from the PsetupdHeat argument list.
 */

int PsetupHeat(realtype tt,
               N_Vector uu, N_Vector up, N_Vector rr,
               realtype c_j, void *prec_data)
{
  sunindextype mm;
  realtype *ppv;
  UserData data;

  data = (UserData) prec_data;
  ppv = N_VGetDeviceArrayPointer_Cuda(data->pp);
  mm = data->mm;
  realtype coeff = data->coeff;

  unsigned block = 256;
  unsigned grid = (mm*mm + block - 1) / block;

  PsetupHeatKernel<<<grid, block>>>(ppv, mm, c_j, coeff);

  return(0);
}

/*
 * PsolveHeat: solve preconditioner linear system.
 * This routine multiplies the input vector rvec by the vector pp
 * containing the inverse diagonal Jacobian elements (previously
 * computed in PrecondHeateq), returning the result in zvec.
 */

int PsolveHeat(realtype tt,
               N_Vector uu, N_Vector up, N_Vector rr,
               N_Vector rvec, N_Vector zvec,
               realtype c_j, realtype delta, void *prec_data)
{
  UserData data;
  data = (UserData) prec_data;
  N_VProd(data->pp, rvec, zvec);
  return(0);
}

/*
 *--------------------------------------------------------------------
 * PRIVATE FUNCTIONS
 *--------------------------------------------------------------------
 */

/*
 * SetInitialProfile: routine to initialize u and up vectors.
 */

static int SetInitialProfile(UserData data, N_Vector uu, N_Vector up,
                             N_Vector res)
{
  sunindextype mm, i, j;
  realtype xfact, yfact, *udata, *updata;

  mm = data->mm;

  udata = N_VGetHostArrayPointer_Cuda(uu);

  /* Initialize uu on all grid points. */
  for (j = 0; j < mm; j++) {
    yfact = data->dx * j;
    for (i = 0; i < mm; i++) {
      xfact = data->dx * i;
      udata[mm*j + i] = RCONST(16.0) * xfact * (ONE - xfact) * yfact * (ONE - yfact);
    }
  }

  N_VCopyToDevice_Cuda(uu);

  /* Initialize up vector to 0. */
  N_VConst(ZERO, up);

  /* resHeat sets res to negative of ODE RHS values at interior points. */
  resHeat(ZERO, uu, up, res, data);

  /* Copy -res into up to get correct interior initial up values. */
  N_VScale(-ONE, res, up);

  /* Set up at boundary points to zero. */
  updata = N_VGetDeviceArrayPointer_Cuda(up);

  unsigned block = 256;
  unsigned grid = (mm*mm + block - 1) / block;

  setInitHeatKernel<<<grid, block>>>(updata, mm);

  return(0);
}

/*
 * Print first lines of output (problem description)
 */

static void PrintHeader(realtype rtol, realtype atol)
{
  printf("\nidaHeat2D_kry: Heat equation, serial example problem for IDA \n");
  printf("         Discretized heat equation on 2D unit square. \n");
  printf("         Zero boundary conditions,");
  printf(" polynomial initial conditions.\n");
  printf("         Mesh dimensions: %d x %d", MGRID, MGRID);
  printf("        Total system size: %d\n\n", NEQ);
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("Tolerance parameters:  rtol = %Lg   atol = %Lg\n", rtol, atol);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#else
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#endif
  printf("Constraints set to force all solution components >= 0. \n");
  printf("Linear solver: SPGMR, preconditioner using diagonal elements. \n");
}

/*
 * PrintOutput: print max norm of solution and current solver statistics
 */

static void PrintOutput(void *mem, realtype t, N_Vector uu)
{
  realtype hused, umax;
  long int nst, nni, nje, nre, nreLS, nli, npe, nps;
  int kused, ier;

  umax = N_VMaxNorm(uu);

  ier = IDAGetLastOrder(mem, &kused);
  check_flag(&ier, "IDAGetLastOrder", 1);
  ier = IDAGetNumSteps(mem, &nst);
  check_flag(&ier, "IDAGetNumSteps", 1);
  ier = IDAGetNumNonlinSolvIters(mem, &nni);
  check_flag(&ier, "IDAGetNumNonlinSolvIters", 1);
  ier = IDAGetNumResEvals(mem, &nre);
  check_flag(&ier, "IDAGetNumResEvals", 1);
  ier = IDAGetLastStep(mem, &hused);
  check_flag(&ier, "IDAGetLastStep", 1);
  ier = IDAGetNumJtimesEvals(mem, &nje);
  check_flag(&ier, "IDAGetNumJtimesEvals", 1);
  ier = IDAGetNumLinIters(mem, &nli);
  check_flag(&ier, "IDAGetNumLinIters", 1);
  ier = IDAGetNumLinResEvals(mem, &nreLS);
  check_flag(&ier, "IDAGetNumLinResEvals", 1);
  ier = IDAGetNumPrecEvals(mem, &npe);
  check_flag(&ier, "IDAGetPrecEvals", 1);
  ier = IDAGetNumPrecSolves(mem, &nps);
  check_flag(&ier, "IDAGetNumPrecSolves", 1);

#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf(" %5.2Lf %13.5Le  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2Le  %3ld %3ld\n",
         t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
         t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#else
  printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
         t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#endif
}

/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns a flag so check if
 *            flag >= 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

static int check_flag(void *flagvalue, const char *funcname, int opt)
{
  int *errflag;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && flagvalue == NULL) {
    fprintf(stderr,
            "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1);
  } else if (opt == 1) {
    /* Check if flag < 0 */
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr,
              "\nSUNDIALS_ERROR: %s() failed with flag = %d\n\n",
              funcname, *errflag);
      return(1);
    }
  } else if (opt == 2 && flagvalue == NULL) {
    /* Check if function returned NULL pointer - no memory allocated */
    fprintf(stderr,
            "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1);
  }

  return(0);
}
