#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Acknowledgements: This example is based on idaHeat2D_kry_p
 *                   example by Daniel R. Reynolds @ SMU and
 *                   Allan Taylor, Alan Hindmarsh and
 *                   Radu Serban @ LLNL
 * -----------------------------------------------------------------
 * Example problem for IDA: 2D heat equation, parallel, GMRES.
 *
 * This example solves a discretized 2D heat equation problem.
 * This version uses the Krylov solver SUNSPGMR.
 *
 * The DAE system solved is a spatial discretization of the PDE
 *          du/dt = d^2u/dx^2 + d^2u/dy^2
 * on the unit square. The boundary condition is u = 0 on all edges.
 * Initial conditions are given by u = 16 x (1 - x) y (1 - y).
 * The PDE is treated with central differences on a uniform MX x MY
 * grid. The values of u at the interior points satisfy ODEs, and
 * equations u = 0 at the boundaries are appended, to form a DAE
 * system of size N = MX * MY. Here MX = MY = 10.
 *
 * The system is actually implemented on submeshes, processor by
 * processor, with an MXSUB by MYSUB mesh on each of NPEX * NPEY
 * processors.
 *
 * The system is solved with IDA using the Krylov linear solver
 * SUNSPGMR. The preconditioner uses the diagonal elements of the
 * Jacobian only. Routines for preconditioning, required by
 * SUNSPGMR, are supplied here. The constraints u >= 0 are posed
 * for all components. Local error testing on the boundary values
 * is suppressed. Output is taken at t = 0, .01, .02, .04,
 * ..., 10.24.
 *
 * This example uses RAJA hardware abstraction layer to create
 * an executable that runs on GPU devices in a distributed memory
 * environment.
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <ida/ida.h>
#include <ida/ida_spils.h>
#include <nvector/nvector_mpiplusx.h>
#include <nvector/nvector_raja.h>
#include <sunlinsol/sunlinsol_spgmr.h>
#include <sundials/sundials_types.h>
#include <sundials/sundials_mpi_types.h>
#include <sundials/sundials_math.h>

#include <RAJA/RAJA.hpp>

#define ZERO  RCONST(0.0)
#define ONE   RCONST(1.0)
#define TWO   RCONST(2.0)

#define NOUT         11    /* Number of output times */

#define NPEX         2     /* No. PEs in x direction of PE array */
#define NPEY         2     /* No. PEs in y direction of PE array */
                           /* Total no. PEs = NPEX*NPEY */
#define MXSUB        5     /* No. x points per subgrid */
#define MYSUB        5     /* No. y points per subgrid */

/* Global spatial mesh is MX x MY = (NPEX x MXSUB) x (NPEY x MYSUB) */

typedef struct {
  int thispe, npex, npey, ixsub, jysub;
  sunindextype mx, my, mxsub, mysub;
  realtype     dx, dy, coeffx, coeffy, coeffxy;
  realtype    *uext; /* device array */
  realtype    *host_send_buff;
  realtype    *host_recv_buff;
  realtype    *dev_send_buff;
  realtype    *dev_recv_buff;
  N_Vector     pp;    /* vector of diagonal preconditioner elements */
  MPI_Comm     comm;
} *UserData;

/* User-supplied residual function and supporting routines */

int resHeat(realtype tt, N_Vector uu, N_Vector up,
            N_Vector rr, void *user_data);

static int rescomm(N_Vector uu, N_Vector up, void *user_data);

static int reslocal(realtype tt, N_Vector uu, N_Vector up,
                    N_Vector res,  void *user_data);

static int BSend(MPI_Comm comm, int thispe,
                 int ixsub, int jysub, int npex, int npey,
                 sunindextype mxsub, sunindextype mysub,
                 const realtype *uarray, realtype *dev_send_buff, realtype *host_send_buff);

static int BRecvPost(MPI_Comm comm, MPI_Request request[], int thispe,
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *host_recv_buff);

static int BRecvWait(MPI_Request request[],
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *uext, const realtype *host_recv_buff, realtype *dev_recv_buff);

/* User-supplied preconditioner routines */

int PsolveHeat(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
               N_Vector rvec, N_Vector zvec, realtype c_j,
               realtype delta, void *user_data);

int PsetupHeat(realtype tt, N_Vector yy, N_Vector yp, N_Vector rr,
               realtype c_j, void *user_data);

/* Private function to check function return values */

static int InitUserData(int thispe, MPI_Comm comm, UserData data);

static int AllocUserData(MPI_Comm comm, N_Vector uu, UserData data);

static int DeleteUserData(UserData data);

static int SetInitialProfile(N_Vector uu, N_Vector up, N_Vector id,
                             N_Vector res, UserData data);

static void PrintHeader(realtype rtol, realtype atol, UserData data);

static void PrintOutput(int id, void *ida_mem, realtype t, N_Vector uu);

static void PrintFinalStats(void *ida_mem);

static int check_flag(void *flagvalue, const char *funcname, int opt, int id);

/*
 *--------------------------------------------------------------------
 * MAIN PROGRAM
 *--------------------------------------------------------------------
 */

int main(int argc, char *argv[])
{
  MPI_Comm comm;
  void *ida_mem;
  SUNLinearSolver LS;
  UserData data;
  int iout, thispe, ier, npes;
  sunindextype local_N;
  realtype rtol, atol, t0, t1, tout, tret;
  N_Vector uulocal, uu, up, constraints, id, res;

  ida_mem = NULL;
  LS = NULL;
  data = NULL;
  uulocal = uu = up = constraints = id = res = NULL;

  /* Get processor number and total number of pe's. */

  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &npes);
  MPI_Comm_rank(comm, &thispe);

  /* Allocate and initialize the data structure */
  data = (UserData) malloc(sizeof *data);
  if(check_flag((void *)data, "malloc", 2, thispe))
    MPI_Abort(comm, 1);

  InitUserData(thispe, comm, data);

  /* Check if the number of MPI processes matches the number of subgrids */
  if (npes != (data->npex * data->npey)) {
    if (thispe == 0)
      fprintf(stderr,
              "\nMPI_ERROR(0): npes = %d is not equal to NPEX*NPEY = %d\n",
              npes, data->npex * data->npey);
    free(data);
    MPI_Finalize();
    return(1);
  }

  /* Set local length local_N */
  local_N = data->mxsub * data->mysub;

  /* Allocate and initialize N-vectors. */

  uulocal = N_VNew_Raja(local_N);
  if(check_flag((void *)uulocal, "N_VNew_Raja", 0, thispe))
    MPI_Abort(comm, 1);
  
  uu = N_VMake_MPIPlusX(comm, uulocal);
  if(check_flag((void *)uu, "N_VMake_MPIPlusX", 0, thispe))
    MPI_Abort(comm, 1);

  up = N_VClone(uu);
  if(check_flag((void *)up, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  res = N_VClone(uu);
  if(check_flag((void *)res, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  constraints = N_VClone(uu);
  if(check_flag((void *)constraints, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  id = N_VClone(uu);
  if(check_flag((void *)id, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  /* Allocate user data, extended vector, and MPI buffers */
  ier = AllocUserData(comm, uu, data);
  if(check_flag(&ier, "AllocUserData", 1, thispe))
    MPI_Abort(comm, 1);


  /* Initialize the uu, up, id, and res profiles. */
  SetInitialProfile(uu, up, id, res, data);

  /* Set constraints to all 1's for nonnegative solution values. */
  N_VConst(ONE, constraints);

  t0 = ZERO; t1 = RCONST(0.01);

  /* Scalar relative and absolute tolerance. */
  rtol = ZERO;
  atol = RCONST(1.0e-3);

  /* Call IDACreate and IDAMalloc to initialize solution. */

  ida_mem = IDACreate();
  if(check_flag((void *)ida_mem, "IDACreate", 0, thispe))
    MPI_Abort(comm, 1);

  ier = IDASetUserData(ida_mem, data);
  if(check_flag(&ier, "IDASetUserData", 1, thispe))
    MPI_Abort(comm, 1);

  ier = IDASetSuppressAlg(ida_mem, SUNTRUE);
  if(check_flag(&ier, "IDASetSuppressAlg", 1, thispe))
    MPI_Abort(comm, 1);

  ier = IDASetId(ida_mem, id);
  if(check_flag(&ier, "IDASetId", 1, thispe))
    MPI_Abort(comm, 1);

  ier = IDASetConstraints(ida_mem, constraints);
  if(check_flag(&ier, "IDASetConstraints", 1, thispe))
    MPI_Abort(comm, 1);
  N_VDestroy(constraints);

  ier = IDAInit(ida_mem, resHeat, t0, uu, up);
  if(check_flag(&ier, "IDAInit", 1, thispe))
    MPI_Abort(comm, 1);

  ier = IDASStolerances(ida_mem, rtol, atol);
  if(check_flag(&ier, "IDASStolerances", 1, thispe))
    MPI_Abort(comm, 1);

  /* Call SUNSPGMR and IDASetLinearSolver to specify the linear solver. */

  LS = SUNSPGMR(uu, PREC_LEFT, 0);  /* use default maxl */
  if(check_flag((void *)LS, "SUNSPGMR", 0, thispe))
    MPI_Abort(comm, 1);

  ier = IDASpilsSetLinearSolver(ida_mem, LS);
  if(check_flag(&ier, "IDASpilsSetLinearSolver", 1, thispe))
    MPI_Abort(comm, 1);

  ier = IDASpilsSetPreconditioner(ida_mem, PsetupHeat, PsolveHeat);
  if(check_flag(&ier, "IDASpilsSetPreconditioner", 1, thispe))
    MPI_Abort(comm, 1);

  /* Print output heading (on processor 0 only) and intial solution  */

  if (thispe == 0) PrintHeader(rtol, atol, data);
  PrintOutput(thispe, ida_mem, t0, uu);

  /* Loop over tout, call IDASolve, print output. */

  for (tout = t1, iout = 1; iout <= NOUT; iout++, tout *= TWO) {

    ier = IDASolve(ida_mem, tout, &tret, uu, up, IDA_NORMAL);
    if(check_flag(&ier, "IDASolve", 1, thispe))
      MPI_Abort(comm, 1);

    PrintOutput(thispe, ida_mem, tret, uu);

  }

  /* Print remaining counters. */

  if (thispe == 0) PrintFinalStats(ida_mem);

  /* Free memory */

  IDAFree(&ida_mem);
  SUNLinSolFree(LS);

  N_VDestroy(id);
  N_VDestroy(res);
  N_VDestroy(up);
  N_VDestroy(uu);

  DeleteUserData(data);
  free(data);

  MPI_Finalize();
  
  return(0);
}

/*
 *--------------------------------------------------------------------
 * FUNCTIONS CALLED BY IDA
 *--------------------------------------------------------------------
 */

/*
 * resHeat: heat equation system residual function
 * This uses 5-point central differencing on the interior points, and
 * includes algebraic equations for the boundary values.
 * So for each interior point, the residual component has the form
 *    res_i = u'_i - (central difference)_i
 * while for each boundary point, it is res_i = u_i.
 *
 * This parallel implementation uses several supporting routines.
 * First a call is made to rescomm to do communication of subgrid boundary
 * data into array uext.  Then reslocal is called to compute the residual
 * on individual processors and their corresponding domains.  The routines
 * BSend, BRecvPost, and BREcvWait handle interprocessor communication
 * of uu required to calculate the residual.
 */

int resHeat(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
            void *user_data)
{
  int retval = 0;

  /* Call rescomm to do inter-processor communication. */
  retval = rescomm(uu, up, user_data);

  /* Call reslocal to calculate res. */
  retval = reslocal(tt, uu, up, rr, user_data);

  return(retval);

}

/*
 * PsetupHeat: setup for diagonal preconditioner for heatsk.
 *
 * The optional user-supplied functions PsetupHeat and
 * PsolveHeat together must define the left preconditoner
 * matrix P approximating the system Jacobian matrix
 *                   J = dF/du + cj*dF/du'
 * (where the DAE system is F(t,u,u') = 0), and solve the linear
 * systems P z = r.   This is done in this case by keeping only
 * the diagonal elements of the J matrix above, storing them as
 * inverses in a vector pp, when computed in PsetupHeat, for
 * subsequent use in PsolveHeat.
 *
 * In this instance, only cj and data (user data structure, with
 * pp etc.) are used from the PsetupHeat argument list.
 *
 */

int PsetupHeat(realtype tt, N_Vector yy, N_Vector yp, N_Vector rr,
               realtype c_j, void *user_data)
{
  const sunindextype zero = 0;
  sunindextype ibc, i0, jbc, j0;

  /* Unwrap the user data */
  UserData data = (UserData) user_data;
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;
  const int npex  = data->npex;
  const int npey  = data->npey;
  const sunindextype mxsub = data->mxsub;
  const sunindextype mysub = data->mysub;
  realtype *ppv = N_VGetDeviceArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(data->pp));

  /* Calculate the value for the inverse element of the diagonal preconditioner */
  const realtype pelinv = ONE/(c_j + data->coeffxy);

  /* Initially set all pp elements on the device to one. */
  N_VConst(ONE, data->pp);

  ibc = (ixsub == 0) || (ixsub == npex-1);
  i0  = (ixsub == 0);
  jbc = (jysub == 0) || (jysub == npey-1);
  j0  = (jysub == 0);

  /* Set inverse of the preconditioner; ppv must be on the device */
  RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, (mxsub - ibc)*(mysub - jbc)),
    [=] __device__(sunindextype tid) {
      sunindextype j = tid / (mxsub - ibc) + j0;
      sunindextype i = tid % (mxsub - ibc) + i0;

      ppv[i + j*mxsub] = pelinv;
    }
  );

  return(0);
}

/*
 * PsolveHeat: solve preconditioner linear system.
 * This routine multiplies the input vector rvec by the vector pp
 * containing the inverse diagonal Jacobian elements (previously
 * computed in PsetupHeat), returning the result in zvec.
 */

int PsolveHeat(realtype tt, N_Vector uu, N_Vector up,
               N_Vector rr, N_Vector rvec, N_Vector zvec,
               realtype c_j, realtype delta, void *user_data)
{
  UserData data = (UserData) user_data;

  N_VProd(data->pp, rvec, zvec);

  return(0);

}

/*
 *--------------------------------------------------------------------
 * SUPPORTING FUNCTIONS
 *--------------------------------------------------------------------
 */


/*
 * rescomm routine.  This routine performs all inter-processor
 * communication of data in u needed to calculate G.
 */

static int rescomm(N_Vector uu, N_Vector up, void* user_data)
{
  UserData data = (UserData) user_data;

  /* Get comm, thispe, subgrid indices, data sizes */
  MPI_Comm comm = data->comm;
  const int thispe = data->thispe;
  const int ixsub  = data->ixsub;
  const int jysub  = data->jysub;
  const int npex   = data->npex;
  const int npey   = data->npey;
  const sunindextype mxsub = data->mxsub;
  const sunindextype mysub = data->mysub;

  /* Get pointers to buffers and extended solution vector data array uext. */
  realtype *uext = data->uext;
  realtype *host_send_buff = data->host_send_buff;
  realtype *host_recv_buff = data->host_recv_buff;
  realtype *dev_send_buff  = data->dev_send_buff;
  realtype *dev_recv_buff  = data->dev_recv_buff;

  /* Get solution vector data. */
  const realtype *uarray = N_VGetDeviceArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(uu));

  /* Set array of MPI requests */
  MPI_Request request[4];

  /* Start receiving boundary data from neighboring PEs. */
  BRecvPost(comm, request, thispe, ixsub, jysub, npex, npey, mxsub, mysub, host_recv_buff);

  /* Send data from boundary of local grid to neighboring PEs. */
  BSend(comm, thispe, ixsub, jysub, npex, npey, mxsub, mysub, uarray, dev_send_buff, host_send_buff);

  /* Finish receiving boundary data from neighboring PEs. */
  BRecvWait(request, ixsub, jysub, npex, npey, mxsub, mysub, uext, host_recv_buff, dev_recv_buff);

  return(0);

}

/*
 * reslocal routine.  Compute res = F(t, uu, up).  This routine assumes
 * that all inter-processor communication of data needed to calculate F
 * has already been done, and that this data is in the work array uext.
 */

static int reslocal(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
                    void *user_data)
{
  UserData data = (UserData) user_data;

  /* Get subgrid indices, array sizes, and grid coefficients. */
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;
  const int npex  = data->npex;
  const int npey  = data->npey;
  const sunindextype mxsub  = data->mxsub;
  const sunindextype mxsub2 = data->mxsub + 2;
  const sunindextype mysub  = data->mysub;
  const realtype coeffx  = data->coeffx;
  const realtype coeffy  = data->coeffy;
  const realtype coeffxy = data->coeffxy;

  /* Vector data arrays, extended work array uext. */
  const realtype *uuv = N_VGetDeviceArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(uu));
  const realtype *upv = N_VGetDeviceArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(up));
  realtype *resv = N_VGetDeviceArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(rr));
  realtype *uext = data->uext;

  const sunindextype zero = 0;
  sunindextype ibc, i0, jbc, j0;

  /* Initialize all elements of rr to uu. This sets the boundary
     elements simply without indexing hassles. */

  N_VScale(ONE, uu, rr);

  /* Copy local segment of u vector into the working extended array uext.
     This completes uext prior to the computation of the rr vector.
     uext and uuv must be on the device.     */
  RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mxsub*mysub),
    [=] __device__(sunindextype tid) {
      sunindextype j = tid/mxsub;
      sunindextype i = tid%mxsub;

      uext[(i+1) + (j+1)*mxsub2] = uuv[i + j*mxsub];
    }
  );

  /* Set loop limits for the interior of the local subgrid. */

  /* Prepare to loop over subgrid. */
  ibc = (ixsub == 0) || (ixsub == npex-1);
  i0  = (ixsub == 0);
  jbc = (jysub == 0) || (jysub == npey-1);
  j0  = (jysub == 0);

  /* Compute local residual; uext, upv, and resv must be on the device */
  RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, (mxsub - ibc)*(mysub - jbc)),
    [=] __device__(sunindextype tid) {
      sunindextype j = tid/(mxsub - ibc) + j0;
      sunindextype i = tid%(mxsub - ibc) + i0;
      sunindextype locu  = i + j*mxsub;
      sunindextype locue = (i+1) + (j+1)*mxsub2;

      realtype termx   = coeffx * (uext[locue-1]      + uext[locue+1]);
      realtype termy   = coeffy * (uext[locue-mxsub2] + uext[locue+mxsub2]);
      realtype termctr = coeffxy * uext[locue];
      resv[locu] = upv[locu] - (termx + termy - termctr);
    }
  );

  return(0);

}

/*
 * Routine to send boundary data to neighboring PEs.
 */

static int BSend(MPI_Comm comm, int thispe,
                 int ixsub, int jysub, int npex, int npey,
                 sunindextype mxsub, sunindextype mysub,
                 const realtype *uarray, realtype *dev_send_buff, realtype *host_send_buff)
{
  hipError_t err;
  const sunindextype zero = 0;
  /* Have left, right, top and bottom device buffers use the same dev_send_buff. */
  realtype *d_bufleft   = dev_send_buff;
  realtype *d_bufright  = dev_send_buff + mysub;
  realtype *d_buftop    = dev_send_buff + 2*mysub;
  realtype *d_bufbottom = dev_send_buff + 2*mysub + mxsub;

  /* Have left, right, top and bottom host buffers use the same host_send_buff. */
  realtype *h_bufleft   = host_send_buff;
  realtype *h_bufright  = host_send_buff + mysub;
  realtype *h_buftop    = host_send_buff + 2*mysub;
  realtype *h_bufbottom = host_send_buff + 2*mysub + mxsub;

  /* If jysub > 0, send data from bottom x-line of u.  (via bufbottom) */

  if (jysub != 0) {
    // Device kernel here to copy from uarray to the buffer on the device
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mxsub),
      [=] __device__(sunindextype lx) {
        d_bufbottom[lx] = uarray[lx];
      }
    );
    // Copy buffer to the host
    err = hipMemcpy(h_bufbottom, d_bufbottom, mxsub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Bottom buffer: Copy from device to host failed with code %d... \n", err);
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufbottom, mxsub, MPI_SUNREALTYPE, thispe-npex, 0, comm);
  }

  /* If jysub < NPEY-1, send data from top x-line of u. (via buftop) */

  if (jysub != npey-1) {
    // Device kernel here to copy from uarray to the buffer on the device
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mxsub),
      [=] __device__(sunindextype lx) {
        d_buftop[lx] = uarray[(mysub-1)*mxsub + lx];
      }
    );
    // Copy buffer to the host
    err = hipMemcpy(h_buftop, d_buftop, mxsub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Top buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_buftop, mxsub, MPI_SUNREALTYPE, thispe+npex, 0, comm);
  }

  /* If ixsub > 0, send data from left y-line of u (via bufleft). */

  if (ixsub != 0) {
    // Device kernel here to copy from uarray to the buffer on the device
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mysub),
      [=] __device__(sunindextype ly) {
        d_bufleft[ly] = uarray[ly*mxsub];
      }
    );
    // Copy buffer to the host
    err = hipMemcpy(h_bufleft, d_bufleft, mysub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Left buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufleft, mysub, MPI_SUNREALTYPE, thispe-1, 0, comm);
  }

  /* If ixsub < NPEX-1, send data from right y-line of u (via bufright). */

  if (ixsub != npex-1) {
    // Device kernel here to copy from uarray to the buffer on the device
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mysub),
      [=] __device__(sunindextype ly) {
        d_bufright[ly] = uarray[ly*mxsub + (mxsub-1)];
      }
    );
    // Copy buffer to the host
    err = hipMemcpy(h_bufright, d_bufright, mysub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Right buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufright, mysub, MPI_SUNREALTYPE, thispe+1, 0, comm);
  }

  return(0);

}

/*
 * Routine to start receiving boundary data from neighboring PEs.
 * Notes:
 *   1) buffer should be able to hold 2*(MYSUB+MYSUB) realtype entries, should
 *      be passed to both the BRecvPost and BRecvWait functions, and should not
 *      be manipulated between the two calls.
 *   2) request should have 4 entries, and should be passed in
 *      both calls also.
 */

static int BRecvPost(MPI_Comm comm, MPI_Request request[], int thispe,
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *host_recv_buff)
{
  /* Have left, right, top and bottom buffers use the same host_recv_buff. */
  realtype *bufleft   = host_recv_buff;
  realtype *bufright  = host_recv_buff + mysub;
  realtype *buftop    = host_recv_buff + 2*mysub;
  realtype *bufbottom = host_recv_buff + 2*mysub + mxsub;

  /* If jysub > 0, receive data for bottom x-line of uext. */
  if (jysub != 0) {
    MPI_Irecv(bufbottom, mxsub, MPI_SUNREALTYPE,
              thispe-npex, 0, comm, &request[0]);
  }

  /* If jysub < NPEY-1, receive data for top x-line of uext. */
  if (jysub != npey-1) {
    MPI_Irecv(buftop, mxsub, MPI_SUNREALTYPE,
              thispe+npex, 0, comm, &request[1]);
  }

  /* If ixsub > 0, receive data for left y-line of uext (via bufleft). */
  if (ixsub != 0) {
    MPI_Irecv(&bufleft[0], mysub, MPI_SUNREALTYPE,
              thispe-1, 0, comm, &request[2]);
  }

  /* If ixsub < NPEX-1, receive data for right y-line of uext (via bufright). */
  if (ixsub != npex-1) {
    MPI_Irecv(&bufright[0], mysub, MPI_SUNREALTYPE,
              thispe+1, 0, comm, &request[3]);
  }

  return(0);

}

/*
 * Routine to finish receiving boundary data from neighboring PEs.
 * Notes:
 *   1) buffer should be able to hold 2*MYSUB realtype entries, should be
 *      passed to both the BRecvPost and BRecvWait functions, and should not
 *      be manipulated between the two calls.
 *   2) request should have four entries, and should be passed in both
 *      calls also.
 */

static int BRecvWait(MPI_Request request[], int ixsub, int jysub,
                     int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *uext, const realtype *host_recv_buff, realtype *dev_recv_buff)
{
  hipError_t err;
  MPI_Status status;
  const sunindextype zero = 0;

  const realtype *h_bufleft   = host_recv_buff;
  const realtype *h_bufright  = host_recv_buff + mysub;
  const realtype *h_buftop    = host_recv_buff + 2*mysub;
  const realtype *h_bufbottom = host_recv_buff + 2*mysub + mxsub;

  realtype *d_bufleft   = dev_recv_buff;
  realtype *d_bufright  = dev_recv_buff + mysub;
  realtype *d_buftop    = dev_recv_buff + 2*mysub;
  realtype *d_bufbottom = dev_recv_buff + 2*mysub + mxsub;

  const sunindextype mxsub2 = mxsub + 2;
  const sunindextype mysub1 = mysub + 1;

  /* If jysub > 0, receive data for bottom x-line of uext. */
  if (jysub != 0) {
    MPI_Wait(&request[0], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufbottom, h_bufbottom, mxsub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the bottom dev_recv_buff to uext. */
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mxsub),
      [=] __device__(sunindextype lx) {
        uext[1 + lx] = d_bufbottom[lx];
      }
    );
  }

  /* If jysub < NPEY-1, receive data for top x-line of uext. */
  if (jysub != npey-1) {
    MPI_Wait(&request[1], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_buftop, h_buftop, mxsub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the top dev_recv_buff to uext. */
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mxsub),
      [=] __device__(sunindextype lx) {
        uext[(1 + mysub1*mxsub2) + lx] = d_buftop[lx];
      }
    );
  }

  /* If ixsub > 0, receive data for left y-line of uext (via bufleft). */
  if (ixsub != 0) {
    MPI_Wait(&request[2], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufleft, h_bufleft, mysub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the left dev_recv_buff to uext. */
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mysub),
      [=] __device__(sunindextype ly) {
        uext[(ly+1)*mxsub2] = d_bufleft[ly];
      }
    );
  }

  /* If ixsub < NPEX-1, receive data for right y-line of uext (via bufright). */
  if (ixsub != npex-1) {
    MPI_Wait(&request[3], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufright, h_bufright, mysub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the right dev_recv_buff to uext. */
    RAJA::forall<RAJA::cuda_exec<256> >(RAJA::RangeSegment(zero, mysub),
      [=] __device__(sunindextype ly) {
        uext[(ly+2)*mxsub2 - 1] = d_bufright[ly];
      }
    );
  }

  return(0);

}

/*
 *--------------------------------------------------------------------
 * PRIVATE FUNCTIONS
 *--------------------------------------------------------------------
 */

/*
 * InitUserData initializes the user's data block data.
 */

static int InitUserData(int thispe, MPI_Comm comm, UserData data)
{

  data->comm    = comm;
  data->thispe  = thispe;
  data->npex    = NPEX;  /* Number of subgrids in x-direction */
  data->npey    = NPEY;  /* Number of subgrids in y-direction */
  data->mxsub   = MXSUB; /* Number of subgrid mesh points in x-direction */
  data->mysub   = MYSUB; /* Number of subgrid mesh points in y-direction */
  data->jysub   = thispe/data->npex;
  data->ixsub   = thispe - (data->jysub * data->npex);
  data->mx      = data->npex * data->mxsub;  /* Mesh size in x-direction */
  data->my      = data->npey * data->mysub;  /* Mesh size in y-direction */
  data->dx      = ONE/(data->mx-ONE); /* Assumes a [0,1] interval in x. */
  data->dy      = ONE/(data->my-ONE); /* Assumes a [0,1] interval in y. */
  data->coeffx  = ONE/(data->dx * data->dx);
  data->coeffy  = ONE/(data->dy * data->dy);
  data->coeffxy = TWO/(data->dx * data->dx) + TWO/(data->dy * data->dy);

  data->uext = NULL;
  data->host_send_buff = NULL;
  data->host_recv_buff = NULL;
  data->dev_send_buff  = NULL;
  data->dev_recv_buff  = NULL;

  return(0);
}


/*
 * AllocUserData allocates memory for the extended vector uext
 * and MPI communication buffers.
 */

static int AllocUserData(MPI_Comm comm, N_Vector uu, UserData data)
{
  hipError_t err;
  sunindextype mxsub = data->mxsub;
  sunindextype mysub = data->mysub;

  /* An N-vector to hold preconditioner. */
  data->pp = N_VClone(uu);
  if(data->pp == NULL) {
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local extended vector (includes ghost nodes) */
  err = hipMalloc((void**) &data->uext, (mxsub + 2)*(mysub +2)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate uext ... \n");
    N_VDestroy(data->pp);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local host send buffer */
  data->host_send_buff = (realtype*) malloc(2*(mxsub + mysub)*sizeof(realtype));
  if(data->host_send_buff == NULL) {
    N_VDestroy(data->pp);
    free(data->uext);
    MPI_Abort(comm, 1);
    return -1;
  }

  data->host_recv_buff = (realtype*) malloc(2*(mxsub + mysub)*sizeof(realtype));
  if(data->host_recv_buff == NULL) {
    N_VDestroy(data->pp);
    free(data->uext);
    free(data->host_send_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local device send buffer */
  err = hipMalloc((void**) &data->dev_send_buff, 2*(mxsub + mysub)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate dev_send_buff ... \n");
    N_VDestroy(data->pp);
    hipFree(data->uext);
    free(data->host_send_buff);
    free(data->host_recv_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local device send buffer */
  err = hipMalloc((void**) &data->dev_recv_buff, 2*(mxsub + mysub)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate dev_recv_buff ... \n");
    N_VDestroy(data->pp);
    hipFree(data->uext);
    free(data->host_send_buff);
    free(data->host_recv_buff);
    hipFree(data->dev_send_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  return 0;
}


static int DeleteUserData(UserData data)
{
  if (data->pp != NULL)
    N_VDestroy(data->pp);
  if (data->uext != NULL)
    hipFree(data->uext);
  if (data->host_send_buff != NULL)
    free(data->host_send_buff);
  if (data->host_recv_buff != NULL)
    free(data->host_recv_buff);
  if (data->dev_send_buff != NULL)
    hipFree(data->dev_send_buff);
  if (data->dev_recv_buff != NULL)
    hipFree(data->dev_recv_buff);
  return 0;
}

/*
 * SetInitialProfile sets the initial values for the problem.
 */

static int SetInitialProfile(N_Vector uu, N_Vector up,  N_Vector id,
                             N_Vector res, UserData data)
{
  sunindextype i, iloc, j, jloc, loc;
  realtype xfact, yfact;

  /* Initialize uu. */

  // Get host pointer
  realtype *uudata = N_VGetHostArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(uu));
  realtype *iddata = N_VGetHostArrayPointer_Raja(N_VGetLocalVector_MPIPlusX(id));

  /* Set mesh spacings and subgrid indices for this PE. */
  const realtype dx = data->dx;
  const realtype dy = data->dy;
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;

  /* Set beginning and ending locations in the global array corresponding
     to the portion of that array assigned to this processor. */
  const sunindextype mxsub   = data->mxsub;
  const sunindextype mysub   = data->mysub;
  const sunindextype ixbegin = mxsub*ixsub;
  const sunindextype ixend   = mxsub*(ixsub+1) - 1;
  const sunindextype jybegin = mysub*jysub;
  const sunindextype jyend   = mysub*(jysub+1) - 1;

  /* Loop over the local array, computing the initial profile value.
     The global indices are (i,j) and the local indices are (iloc,jloc).
     Also set the id vector to zero for boundary points, one otherwise. */

  for (j = jybegin, jloc = 0; j <= jyend; j++, jloc++) {
    yfact = dy*j;
    for (i = ixbegin, iloc = 0; i <= ixend; i++, iloc++) {
      xfact = dx*i;
      loc = iloc + jloc*mxsub;
      uudata[loc] = RCONST(16.0) * xfact * (ONE - xfact) * yfact * (ONE - yfact);

      if (i == 0 || i == data->mx - 1 || j == 0 || j == data->my - 1)
        iddata[loc] = ZERO;
      else
        iddata[loc] = ONE;
    }
  }

  // Synchronize data from the host to the device for uu and id vectors
  N_VCopyToDevice_Raja(N_VGetLocalVector_MPIPlusX(uu));
  N_VCopyToDevice_Raja(N_VGetLocalVector_MPIPlusX(id));

  /* Initialize up. */

  N_VConst(ZERO, up);    /* Initially set up = 0. */

  /* resHeat sets res to negative of ODE RHS values at interior points. */
  resHeat(ZERO, uu, up, res, data);

  /* Copy -res into up to get correct initial up values on the device only! */
  N_VScale(-ONE, res, up);

  return(0);
}

/*
 * Print first lines of output and table heading
 */

static void PrintHeader(realtype rtol, realtype atol, UserData data)
{
  printf("\nidaHeat2D_kry_p: Heat equation, parallel example problem for IDA\n");
  printf("            Discretized heat equation on 2D unit square.\n");
  printf("            Zero boundary conditions,");
  printf(" polynomial initial conditions.\n");
  printf("            Mesh dimensions: %d x %d", (int) data->mx, (int) data->my);
  printf("        Total system size: %ld\n\n", (long) data->mx * data->my);
  printf("Subgrid dimensions: %d x %d", (int) data->mxsub, (int) data->mysub);
  printf("        Processor array: %d x %d\n", (int) data->npex, (int) data->npey);
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("Tolerance parameters:  rtol = %Lg   atol = %Lg\n", rtol, atol);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#else
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#endif
  printf("Constraints set to force all solution components >= 0. \n");
  printf("SUPPRESSALG = SUNTRUE to suppress local error testing on ");
  printf("all boundary components. \n");
  printf("Linear solver: SUNSPGMR  ");
  printf("Preconditioner: diagonal elements only.\n");

  /* Print output table heading and initial line of table. */
  printf("\n   Output Summary (umax = max-norm of solution) \n\n");
  printf("  time     umax       k  nst  nni  nli   nre   nreLS    h      npe nps\n");
  printf("----------------------------------------------------------------------\n");
}

/*
 * PrintOutput: print max norm of solution and current solver statistics
 */

static void PrintOutput(int id, void *ida_mem, realtype t, N_Vector uu)
{
  realtype hused, umax;
  long int nst, nni, nje, nre, nreLS, nli, npe, nps;
  int kused, ier;

  umax = N_VMaxNorm(uu);

  if (id == 0) {

    ier = IDAGetLastOrder(ida_mem, &kused);
    check_flag(&ier, "IDAGetLastOrder", 1, id);
    ier = IDAGetNumSteps(ida_mem, &nst);
    check_flag(&ier, "IDAGetNumSteps", 1, id);
    ier = IDAGetNumNonlinSolvIters(ida_mem, &nni);
    check_flag(&ier, "IDAGetNumNonlinSolvIters", 1, id);
    ier = IDAGetNumResEvals(ida_mem, &nre);
    check_flag(&ier, "IDAGetNumResEvals", 1, id);
    ier = IDAGetLastStep(ida_mem, &hused);
    check_flag(&ier, "IDAGetLastStep", 1, id);
    ier = IDASpilsGetNumJtimesEvals(ida_mem, &nje);
    check_flag(&ier, "IDASpilsGetNumJtimesEvals", 1, id);
    ier = IDASpilsGetNumLinIters(ida_mem, &nli);
    check_flag(&ier, "IDASpilsGetNumLinIters", 1, id);
    ier = IDASpilsGetNumResEvals(ida_mem, &nreLS);
    check_flag(&ier, "IDASpilsGetNumResEvals", 1, id);
    ier = IDASpilsGetNumPrecEvals(ida_mem, &npe);
    check_flag(&ier, "IDASpilsGetPrecEvals", 1, id);
    ier = IDASpilsGetNumPrecSolves(ida_mem, &nps);
    check_flag(&ier, "IDASpilsGetNumPrecSolves", 1, id);

#if defined(SUNDIALS_EXTENDED_PRECISION)
    printf(" %5.2Lf %13.5Le  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2Le  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#else
    printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#endif

  }
}

/*
 * Print some final integrator statistics
 */

static void PrintFinalStats(void *ida_mem)
{
  long int netf, ncfn, ncfl;

  IDAGetNumErrTestFails(ida_mem, &netf);
  IDAGetNumNonlinSolvConvFails(ida_mem, &ncfn);
  IDASpilsGetNumConvFails(ida_mem, &ncfl);

  printf("\nError test failures            = %ld\n", netf);
  printf("Nonlinear convergence failures = %ld\n", ncfn);
  printf("Linear convergence failures    = %ld\n", ncfl);
}

/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns a flag so check if
 *            flag >= 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

static int check_flag(void *flagvalue, const char *funcname, int opt, int id)
{
  int *errflag;

  if (opt == 0 && flagvalue == NULL) {
    /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
    fprintf(stderr,
            "\nSUNDIALS_ERROR(%d): %s() failed - returned NULL pointer\n\n",
            id, funcname);
    return(1);
  } else if (opt == 1) {
    /* Check if flag < 0 */
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr,
              "\nSUNDIALS_ERROR(%d): %s() failed with flag = %d\n\n",
              id, funcname, *errflag);
      return(1);
    }
  } else if (opt == 2 && flagvalue == NULL) {
    /* Check if function returned NULL pointer - no memory allocated */
    fprintf(stderr,
            "\nMEMORY_ERROR(%d): %s() failed - returned NULL pointer\n\n",
            id, funcname);
    return(1);
  }

  return(0);
}
