#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, Cody J. Balos @ LLNL 
 * -----------------------------------------------------------------
 * Based on work by Daniel R. Reynolds @ SMU
 *         Allan Taylor, Alan Hindmarsh and Radu Serban @ LLNL
 * -----------------------------------------------------------------
 * Example problem for IDA: 2D heat equation, parallel, GMRES.
 *
 * This example solves a discretized 2D heat equation problem.
 * This version uses the Krylov solver SUNSPGMR.
 *
 * The DAE system solved is a spatial discretization of the PDE
 *          du/dt = d^2u/dx^2 + d^2u/dy^2
 * on the unit square. The boundary condition is u = 0 on all edges.
 * Initial conditions are given by u = 16 x (1 - x) y (1 - y).
 * The PDE is treated with central differences on a uniform MX x MY
 * grid. The values of u at the interior points satisfy ODEs, and
 * equations u = 0 at the boundaries are appended, to form a DAE
 * system of size N = MX * MY. Here MX = MY = 10.
 *
 * The system is actually implemented on submeshes, processor by
 * processor, with an MXSUB by MYSUB mesh on each of NPEX * NPEY
 * processors.
 *
 * The system is solved with IDA using the Krylov linear solver
 * SUNSPGMR. The preconditioner uses the diagonal elements of the
 * Jacobian only. Routines for preconditioning, required by
 * SUNSPGMR, are supplied here. The constraints u >= 0 are posed
 * for all components. Local error testing on the boundary values
 * is suppressed. Output is taken at t = 0, .01, .02, .04,
 * ..., 10.24.
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <ida/ida.h>
#include <ida/ida_spils.h>
#include <nvector/nvector_cuda.h>
#include <nvector/nvector_mpiplusx.h>
#include <sunlinsol/sunlinsol_spgmr.h>
#include <sundials/sundials_types.h>
#include <sundials/sundials_mpi_types.h>
#include <sundials/sundials_math.h>

#define ZERO  RCONST(0.0)
#define ONE   RCONST(1.0)
#define TWO   RCONST(2.0)

#define NOUT         11    /* Number of output times */

#define NPEX         2     /* No. PEs in x direction of PE array */
#define NPEY         2     /* No. PEs in y direction of PE array */
                           /* Total no. PEs = NPEX*NPEY */
#define MXSUB        5     /* No. x points per subgrid */
#define MYSUB        5     /* No. y points per subgrid */

/* Global spatial mesh is MX x MY = (NPEX x MXSUB) x (NPEY x MYSUB) */

typedef struct {
  int thispe, npex, npey, ixsub, jysub;
  sunindextype mx, my, mxsub, mysub;
  realtype     dx, dy, coeffx, coeffy, coeffxy;
  realtype    *uext; /* device array */
  realtype    *host_send_buff;
  realtype    *host_recv_buff;
  realtype    *dev_send_buff;
  realtype    *dev_recv_buff;
  N_Vector     pp;    /* vector of diagonal preconditioner elements */
  MPI_Comm  comm;
} *UserData;

/* User-supplied residual function and supporting routines */

int resHeat(realtype tt, N_Vector uu, N_Vector up,
            N_Vector rr, void *user_data);

static int rescomm(N_Vector uu, N_Vector up, void *user_data);

static int reslocal(realtype tt, N_Vector uu, N_Vector up,
                    N_Vector res,  void *user_data);

static int BSend(MPI_Comm comm, int thispe,
                 int ixsub, int jysub, int npex, int npey,
                 sunindextype mxsub, sunindextype mysub,
                 const realtype *uarray, realtype *dev_send_buff, realtype *host_send_buff);

static int BRecvPost(MPI_Comm comm, MPI_Request request[], int thispe,
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *host_recv_buff);

static int BRecvWait(MPI_Request request[],
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *uext, const realtype *host_recv_buff, realtype *dev_recv_buff);

/* User-supplied preconditioner routines */

int PsolveHeat(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
               N_Vector rvec, N_Vector zvec, realtype c_j,
               realtype delta, void *user_data);

int PsetupHeat(realtype tt, N_Vector yy, N_Vector yp, N_Vector rr,
               realtype c_j, void *user_data);

/* Private function to check function return values */

static int InitUserData(int thispe, MPI_Comm comm, UserData data);

static int AllocUserData(int thispe, MPI_Comm comm, N_Vector uu, UserData data);

static int DeleteUserData(UserData data);

static int SetInitialProfile(N_Vector uu, N_Vector up, N_Vector id,
                             N_Vector res, UserData data);

static void PrintHeader(realtype rtol, realtype atol, UserData data);

static void PrintOutput(int id, void *ida_mem, realtype t, N_Vector uu);

static void PrintFinalStats(void *ida_mem);

static int check_flag(void *flagvalue, const char *funcname, int opt, int id);


/*
 *--------------------------------------------------------------------
 * CUDA Kernels
 *--------------------------------------------------------------------
 */

__global__
void PsetupHeatKernel(realtype *ppv, sunindextype mx, sunindextype my,
                      sunindextype ibc, sunindextype jbc,
                      sunindextype i0, sunindextype j0, realtype pelinv)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < (mx - ibc)*(my - jbc)) {
    i = tid % (mx - ibc) + i0;
    j = tid / (mx - ibc) + j0;

    /* Loop over interior points; ppv_i = 1/J_ii */
    ppv[i + j*mx] = pelinv;
  }
}


__global__
void CopyLocalToExtendedArray(const realtype *uuv, realtype *uext,
                              sunindextype mx, sunindextype my)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mx*my) {
    i = tid % mx;
    j = tid / mx;

    uext[(i+1) + (j+1)*(mx+2)] = uuv[i + j*mx];
  }
}


__global__
void LocalResidualKernel(const realtype *uext, const realtype *upv, realtype *resv,
                         sunindextype mx, sunindextype my,
                         sunindextype ibc, sunindextype jbc,
                         sunindextype i0, sunindextype j0,
                         realtype coeffx, realtype coeffy, realtype coeffxy)
{
  sunindextype i, j, tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < (mx - ibc)*(my - jbc)) {
    i = tid % (mx - ibc) + i0;
    j = tid / (mx - ibc) + j0;

    sunindextype locu  = i + j*mx;
    sunindextype locue = (i+1) + (j+1)*(mx+2);

    realtype termx   = coeffx * (uext[locue-1]      + uext[locue+1]);
    realtype termy   = coeffy * (uext[locue-(mx+2)] + uext[locue+(mx+2)]);
    realtype termctr = coeffxy * uext[locue];
    resv[locu] = upv[locu] - (termx + termy - termctr);
  }
}


__global__
void CopyToBottomBuffer(const realtype *uarray, realtype *bufbottom,
                        sunindextype mx)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mx) {
      bufbottom[tid] = uarray[tid];
  }
}


__global__
void CopyToTopBuffer(const realtype *uarray, realtype *buftop,
                     sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mx) {
      buftop[tid] = uarray[(my-1)*mx + tid];
  }
}


__global__
void CopyToLeftBuffer(const realtype *uarray, realtype *bufleft,
                      sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < my) {
      bufleft[tid] = uarray[tid*mx];
  }
}


__global__
void CopyToRightBuffer(const realtype *uarray, realtype *bufright,
                       sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < my) {
      bufright[tid] = uarray[tid*mx + (mx-1)];
  }
}


__global__
void CopyFromBottomBuffer(const realtype *bufbottom, realtype *uext,
                          sunindextype mx)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mx) {
      uext[1 + tid] = bufbottom[tid];
  }
}


__global__
void CopyFromTopBuffer(const realtype *buftop, realtype *uext,
                       sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < mx) {
      uext[(1 + (my+1)*(mx+2)) + tid] = buftop[tid];
  }
}


__global__
void CopyFromLeftBuffer(const realtype *bufleft, realtype *uext,
                        sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < my) {
      uext[(tid+1)*(mx+2)] = bufleft[tid];
  }
}


__global__
void CopyFromRightBuffer(const realtype *bufright, realtype *uext,
                         sunindextype mx, sunindextype my)
{
  sunindextype tid;

  /* Loop over all grid points. */
  tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < my) {
      uext[(tid+2)*(mx+2) - 1] = bufright[tid];
  }
}


/*
 *--------------------------------------------------------------------
 * MAIN PROGRAM
 *--------------------------------------------------------------------
 */

int main(int argc, char *argv[])
{
  MPI_Comm comm;
  void *ida_mem;
  SUNLinearSolver LS;
  UserData data;
  int iout, thispe, ier, npes;
  sunindextype local_N;
  realtype rtol, atol, t0, t1, tout, tret;
  N_Vector uulocal, uu, up, constraints, id, res;

  ida_mem = NULL;
  LS = NULL;
  data = NULL;
  uulocal = uu = up = constraints = id = res = NULL;

  /* Get processor number and total number of pe's. */

  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &npes);
  MPI_Comm_rank(comm, &thispe);

  /* Allocate and initialize the data structure */
  data = (UserData) malloc(sizeof *data);
  if(check_flag((void *)data, "malloc", 2, thispe))
    MPI_Abort(comm, 1);

  InitUserData(thispe, comm, data);

  /* Check if the number of MPI processes matches the number of subgrids */
  if (npes != (data->npex * data->npey)) {
    if (thispe == 0)
      fprintf(stderr,
              "\nMPI_ERROR(0): npes = %d is not equal to NPEX*NPEY = %d\n",
              npes, data->npex * data->npey);
    free(data);
    MPI_Finalize();
    return(1);
  }

  /* Set local length local_N */
  local_N = data->mxsub * data->mysub;

  /* Allocate and initialize N-vectors. */

  uulocal = N_VNew_Cuda(local_N);
  if(check_flag((void *)uulocal, "N_VNew_Cuda", 0, thispe))
    MPI_Abort(comm, 1);

  uu = N_VMake_MPIPlusX(comm, uulocal);
  if(check_flag((void *)uu, "N_VMake_MPIPlusX", 0, thispe))
    MPI_Abort(comm, 1);

  up = N_VClone(uu);
  if(check_flag((void *)up, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  res = N_VClone(uu);
  if(check_flag((void *)res, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  constraints = N_VClone(uu);
  if(check_flag((void *)constraints, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  id = N_VClone(uu);
  if(check_flag((void *)id, "N_VClone", 0, thispe))
    MPI_Abort(comm, 1);

  /* Allocate user data extended vector and MPI buffers */
  ier = AllocUserData(thispe, comm, uu, data);
  if(check_flag(&ier, "AllocUserData", 1, thispe)) MPI_Abort(comm, 1);


  /* Initialize the uu, up, id, and res profiles. */
  SetInitialProfile(uu, up, id, res, data);

  /* Set constraints to all 1's for nonnegative solution values. */
  N_VConst(ONE, constraints);

  t0 = ZERO; t1 = RCONST(0.01);

  /* Scalar relative and absolute tolerance. */
  rtol = ZERO;
  atol = RCONST(1.0e-3);

  /* Call IDACreate and IDAMalloc to initialize solution. */

  ida_mem = IDACreate();
  if(check_flag((void *)ida_mem, "IDACreate", 0, thispe)) MPI_Abort(comm, 1);

  ier = IDASetUserData(ida_mem, data);
  if(check_flag(&ier, "IDASetUserData", 1, thispe)) MPI_Abort(comm, 1);

  ier = IDASetSuppressAlg(ida_mem, SUNTRUE);
  if(check_flag(&ier, "IDASetSuppressAlg", 1, thispe)) MPI_Abort(comm, 1);

  ier = IDASetId(ida_mem, id);
  if(check_flag(&ier, "IDASetId", 1, thispe)) MPI_Abort(comm, 1);

  ier = IDASetConstraints(ida_mem, constraints);
  if(check_flag(&ier, "IDASetConstraints", 1, thispe)) MPI_Abort(comm, 1);
  N_VDestroy(constraints);

  ier = IDAInit(ida_mem, resHeat, t0, uu, up);
  if(check_flag(&ier, "IDAInit", 1, thispe)) MPI_Abort(comm, 1);

  ier = IDASStolerances(ida_mem, rtol, atol);
  if(check_flag(&ier, "IDASStolerances", 1, thispe)) MPI_Abort(comm, 1);

  /* Call SUNSPGMR and IDASetLinearSolver to specify the linear solver. */

  LS = SUNSPGMR(uu, PREC_LEFT, 0);  /* use default maxl */
  if(check_flag((void *)LS, "SUNSPGMR", 0, thispe)) MPI_Abort(comm, 1);

  ier = IDASpilsSetLinearSolver(ida_mem, LS);
  if(check_flag(&ier, "IDASpilsSetLinearSolver", 1, thispe)) MPI_Abort(comm, 1);

  ier = IDASpilsSetPreconditioner(ida_mem, PsetupHeat, PsolveHeat);
  if(check_flag(&ier, "IDASpilsSetPreconditioner", 1, thispe)) MPI_Abort(comm, 1);

  /* Print output heading (on processor 0 only) and intial solution  */

  if (thispe == 0) PrintHeader(rtol, atol, data);
  PrintOutput(thispe, ida_mem, t0, uu);

  /* Loop over tout, call IDASolve, print output. */

  for (tout = t1, iout = 1; iout <= NOUT; iout++, tout *= TWO) {

    ier = IDASolve(ida_mem, tout, &tret, uu, up, IDA_NORMAL);
    if(check_flag(&ier, "IDASolve", 1, thispe)) MPI_Abort(comm, 1);

    PrintOutput(thispe, ida_mem, tret, uu);

  }

  /* Print remaining counters. */

  if (thispe == 0) PrintFinalStats(ida_mem);

  /* Free memory */

  IDAFree(&ida_mem);
  SUNLinSolFree(LS);

  N_VDestroy(id);
  N_VDestroy(res);
  N_VDestroy(up);
  N_VDestroy(uu);

  DeleteUserData(data);
  free(data);

  MPI_Finalize();

  return(0);

}

/*
 *--------------------------------------------------------------------
 * FUNCTIONS CALLED BY IDA
 *--------------------------------------------------------------------
 */

/*
 * resHeat: heat equation system residual function
 * This uses 5-point central differencing on the interior points, and
 * includes algebraic equations for the boundary values.
 * So for each interior point, the residual component has the form
 *    res_i = u'_i - (central difference)_i
 * while for each boundary point, it is res_i = u_i.
 *
 * This parallel implementation uses several supporting routines.
 * First a call is made to rescomm to do communication of subgrid boundary
 * data into array uext.  Then reslocal is called to compute the residual
 * on individual processors and their corresponding domains.  The routines
 * BSend, BRecvPost, and BREcvWait handle interprocessor communication
 * of uu required to calculate the residual.
 */

int resHeat(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
            void *user_data)
{
  int retval = 0;

  /* Call rescomm to do inter-processor communication. */
  retval = rescomm(uu, up, user_data);

  /* Call reslocal to calculate res. */
  retval = reslocal(tt, uu, up, rr, user_data);

  return(retval);

}

/*
 * PsetupHeat: setup for diagonal preconditioner for heatsk.
 *
 * The optional user-supplied functions PsetupHeat and
 * PsolveHeat together must define the left preconditoner
 * matrix P approximating the system Jacobian matrix
 *                   J = dF/du + cj*dF/du'
 * (where the DAE system is F(t,u,u') = 0), and solve the linear
 * systems P z = r.   This is done in this case by keeping only
 * the diagonal elements of the J matrix above, storing them as
 * inverses in a vector pp, when computed in PsetupHeat, for
 * subsequent use in PsolveHeat.
 *
 * In this instance, only cj and data (user data structure, with
 * pp etc.) are used from the PsetupHeat argument list.
 *
 */

int PsetupHeat(realtype tt, N_Vector yy, N_Vector yp, N_Vector rr,
               realtype c_j, void *user_data)
{
  sunindextype ibc, i0, jbc, j0;

  /* Unwrap the user data */
  UserData data = (UserData) user_data;
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;
  const int npex  = data->npex;
  const int npey  = data->npey;
  const sunindextype mxsub = data->mxsub;
  const sunindextype mysub = data->mysub;
  realtype *ppv = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(data->pp));

  /* Calculate the value for the inverse element of the diagonal preconditioner */
  const realtype pelinv = ONE/(c_j + data->coeffxy);

  /* Initially set all pp elements on the device to one. */
  N_VConst(ONE, data->pp);

  ibc = (ixsub == 0) || (ixsub == npex-1);
  i0  = (ixsub == 0);
  jbc = (jysub == 0) || (jysub == npey-1);
  j0  = (jysub == 0);

  unsigned block = 256;
  unsigned grid = ((mxsub - ibc)*(mysub - jbc) + block - 1) / block;

  PsetupHeatKernel<<<grid, block>>>(ppv, mxsub, mysub, ibc, jbc, i0, j0, pelinv);

  return(0);
}

/*
 * PsolveHeat: solve preconditioner linear system.
 * This routine multiplies the input vector rvec by the vector pp
 * containing the inverse diagonal Jacobian elements (previously
 * computed in PsetupHeat), returning the result in zvec.
 */

int PsolveHeat(realtype tt, N_Vector uu, N_Vector up,
               N_Vector rr, N_Vector rvec, N_Vector zvec,
               realtype c_j, realtype delta, void *user_data)
{
  UserData data = (UserData) user_data;

  N_VProd(data->pp, rvec, zvec);

  return(0);

}

/*
 *--------------------------------------------------------------------
 * SUPPORTING FUNCTIONS
 *--------------------------------------------------------------------
 */


/*
 * rescomm routine.  This routine performs all inter-processor
 * communication of data in u needed to calculate G.
 */

static int rescomm(N_Vector uu, N_Vector up, void* user_data)
{
  UserData data = (UserData) user_data;

  /* Get comm, thispe, subgrid indices, data sizes */
  MPI_Comm comm = data->comm;
  const int thispe = data->thispe;
  const int ixsub  = data->ixsub;
  const int jysub  = data->jysub;
  const int npex   = data->npex;
  const int npey   = data->npey;
  const sunindextype mxsub = data->mxsub;
  const sunindextype mysub = data->mysub;

  /* Get pointers to buffers and extended solution vector data array uext. */
  realtype *uext = data->uext;
  realtype *host_send_buff = data->host_send_buff;
  realtype *host_recv_buff = data->host_recv_buff;
  realtype *dev_send_buff  = data->dev_send_buff;
  realtype *dev_recv_buff  = data->dev_recv_buff;

  /* Get solution vector data. */
  const realtype *uarray = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(uu));

  /* Set array of MPI requests */
  MPI_Request request[4];

  /* Start receiving boundary data from neighboring PEs. */
  BRecvPost(comm, request, thispe, ixsub, jysub, npex, npey, mxsub, mysub, host_recv_buff);

  /* Send data from boundary of local grid to neighboring PEs. */
  BSend(comm, thispe, ixsub, jysub, npex, npey, mxsub, mysub, uarray, dev_send_buff, host_send_buff);

  /* Finish receiving boundary data from neighboring PEs. */
  BRecvWait(request, ixsub, jysub, npex, npey, mxsub, mysub, uext, host_recv_buff, dev_recv_buff);

  return(0);

}

/*
 * reslocal routine.  Compute res = F(t, uu, up).  This routine assumes
 * that all inter-processor communication of data needed to calculate F
 * has already been done, and that this data is in the work array uext.
 */

static int reslocal(realtype tt, N_Vector uu, N_Vector up, N_Vector rr,
                    void *user_data)
{
  UserData data = (UserData) user_data;

  /* Get subgrid indices, array sizes, and grid coefficients. */
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;
  const int npex  = data->npex;
  const int npey  = data->npey;
  const sunindextype mxsub  = data->mxsub;
  const sunindextype mysub  = data->mysub;
  const realtype coeffx  = data->coeffx;
  const realtype coeffy  = data->coeffy;
  const realtype coeffxy = data->coeffxy;

  /* Vector data arrays, extended work array uext. */
  const realtype *uuv = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(uu));
  const realtype *upv = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(up));
  realtype *resv = N_VGetDeviceArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(rr));
  realtype *uext = data->uext;

  sunindextype ibc, i0, jbc, j0;

  /* Initialize all elements of rr to uu. This sets the boundary
     elements simply without indexing hassles. */

  N_VScale(ONE, uu, rr);

  /* Copy local segment of u vector into the working extended array uext.
     This completes uext prior to the computation of the rr vector.
     uext and uuv must be on the device.     */
  unsigned block = 256;
  unsigned grid = (mxsub*mysub + block - 1) / block;

  CopyLocalToExtendedArray<<<grid, block>>>(uuv, uext, mxsub, mysub);

  /* Set loop limits for the interior of the local subgrid. */
  ibc = (ixsub == 0) || (ixsub == npex-1);
  i0  = (ixsub == 0);
  jbc = (jysub == 0) || (jysub == npey-1);
  j0  = (jysub == 0);

  /* Compute local residual; uext, upv, and resv must be on the device */
  block = 256;
  grid = ((mxsub - ibc)*(mysub - jbc) + block - 1) / block;

  LocalResidualKernel<<<grid, block>>>(uext, upv, resv, mxsub, mysub, ibc, jbc,
                                       i0, j0, coeffx, coeffy, coeffxy);

  return(0);

}

/*
 * Routine to send boundary data to neighboring PEs.
 */

static int BSend(MPI_Comm comm, int thispe,
                 int ixsub, int jysub, int npex, int npey,
                 sunindextype mxsub, sunindextype mysub,
                 const realtype *uarray, realtype *dev_send_buff,
                 realtype *host_send_buff)
{
  hipError_t err;

  /* Have left, right, top and bottom device buffers use the same dev_send_buff. */
  realtype *d_bufleft   = dev_send_buff;
  realtype *d_bufright  = dev_send_buff + mysub;
  realtype *d_buftop    = dev_send_buff + 2*mysub;
  realtype *d_bufbottom = dev_send_buff + 2*mysub + mxsub;

  /* Have left, right, top and bottom host buffers use the same host_send_buff. */
  realtype *h_bufleft   = host_send_buff;
  realtype *h_bufright  = host_send_buff + mysub;
  realtype *h_buftop    = host_send_buff + 2*mysub;
  realtype *h_bufbottom = host_send_buff + 2*mysub + mxsub;

  /* If jysub > 0, send data from bottom x-line of u.  (via bufbottom) */

  if (jysub != 0) {
    // Device kernel here to copy from uarray to the buffer on the device
    unsigned block = 256;
    unsigned grid = (mxsub + block - 1) / block;
    CopyToBottomBuffer<<<grid, block>>>(uarray, d_bufbottom, mxsub);

    // Copy buffer to the host
    err = hipMemcpy(h_bufbottom, d_bufbottom, mxsub*sizeof(realtype),
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Bottom buffer: Copy from device to host failed with code %d... \n", err);
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufbottom, mxsub, MPI_SUNREALTYPE, thispe-npex, 0, comm);
  }

  /* If jysub < NPEY-1, send data from top x-line of u. (via buftop) */

  if (jysub != npey-1) {
    // Device kernel here to copy from uarray to the buffer on the device
    unsigned block = 256;
    unsigned grid = (mxsub + block - 1) / block;
    CopyToTopBuffer<<<grid, block>>>(uarray, d_buftop, mxsub, mysub);

    // Copy buffer to the host
    err = hipMemcpy(h_buftop, d_buftop, mxsub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Top buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_buftop, mxsub, MPI_SUNREALTYPE, thispe+npex, 0, comm);
  }

  /* If ixsub > 0, send data from left y-line of u (via bufleft). */

  if (ixsub != 0) {
    // Device kernel here to copy from uarray to the buffer on the device
    unsigned block = 256;
    unsigned grid = (mysub + block - 1) / block;
    CopyToLeftBuffer<<<grid, block>>>(uarray, d_bufleft, mxsub, mysub);

    // Copy buffer to the host
    err = hipMemcpy(h_bufleft, d_bufleft, mysub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Left buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufleft, mysub, MPI_SUNREALTYPE, thispe-1, 0, comm);
  }

  /* If ixsub < NPEX-1, send data from right y-line of u (via bufright). */

  if (ixsub != npex-1) {
    // Device kernel here to copy from uarray to the buffer on the device
    unsigned block = 256;
    unsigned grid = (mysub + block - 1) / block;
    CopyToRightBuffer<<<grid, block>>>(uarray, d_bufright, mxsub, mysub);

    // Copy buffer to the host
    err = hipMemcpy(h_bufright, d_bufright, mysub*sizeof(realtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      printf("Right buffer: Copy from device to host failed ... \n");
      return -1;
    }
    // MPI send buffer
    MPI_Send(h_bufright, mysub, MPI_SUNREALTYPE, thispe+1, 0, comm);
  }

  return(0);

}

/*
 * Routine to start receiving boundary data from neighboring PEs.
 * Notes:
 *   1) buffer should be able to hold 2*(MYSUB+MYSUB) realtype entries, should
 *      be passed to both the BRecvPost and BRecvWait functions, and should not
 *      be manipulated between the two calls.
 *   2) request should have 4 entries, and should be passed in
 *      both calls also.
 */

static int BRecvPost(MPI_Comm comm, MPI_Request request[], int thispe,
                     int ixsub, int jysub, int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *host_recv_buff)
{
  /* Have left, right, top and bottom buffers use the same host_recv_buff. */
  realtype *bufleft   = host_recv_buff;
  realtype *bufright  = host_recv_buff + mysub;
  realtype *buftop    = host_recv_buff + 2*mysub;
  realtype *bufbottom = host_recv_buff + 2*mysub + mxsub;

  /* If jysub > 0, receive data for bottom x-line of uext. */
  if (jysub != 0) {
    MPI_Irecv(bufbottom, mxsub, MPI_SUNREALTYPE,
              thispe-npex, 0, comm, &request[0]);
  }

  /* If jysub < NPEY-1, receive data for top x-line of uext. */
  if (jysub != npey-1) {
    MPI_Irecv(buftop, mxsub, MPI_SUNREALTYPE,
              thispe+npex, 0, comm, &request[1]);
  }

  /* If ixsub > 0, receive data for left y-line of uext (via bufleft). */
  if (ixsub != 0) {
    MPI_Irecv(&bufleft[0], mysub, MPI_SUNREALTYPE,
              thispe-1, 0, comm, &request[2]);
  }

  /* If ixsub < NPEX-1, receive data for right y-line of uext (via bufright). */
  if (ixsub != npex-1) {
    MPI_Irecv(&bufright[0], mysub, MPI_SUNREALTYPE,
              thispe+1, 0, comm, &request[3]);
  }

  return(0);

}

/*
 * Routine to finish receiving boundary data from neighboring PEs.
 * Notes:
 *   1) buffer should be able to hold 2*MYSUB realtype entries, should be
 *      passed to both the BRecvPost and BRecvWait functions, and should not
 *      be manipulated between the two calls.
 *   2) request should have four entries, and should be passed in both
 *      calls also.
 */

static int BRecvWait(MPI_Request request[], int ixsub, int jysub,
                     int npex, int npey,
                     sunindextype mxsub, sunindextype mysub,
                     realtype *uext, const realtype *host_recv_buff, realtype *dev_recv_buff)
{
  hipError_t err;
  MPI_Status status;

  const realtype *h_bufleft   = host_recv_buff;
  const realtype *h_bufright  = host_recv_buff + mysub;
  const realtype *h_buftop    = host_recv_buff + 2*mysub;
  const realtype *h_bufbottom = host_recv_buff + 2*mysub + mxsub;

  realtype *d_bufleft   = dev_recv_buff;
  realtype *d_bufright  = dev_recv_buff + mysub;
  realtype *d_buftop    = dev_recv_buff + 2*mysub;
  realtype *d_bufbottom = dev_recv_buff + 2*mysub + mxsub;

  /* If jysub > 0, receive data for bottom x-line of uext. */
  if (jysub != 0) {
    MPI_Wait(&request[0], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufbottom, h_bufbottom, mxsub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the bottom dev_recv_buff to uext. */
    unsigned block = 256;
    unsigned grid = (mxsub + block - 1) / block;
    CopyFromBottomBuffer<<<grid, block>>>(d_bufbottom, uext, mxsub);
  }

  /* If jysub < NPEY-1, receive data for top x-line of uext. */
  if (jysub != npey-1) {
    MPI_Wait(&request[1], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_buftop, h_buftop, mxsub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the top dev_recv_buff to uext. */
    unsigned block = 256;
    unsigned grid = (mxsub + block - 1) / block;
    CopyFromTopBuffer<<<grid, block>>>(d_buftop, uext, mxsub, mysub);
  }

  /* If ixsub > 0, receive data for left y-line of uext (via bufleft). */
  if (ixsub != 0) {
    MPI_Wait(&request[2], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufleft, h_bufleft, mysub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the left dev_recv_buff to uext. */
    unsigned block = 256;
    unsigned grid = (mysub + block - 1) / block;
    CopyFromLeftBuffer<<<grid, block>>>(d_bufleft, uext, mxsub, mysub);
  }

  /* If ixsub < NPEX-1, receive data for right y-line of uext (via bufright). */
  if (ixsub != npex-1) {
    MPI_Wait(&request[3], &status);
    /* Copy the buffer from the host to the device */
    err = hipMemcpy(d_bufright, h_bufright, mysub*sizeof(realtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      printf("Copy from host to device failed ... \n");
      return -1;
    }
    /* Copy the right dev_recv_buff to uext. */
    unsigned block = 256;
    unsigned grid = (mysub + block - 1) / block;
    CopyFromRightBuffer<<<grid, block>>>(d_bufright, uext, mxsub, mysub);
  }

  return(0);

}

/*
 *--------------------------------------------------------------------
 * PRIVATE FUNCTIONS
 *--------------------------------------------------------------------
 */

/*
 * InitUserData initializes the user's data block data.
 */

static int InitUserData(int thispe, MPI_Comm comm, UserData data)
{

  data->comm    = comm;
  data->thispe  = thispe;
  data->npex    = NPEX;  /* Number of subgrids in x-direction */
  data->npey    = NPEY;  /* Number of subgrids in y-direction */
  data->mxsub   = MXSUB; /* Number of subgrid mesh points in x-direction */
  data->mysub   = MYSUB; /* Number of subgrid mesh points in y-direction */
  data->jysub   = thispe/data->npex;
  data->ixsub   = thispe - (data->jysub * data->npex);
  data->mx      = data->npex * data->mxsub;  /* Mesh size in x-direction */
  data->my      = data->npey * data->mysub;  /* Mesh size in y-direction */
  data->dx      = ONE/(data->mx-ONE); /* Assumes a [0,1] interval in x. */
  data->dy      = ONE/(data->my-ONE); /* Assumes a [0,1] interval in y. */
  data->coeffx  = ONE/(data->dx * data->dx);
  data->coeffy  = ONE/(data->dy * data->dy);
  data->coeffxy = TWO/(data->dx * data->dx) + TWO/(data->dy * data->dy);

  data->uext = NULL;
  data->host_send_buff = NULL;
  data->host_recv_buff = NULL;
  data->dev_send_buff  = NULL;
  data->dev_recv_buff  = NULL;

  return(0);
}


/*
 * AllocUserData allocates memory for the extended vector uext
 * and MPI communication buffers.
 */

static int AllocUserData(int thispe, MPI_Comm comm, N_Vector uu, UserData data)
{
  hipError_t err;
  sunindextype mxsub = data->mxsub;
  sunindextype mysub = data->mysub;

  /* An N-vector to hold preconditioner. */
  data->pp = N_VClone(uu);
  if(data->pp == NULL) {
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local extended vector (includes ghost nodes) */
  err = hipMalloc((void**) &data->uext, (mxsub + 2)*(mysub +2)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate uext ... \n");
    N_VDestroy(data->pp);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local host send buffer */
  data->host_send_buff = (realtype*) malloc(2*(mxsub + mysub)*sizeof(realtype));
  if(data->host_send_buff == NULL) {
    N_VDestroy(data->pp);
    free(data->uext);
    MPI_Abort(comm, 1);
    return -1;
  }

  data->host_recv_buff = (realtype*) malloc(2*(mxsub + mysub)*sizeof(realtype));
  if(data->host_recv_buff == NULL) {
    N_VDestroy(data->pp);
    free(data->uext);
    free(data->host_send_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local device send buffer */
  err = hipMalloc((void**) &data->dev_send_buff, 2*(mxsub + mysub)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate dev_send_buff ... \n");
    N_VDestroy(data->pp);
    hipFree(data->uext);
    free(data->host_send_buff);
    free(data->host_recv_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  /* Allocate local device send buffer */
  err = hipMalloc((void**) &data->dev_recv_buff, 2*(mxsub + mysub)*sizeof(realtype));
  if(err != hipSuccess) {
    printf("Failed to allocate dev_recv_buff ... \n");
    N_VDestroy(data->pp);
    hipFree(data->uext);
    free(data->host_send_buff);
    free(data->host_recv_buff);
    hipFree(data->dev_send_buff);
    MPI_Abort(comm, 1);
    return -1;
  }

  return 0;
}


static int DeleteUserData(UserData data)
{
  if (data->pp != NULL)
    N_VDestroy(data->pp);
  if (data->uext != NULL)
    hipFree(data->uext);
  if (data->host_send_buff != NULL)
    free(data->host_send_buff);
  if (data->host_recv_buff != NULL)
    free(data->host_recv_buff);
  if (data->dev_send_buff != NULL)
    hipFree(data->dev_send_buff);
  if (data->dev_recv_buff != NULL)
    hipFree(data->dev_recv_buff);
  return 0;
}

/*
 * SetInitialProfile sets the initial values for the problem.
 */

static int SetInitialProfile(N_Vector uu, N_Vector up,  N_Vector id,
                             N_Vector res, UserData data)
{
  sunindextype i, iloc, j, jloc, loc;
  realtype xfact, yfact;

  /* Initialize uu. */

  // Get host pointer
  realtype *uudata = N_VGetHostArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(uu));
  realtype *iddata = N_VGetHostArrayPointer_Cuda(N_VGetLocalVector_MPIPlusX(id));

  /* Set mesh spacings and subgrid indices for this PE. */
  const realtype dx = data->dx;
  const realtype dy = data->dy;
  const int ixsub = data->ixsub;
  const int jysub = data->jysub;

  /* Set beginning and ending locations in the global array corresponding
     to the portion of that array assigned to this processor. */
  const sunindextype mxsub   = data->mxsub;
  const sunindextype mysub   = data->mysub;
  const sunindextype ixbegin = mxsub*ixsub;
  const sunindextype ixend   = mxsub*(ixsub+1) - 1;
  const sunindextype jybegin = mysub*jysub;
  const sunindextype jyend   = mysub*(jysub+1) - 1;

  /* Loop over the local array, computing the initial profile value.
     The global indices are (i,j) and the local indices are (iloc,jloc).
     Also set the id vector to zero for boundary points, one otherwise. */

  for (j = jybegin, jloc = 0; j <= jyend; j++, jloc++) {
    yfact = dy*j;
    for (i = ixbegin, iloc = 0; i <= ixend; i++, iloc++) {
      xfact = dx*i;
      loc = iloc + jloc*mxsub;
      uudata[loc] = RCONST(16.0) * xfact * (ONE - xfact) * yfact * (ONE - yfact);

      if (i == 0 || i == data->mx - 1 || j == 0 || j == data->my - 1)
        iddata[loc] = ZERO;
      else
        iddata[loc] = ONE;
    }
  }

  // Synchronize data from the host to the device for uu and id vectors
  N_VCopyToDevice_Cuda(N_VGetLocalVector_MPIPlusX(uu));
  N_VCopyToDevice_Cuda(N_VGetLocalVector_MPIPlusX(id));

  /* Initialize up. */

  N_VConst(ZERO, up);    /* Initially set up = 0. */

  /* resHeat sets res to negative of ODE RHS values at interior points. */
  resHeat(ZERO, uu, up, res, data);

  /* Copy -res into up to get correct initial up values on the device only! */
  N_VScale(-ONE, res, up);

  return(0);
}

/*
 * Print first lines of output and table heading
 */

static void PrintHeader(realtype rtol, realtype atol, UserData data)
{
  printf("\nidaHeat2D_kry_p: Heat equation, parallel example problem for IDA\n");
  printf("            Discretized heat equation on 2D unit square.\n");
  printf("            Zero boundary conditions,");
  printf(" polynomial initial conditions.\n");
  printf("            Mesh dimensions: %d x %d", (int) data->mx, (int) data->my);
  printf("        Total system size: %ld\n\n", (long) data->mx * data->my);
  printf("Subgrid dimensions: %d x %d", (int) data->mxsub, (int) data->mysub);
  printf("        Processor array: %d x %d\n", (int) data->npex, (int) data->npey);
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("Tolerance parameters:  rtol = %Lg   atol = %Lg\n", rtol, atol);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#else
  printf("Tolerance parameters:  rtol = %g   atol = %g\n", rtol, atol);
#endif
  printf("Constraints set to force all solution components >= 0. \n");
  printf("SUPPRESSALG = SUNTRUE to suppress local error testing on ");
  printf("all boundary components. \n");
  printf("Linear solver: SUNSPGMR  ");
  printf("Preconditioner: diagonal elements only.\n");

  /* Print output table heading and initial line of table. */
  printf("\n   Output Summary (umax = max-norm of solution) \n\n");
  printf("  time     umax       k  nst  nni  nli   nre   nreLS    h      npe nps\n");
  printf("----------------------------------------------------------------------\n");
}

/*
 * PrintOutput: print max norm of solution and current solver statistics
 */

static void PrintOutput(int id, void *ida_mem, realtype t, N_Vector uu)
{
  realtype hused, umax;
  long int nst, nni, nje, nre, nreLS, nli, npe, nps;
  int kused, ier;

  umax = N_VMaxNorm(uu);

  if (id == 0) {

    ier = IDAGetLastOrder(ida_mem, &kused);
    check_flag(&ier, "IDAGetLastOrder", 1, id);
    ier = IDAGetNumSteps(ida_mem, &nst);
    check_flag(&ier, "IDAGetNumSteps", 1, id);
    ier = IDAGetNumNonlinSolvIters(ida_mem, &nni);
    check_flag(&ier, "IDAGetNumNonlinSolvIters", 1, id);
    ier = IDAGetNumResEvals(ida_mem, &nre);
    check_flag(&ier, "IDAGetNumResEvals", 1, id);
    ier = IDAGetLastStep(ida_mem, &hused);
    check_flag(&ier, "IDAGetLastStep", 1, id);
    ier = IDASpilsGetNumJtimesEvals(ida_mem, &nje);
    check_flag(&ier, "IDASpilsGetNumJtimesEvals", 1, id);
    ier = IDASpilsGetNumLinIters(ida_mem, &nli);
    check_flag(&ier, "IDASpilsGetNumLinIters", 1, id);
    ier = IDASpilsGetNumResEvals(ida_mem, &nreLS);
    check_flag(&ier, "IDASpilsGetNumResEvals", 1, id);
    ier = IDASpilsGetNumPrecEvals(ida_mem, &npe);
    check_flag(&ier, "IDASpilsGetPrecEvals", 1, id);
    ier = IDASpilsGetNumPrecSolves(ida_mem, &nps);
    check_flag(&ier, "IDASpilsGetNumPrecSolves", 1, id);

#if defined(SUNDIALS_EXTENDED_PRECISION)
    printf(" %5.2Lf %13.5Le  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2Le  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#else
    printf(" %5.2f %13.5e  %d  %3ld  %3ld  %3ld  %4ld  %4ld  %9.2e  %3ld %3ld\n",
           t, umax, kused, nst, nni, nje, nre, nreLS, hused, npe, nps);
#endif

  }
}

/*
 * Print some final integrator statistics
 */

static void PrintFinalStats(void *ida_mem)
{
  long int netf, ncfn, ncfl;

  IDAGetNumErrTestFails(ida_mem, &netf);
  IDAGetNumNonlinSolvConvFails(ida_mem, &ncfn);
  IDASpilsGetNumConvFails(ida_mem, &ncfl);

  printf("\nError test failures            = %ld\n", netf);
  printf("Nonlinear convergence failures = %ld\n", ncfn);
  printf("Linear convergence failures    = %ld\n", ncfl);
}

/*
 * Check function return value...
 *   opt == 0 means SUNDIALS function allocates memory so check if
 *            returned NULL pointer
 *   opt == 1 means SUNDIALS function returns a flag so check if
 *            flag >= 0
 *   opt == 2 means function allocates memory so check if returned
 *            NULL pointer
 */

static int check_flag(void *flagvalue, const char *funcname, int opt, int id)
{
  int *errflag;

  if (opt == 0 && flagvalue == NULL) {
    /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
    fprintf(stderr,
            "\nSUNDIALS_ERROR(%d): %s() failed - returned NULL pointer\n\n",
            id, funcname);
    return(1);
  } else if (opt == 1) {
    /* Check if flag < 0 */
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr,
              "\nSUNDIALS_ERROR(%d): %s() failed with flag = %d\n\n",
              id, funcname, *errflag);
      return(1);
    }
  } else if (opt == 2 && flagvalue == NULL) {
    /* Check if function returned NULL pointer - no memory allocated */
    fprintf(stderr,
            "\nMEMORY_ERROR(%d): %s() failed - returned NULL pointer\n\n",
            id, funcname);
    return(1);
  }

  return(0);
}
