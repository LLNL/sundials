/* -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2024, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * SUNDIALS CUDA memory helper implementation.
 * ----------------------------------------------------------------*/

#include <cstdlib>
#include <sundials/sundials_math.h>
#include <sunmemory/sunmemory_cuda.h>

#include "sundials/priv/sundials_errors_impl.h"
#include "sundials/sundials_errors.h"
#include "sundials_cuda.h"
#include "sundials_debug.h"

struct SUNMemoryHelper_Content_Cuda_
{
  unsigned long num_allocations_host;
  unsigned long num_deallocations_host;
  unsigned long num_allocations_device;
  unsigned long num_deallocations_device;
  unsigned long num_allocations_pinned;
  unsigned long num_deallocations_pinned;
  unsigned long num_allocations_uvm;
  unsigned long num_deallocations_uvm;
  size_t bytes_allocated_host;
  size_t bytes_high_watermark_host;
  size_t bytes_allocated_device;
  size_t bytes_high_watermark_device;
  size_t bytes_allocated_pinned;
  size_t bytes_high_watermark_pinned;
  size_t bytes_allocated_uvm;
  size_t bytes_high_watermark_uvm;
};

typedef struct SUNMemoryHelper_Content_Cuda_ SUNMemoryHelper_Content_Cuda;

#define SUNHELPER_CONTENT(h) ((SUNMemoryHelper_Content_Cuda*)h->content)

SUNMemoryHelper SUNMemoryHelper_Cuda(SUNContext sunctx)
{
  SUNFunctionBegin(sunctx);

  SUNMemoryHelper helper;

  /* Allocate the helper */
  helper = SUNMemoryHelper_NewEmpty(sunctx);
  SUNCheckLastErrNull();

  /* Set the ops */
  helper->ops->alloc         = SUNMemoryHelper_Alloc_Cuda;
  helper->ops->dealloc       = SUNMemoryHelper_Dealloc_Cuda;
  helper->ops->copy          = SUNMemoryHelper_Copy_Cuda;
  helper->ops->copyasync     = SUNMemoryHelper_CopyAsync_Cuda;
  helper->ops->getallocstats = SUNMemoryHelper_GetAllocStats_Cuda;
  helper->ops->clone         = SUNMemoryHelper_Clone_Cuda;
  helper->ops->destroy       = SUNMemoryHelper_Destroy_Cuda;

  /* Attach content */
  helper->content =
    (SUNMemoryHelper_Content_Cuda*)malloc(sizeof(SUNMemoryHelper_Content_Cuda));
  SUNAssertNull(helper->content, SUN_ERR_MALLOC_FAIL);

  SUNHELPER_CONTENT(helper)->num_allocations_host        = 0;
  SUNHELPER_CONTENT(helper)->num_deallocations_host      = 0;
  SUNHELPER_CONTENT(helper)->bytes_allocated_host        = 0;
  SUNHELPER_CONTENT(helper)->bytes_high_watermark_host   = 0;
  SUNHELPER_CONTENT(helper)->num_allocations_device      = 0;
  SUNHELPER_CONTENT(helper)->num_deallocations_device    = 0;
  SUNHELPER_CONTENT(helper)->bytes_allocated_device      = 0;
  SUNHELPER_CONTENT(helper)->bytes_high_watermark_device = 0;
  SUNHELPER_CONTENT(helper)->num_allocations_pinned      = 0;
  SUNHELPER_CONTENT(helper)->num_deallocations_pinned    = 0;
  SUNHELPER_CONTENT(helper)->bytes_allocated_pinned      = 0;
  SUNHELPER_CONTENT(helper)->bytes_high_watermark_pinned = 0;
  SUNHELPER_CONTENT(helper)->num_allocations_uvm         = 0;
  SUNHELPER_CONTENT(helper)->num_deallocations_uvm       = 0;
  SUNHELPER_CONTENT(helper)->bytes_allocated_uvm         = 0;
  SUNHELPER_CONTENT(helper)->bytes_high_watermark_uvm    = 0;

  return helper;
}

SUNMemoryHelper SUNMemoryHelper_Clone_Cuda(SUNMemoryHelper helper)
{
  SUNFunctionBegin(helper->sunctx);
  SUNMemoryHelper hclone = SUNMemoryHelper_Cuda(helper->sunctx);
  SUNCheckLastErrNull();
  return hclone;
}

SUNErrCode SUNMemoryHelper_Alloc_Cuda(SUNMemoryHelper helper, SUNMemory* memptr,
                                      size_t mem_size, SUNMemoryType mem_type,
                                      void* queue)
{
  SUNFunctionBegin(helper->sunctx);

  SUNMemory mem = SUNMemoryNewEmpty(helper->sunctx);
  SUNCheckLastErr();

  mem->ptr   = NULL;
  mem->own   = SUNTRUE;
  mem->type  = mem_type;
  mem->bytes = mem_size;

  if (mem_type == SUNMEMTYPE_HOST)
  {
    mem->ptr = malloc(mem_size);
    SUNAssert(mem->ptr, SUN_ERR_MALLOC_FAIL);
    SUNHELPER_CONTENT(helper)->bytes_allocated_host += mem_size;
    SUNHELPER_CONTENT(helper)->num_allocations_host++;
    SUNHELPER_CONTENT(helper)->bytes_high_watermark_host =
      SUNMAX(SUNHELPER_CONTENT(helper)->bytes_allocated_host,
             SUNHELPER_CONTENT(helper)->bytes_high_watermark_host);
  }
  else if (mem_type == SUNMEMTYPE_PINNED)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipHostMalloc(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT(
        "ERROR in SUNMemoryHelper_Alloc_Cuda: hipHostMalloc failed\n");
      free(mem);
      return SUN_ERR_EXT_FAIL;
    }
    else
    {
      SUNHELPER_CONTENT(helper)->bytes_allocated_pinned += mem_size;
      SUNHELPER_CONTENT(helper)->num_allocations_pinned++;
      SUNHELPER_CONTENT(helper)->bytes_high_watermark_pinned =
        SUNMAX(SUNHELPER_CONTENT(helper)->bytes_allocated_pinned,
               SUNHELPER_CONTENT(helper)->bytes_high_watermark_pinned);
    }
  }
  else if (mem_type == SUNMEMTYPE_DEVICE)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipMalloc(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT(
        "ERROR in SUNMemoryHelper_Alloc_Cuda: hipMalloc failed\n");
      free(mem);
      return SUN_ERR_EXT_FAIL;
    }
    else
    {
      SUNHELPER_CONTENT(helper)->bytes_allocated_device += mem_size;
      SUNHELPER_CONTENT(helper)->num_allocations_device++;
      SUNHELPER_CONTENT(helper)->bytes_high_watermark_device =
        SUNMAX(SUNHELPER_CONTENT(helper)->bytes_allocated_device,
               SUNHELPER_CONTENT(helper)->bytes_high_watermark_device);
    }
  }
  else if (mem_type == SUNMEMTYPE_UVM)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipMallocManaged(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT(
        "ERROR in SUNMemoryHelper_Alloc_Cuda: hipMallocManaged failed\n");
      free(mem);
      return SUN_ERR_EXT_FAIL;
    }
    else
    {
      SUNHELPER_CONTENT(helper)->bytes_allocated_uvm += mem_size;
      SUNHELPER_CONTENT(helper)->num_allocations_uvm++;
      SUNHELPER_CONTENT(helper)->bytes_high_watermark_uvm =
        SUNMAX(SUNHELPER_CONTENT(helper)->bytes_allocated_uvm,
               SUNHELPER_CONTENT(helper)->bytes_high_watermark_uvm);
    }
  }
  else
  {
    SUNDIALS_DEBUG_PRINT(
      "ERROR in SUNMemoryHelper_Alloc_Cuda: unknown memory type\n");
    free(mem);
    return SUN_ERR_ARG_OUTOFRANGE;
  }

  *memptr = mem;
  return SUN_SUCCESS;
}

SUNErrCode SUNMemoryHelper_Dealloc_Cuda(SUNMemoryHelper helper, SUNMemory mem,
                                        void* queue)
{
  if (mem == NULL) { return SUN_SUCCESS; }

  if (mem->ptr != NULL && mem->own)
  {
    if (mem->type == SUNMEMTYPE_HOST)
    {
      SUNHELPER_CONTENT(helper)->num_deallocations_host++;
      SUNHELPER_CONTENT(helper)->bytes_allocated_host -= mem->bytes;
      free(mem->ptr);
      mem->ptr = NULL;
    }
    else if (mem->type == SUNMEMTYPE_PINNED)
    {
      SUNHELPER_CONTENT(helper)->num_deallocations_pinned++;
      SUNHELPER_CONTENT(helper)->bytes_allocated_pinned -= mem->bytes;
      if (!SUNDIALS_CUDA_VERIFY(hipHostFree(mem->ptr)))
      {
        SUNDIALS_DEBUG_PRINT(
          "ERROR in SUNMemoryHelper_Dealloc_Cuda: hipHostFree failed\n");
        return SUN_ERR_EXT_FAIL;
      }
      mem->ptr = NULL;
    }
    else if (mem->type == SUNMEMTYPE_DEVICE)
    {
      SUNHELPER_CONTENT(helper)->num_deallocations_device++;
      SUNHELPER_CONTENT(helper)->bytes_allocated_device -= mem->bytes;
      if (!SUNDIALS_CUDA_VERIFY(hipFree(mem->ptr)))
      {
        SUNDIALS_DEBUG_PRINT(
          "ERROR in SUNMemoryHelper_Dealloc_Cuda: hipFree failed\n");
        return SUN_ERR_EXT_FAIL;
      }
      mem->ptr = NULL;
    }
    else if (mem->type == SUNMEMTYPE_UVM)
    {
      SUNHELPER_CONTENT(helper)->num_deallocations_uvm++;
      SUNHELPER_CONTENT(helper)->bytes_allocated_uvm -= mem->bytes;
      if (!SUNDIALS_CUDA_VERIFY(hipFree(mem->ptr)))
      {
        SUNDIALS_DEBUG_PRINT(
          "ERROR in SUNMemoryHelper_Dealloc_Cuda: hipFree failed\n");
        return SUN_ERR_EXT_FAIL;
      }
      mem->ptr = NULL;
    }
    else
    {
      SUNDIALS_DEBUG_PRINT(
        "ERROR in SUNMemoryHelper_Dealloc_Cuda: unknown memory type\n");
      return SUN_ERR_EXT_FAIL;
    }
  }

  free(mem);
  return SUN_SUCCESS;
}

SUNErrCode SUNMemoryHelper_Copy_Cuda(SUNMemoryHelper helper, SUNMemory dst,
                                     SUNMemory src, size_t memory_size,
                                     void* queue)
{
  int retval        = SUN_SUCCESS;
  hipError_t cuerr = hipSuccess;

  switch (src->type)
  {
  case SUNMEMTYPE_HOST:
  case SUNMEMTYPE_PINNED:
    if (dst->type == SUNMEMTYPE_HOST || dst->type == SUNMEMTYPE_PINNED)
    {
      memcpy(dst->ptr, src->ptr, memory_size);
    }
    else if (dst->type == SUNMEMTYPE_DEVICE || dst->type == SUNMEMTYPE_UVM)
    {
      cuerr = hipMemcpy(dst->ptr, src->ptr, memory_size, hipMemcpyHostToDevice);
    }
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) { retval = SUN_ERR_EXT_FAIL; }
    break;
  case SUNMEMTYPE_UVM:
  case SUNMEMTYPE_DEVICE:
    if (dst->type == SUNMEMTYPE_HOST || dst->type == SUNMEMTYPE_PINNED)
    {
      cuerr = hipMemcpy(dst->ptr, src->ptr, memory_size, hipMemcpyDeviceToHost);
    }
    else if (dst->type == SUNMEMTYPE_DEVICE || dst->type == SUNMEMTYPE_UVM)
    {
      cuerr = hipMemcpy(dst->ptr, src->ptr, memory_size,
                         hipMemcpyDeviceToDevice);
    }
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) { retval = SUN_ERR_EXT_FAIL; }
    break;
  default:
    SUNDIALS_DEBUG_PRINT(
      "ERROR in SUNMemoryHelper_CopyAsync_Cuda: unknown memory type\n");
    retval = SUN_ERR_OUTOFRANGE;
  }

  return (retval);
}

SUNErrCode SUNMemoryHelper_CopyAsync_Cuda(SUNMemoryHelper helper, SUNMemory dst,
                                          SUNMemory src, size_t memory_size,
                                          void* queue)
{
  int retval          = SUN_SUCCESS;
  hipError_t cuerr   = hipSuccess;
  hipStream_t stream = 0;

  if (queue != NULL) { stream = *((hipStream_t*)queue); }

  switch (src->type)
  {
  case SUNMEMTYPE_HOST:
  case SUNMEMTYPE_PINNED:
    if (dst->type == SUNMEMTYPE_HOST || dst->type == SUNMEMTYPE_PINNED)
    {
      memcpy(dst->ptr, src->ptr, memory_size);
    }
    else if (dst->type == SUNMEMTYPE_DEVICE || dst->type == SUNMEMTYPE_UVM)
    {
      cuerr = hipMemcpyAsync(dst->ptr, src->ptr, memory_size,
                              hipMemcpyHostToDevice, stream);
    }
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) { retval = SUN_ERR_EXT_FAIL; }
    break;
  case SUNMEMTYPE_UVM:
  case SUNMEMTYPE_DEVICE:
    if (dst->type == SUNMEMTYPE_HOST || dst->type == SUNMEMTYPE_PINNED)
    {
      cuerr = hipMemcpyAsync(dst->ptr, src->ptr, memory_size,
                              hipMemcpyDeviceToHost, stream);
    }
    else if (dst->type == SUNMEMTYPE_DEVICE || dst->type == SUNMEMTYPE_UVM)
    {
      cuerr = hipMemcpyAsync(dst->ptr, src->ptr, memory_size,
                              hipMemcpyDeviceToDevice, stream);
    }
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) { retval = SUN_ERR_EXT_FAIL; }
    break;
  default:
    SUNDIALS_DEBUG_PRINT(
      "ERROR in SUNMemoryHelper_CopyAsync_Cuda: unknown memory type\n");
    retval = SUN_ERR_OUTOFRANGE;
  }

  return (retval);
}

SUNErrCode SUNMemoryHelper_Destroy_Cuda(SUNMemoryHelper helper)
{
  if (helper)
  {
    if (helper->content) { free(helper->content); }
    if (helper->ops) { free(helper->ops); }
    free(helper);
  }
  return SUN_SUCCESS;
}

SUNErrCode SUNMemoryHelper_GetAllocStats_Cuda(SUNMemoryHelper helper,
                                              SUNMemoryType mem_type,
                                              unsigned long* num_allocations,
                                              unsigned long* num_deallocations,
                                              size_t* bytes_allocated,
                                              size_t* bytes_high_watermark)
{
  if (mem_type == SUNMEMTYPE_HOST)
  {
    *num_allocations   = SUNHELPER_CONTENT(helper)->num_allocations_host;
    *num_deallocations = SUNHELPER_CONTENT(helper)->num_deallocations_host;
    *bytes_allocated   = SUNHELPER_CONTENT(helper)->bytes_allocated_host;
    *bytes_high_watermark = SUNHELPER_CONTENT(helper)->bytes_high_watermark_host;
  }
  else if (mem_type == SUNMEMTYPE_PINNED)
  {
    *num_allocations   = SUNHELPER_CONTENT(helper)->num_allocations_pinned;
    *num_deallocations = SUNHELPER_CONTENT(helper)->num_deallocations_pinned;
    *bytes_allocated   = SUNHELPER_CONTENT(helper)->bytes_allocated_pinned;
    *bytes_high_watermark = SUNHELPER_CONTENT(helper)->bytes_high_watermark_pinned;
  }
  else if (mem_type == SUNMEMTYPE_DEVICE)
  {
    *num_allocations   = SUNHELPER_CONTENT(helper)->num_allocations_device;
    *num_deallocations = SUNHELPER_CONTENT(helper)->num_deallocations_device;
    *bytes_allocated   = SUNHELPER_CONTENT(helper)->bytes_allocated_device;
    *bytes_high_watermark = SUNHELPER_CONTENT(helper)->bytes_high_watermark_device;
  }
  else if (mem_type == SUNMEMTYPE_UVM)
  {
    *num_allocations      = SUNHELPER_CONTENT(helper)->num_allocations_uvm;
    *num_deallocations    = SUNHELPER_CONTENT(helper)->num_deallocations_uvm;
    *bytes_allocated      = SUNHELPER_CONTENT(helper)->bytes_allocated_uvm;
    *bytes_high_watermark = SUNHELPER_CONTENT(helper)->bytes_high_watermark_uvm;
  }
  else { return SUN_ERR_ARG_OUTOFRANGE; }
  return SUN_SUCCESS;
}
