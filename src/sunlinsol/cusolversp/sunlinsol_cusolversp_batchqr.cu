/* ----------------------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * ----------------------------------------------------------------------------
 * Based on work by Donald Wilcox @ LBNL
 * ----------------------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * ----------------------------------------------------------------------------
 * Implementation file for cuSolverSp batched QR SUNLinearSolver interface.
 * ----------------------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/nvector_cuda.h>
#include <sunmatrix/sunmatrix_sparse.h>
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>
#include <sundials/sundials_math.h>

#define ZERO RCONST(0.0)
#define ONE  RCONST(1.0)
#define TWO  RCONST(2.0)

/* macros for handling the different function names based on precision */
#if defined(SUNDIALS_DOUBLE_PRECISION)
#define _cusolverSpXcsrqrBufferInfoBatched cusolverSpDcsrqrBufferInfoBatched
#define _cusolverSpXcsrqrsvBatched cusolverSpDcsrqrsvBatched
#elif defined(SUNDIALS_SINGLE_PRECISION)
#define _cusolverSpXcsrqrBufferInfoBatched cusolverSpScsrqrBufferInfoBatched
#define _cusolverSpXcsrqrsvBatched cusolverSpScsrqrsvBatched
#endif

/*
 * -----------------------------------------------------------------
 * cuSolverSp solver structure accessibility macros:
 * -----------------------------------------------------------------
 */

#define SUN_CUSP_CONTENT(S)        ( (SUNLinearSolverContent_cuSolverSp_batchQR)(S->content) )
#define SUN_CUSP_QRWORKSPACE(S)    ( SUN_CUSP_CONTENT(S)->workspace )
#define SUN_CUSP_FIRSTFACTORIZE(S) ( SUN_CUSP_CONTENT(S)->first_factorize )
#define SUN_CUSP_LASTFLAG(S)       ( SUN_CUSP_CONTENT(S)->last_flag )
#define SUN_CUSP_HANDLE(S)         ( SUN_CUSP_CONTENT(S)->cusolver_handle )
#define SUN_CUSP_SUBSYS_SIZE(S)    ( SUN_CUSP_CONTENT(S)->subsys_size )
#define SUN_CUSP_SUBSYS_NNZ(S)     ( SUN_CUSP_CONTENT(S)->subsys_nnz )
#define SUN_CUSP_MATDESC(S)        ( SUN_CUSP_CONTENT(S)->system_description )
#define SUN_CUSP_NUM_SUBSYS(S)     ( SUN_CUSP_CONTENT(S)->nsubsys )
#define SUN_CUSP_DROWPTR(S)        ( SUN_CUSP_CONTENT(S)->d_rowptr )
#define SUN_CUSP_DCOLIND(S)        ( SUN_CUSP_CONTENT(S)->d_colind )
#define SUN_CUSP_DVALUES(S)        ( SUN_CUSP_CONTENT(S)->d_values )
#define SUN_CUSP_DESC(S)           ( SUN_CUSP_CONTENT(S)->desc )
#define SUN_CUSP_QRINFO(S)         ( SUN_CUSP_CONTENT(S)->info )
#define SUN_CUSP_INTERNAL_SIZE(S)  ( SUN_CUSP_CONTENT(S)->internal_size )
#define SUN_CUSP_WORK_SIZE(S)      ( SUN_CUSP_CONTENT(S)->workspace_size )

/*
 * ----------------------------------------------------------------------------
 *  Implementations of exported functions.
 * ----------------------------------------------------------------------------
 */

SUNLinearSolver SUNLinSol_cuSolverSp_batchQR(N_Vector y, SUNMatrix A, int nsubsys,
                                             int subsys_size, int subsys_nnz)
{
  /* Check that required arguments are not NULL */
  if (y == NULL || A == NULL) return(NULL);

  /* Check compatibility with supplied SUNMatrix and N_Vector */
  if (SUNMatGetID(A) != SUNMATRIX_SPARSE ||
      N_VGetVectorID(y) != SUNDIALS_NVEC_CUDA) return(NULL);

  /* Check that it is a CSR matrix */
  if (SUNSparseMatrix_SparseType(A) != CSR_MAT) return(NULL);

  /* Check that the vector is using managed memory */
  if (!N_VIsManagedMemory_Cuda(y)) return(NULL);

  /* Matrix must be square */
  if (SUNSparseMatrix_Columns(A) != SUNSparseMatrix_Rows(A)) return(NULL);

  /* Matrix and vector dimensions must agree */
  if (N_VGetLength(y) != SUNSparseMatrix_Columns(A)) return(NULL);

  /* All subsystems must be the same size */
  if (SUNSparseMatrix_Columns(A) != (subsys_size * nsubsys)) return(NULL);

  /* Number of nonzeros per subsys must be the same */
  if (SUNSparseMatrix_NNZ(A) != (subsys_nnz * nsubsys)) return(NULL);

  /* Allocate device memory for the matrix */
  int *d_colind, *d_rowptr;
  realtype *d_values;

  d_colind = NULL;
  d_rowptr = NULL;
  d_values = NULL;

  hipError_t cuerr;
  cuerr = hipMalloc((void **) &d_colind, sizeof(*d_colind) * subsys_nnz);
  if (cuerr != hipSuccess) return(NULL);
  cuerr = hipMalloc((void **) &d_rowptr, sizeof(*d_rowptr) * (subsys_size + 1));
  if (cuerr != hipSuccess) { hipFree(d_colind); return(NULL); }
  cuerr = hipMalloc((void **) &d_values, sizeof(*d_values) * subsys_nnz * nsubsys);
  if (cuerr != hipSuccess) { hipFree(d_rowptr); hipFree(d_colind); return(NULL); }

  /* Create an empty linear solver */
  SUNLinearSolver S;

  S = NULL;
  S = SUNLinSolNewEmpty();
  if (S == NULL) {
    hipFree(d_rowptr); hipFree(d_colind); hipFree(d_values);
    return(NULL);
  }

  /* Attach operations */
  S->ops->gettype    = SUNLinSolGetType_cuSolverSp_batchQR;
  S->ops->getid      = SUNLinSolGetID_cuSolverSp_batchQR;
  S->ops->initialize = SUNLinSolInitialize_cuSolverSp_batchQR;
  S->ops->setup      = SUNLinSolSetup_cuSolverSp_batchQR;
  S->ops->solve      = SUNLinSolSolve_cuSolverSp_batchQR;
  S->ops->lastflag   = SUNLinSolLastFlag_cuSolverSp_batchQR;
  S->ops->free       = SUNLinSolFree_cuSolverSp_batchQR;

  /* Create content */
  SUNLinearSolverContent_cuSolverSp_batchQR content;

  content = NULL;
  content = (SUNLinearSolverContent_cuSolverSp_batchQR) malloc(sizeof *content);
  if (S == NULL) {
    hipFree(d_rowptr); hipFree(d_colind); hipFree(d_values);
    SUNLinSolFree(S);
    return(NULL);
  }

  /* Attach content */
  S->content = content;

  /* Fill content */
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;

  cusolver_status = hipsolverSpCreate(&content->cusolver_handle);
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNLinSolFree(S);
    return(NULL);
  }

  cusparse_status = hipsparseCreateMatDescr(&content->system_description);
  if (cusparse_status != HIPSPARSE_STATUS_SUCCESS) {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNLinSolFree(S);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatType(content->system_description, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (cusparse_status != HIPSPARSE_STATUS_SUCCESS) {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNLinSolFree(S);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatIndexBase(content->system_description, HIPSPARSE_INDEX_BASE_ZERO);
  if (cusparse_status != HIPSPARSE_STATUS_SUCCESS) {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNLinSolFree(S);
    return(NULL);
  }

  content->info        = NULL;
  content->workspace   = NULL;
  content->subsys_size = subsys_size;
  content->subsys_nnz  = subsys_nnz;
  content->nsubsys     = nsubsys;
  content->d_colind    = d_colind;
  content->d_rowptr    = d_rowptr;
  content->d_values    = d_values;
  content->desc        = NULL;

  return(S);
}

/*
 * -----------------------------------------------------------------
 * Implementation of accessor and setter functions.
 * -----------------------------------------------------------------
 */

void SUNLinSol_cuSolverSp_batchQR_GetDescription(SUNLinearSolver S, const char** desc)
{
  *desc = SUN_CUSP_DESC(S);
}

void SUNLinSol_cuSolverSp_batchQR_SetDescription(SUNLinearSolver S, const char* desc)
{
  SUN_CUSP_DESC(S) = desc;
}

/*
 * -----------------------------------------------------------------
 * Implementation of linear solver operations
 * -----------------------------------------------------------------
 */

SUNLinearSolver_Type SUNLinSolGetType_cuSolverSp_batchQR(SUNLinearSolver S)
{
  return(SUNLINEARSOLVER_DIRECT);
}

SUNLinearSolver_ID SUNLinSolGetID_cuSolverSp_batchQR(SUNLinearSolver S)
{
  return(SUNLINEARSOLVER_CUSOLVERSP_BATCHQR);
}

int SUNLinSolInitialize_cuSolverSp_batchQR(SUNLinearSolver S)
{
  SUN_CUSP_FIRSTFACTORIZE(S) = SUNTRUE;
  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;
  return(SUN_CUSP_LASTFLAG(S));
}

int SUNLinSolSetup_cuSolverSp_batchQR(SUNLinearSolver S, SUNMatrix A)
{
  hipError_t cuerr;
  hipsolverStatus_t status;

  /* copy matrix to the device */
  cuerr = hipMemcpy(SUN_CUSP_DCOLIND(S), SUNSparseMatrix_IndexValues(A),
                     sizeof(int) * SUN_CUSP_SUBSYS_NNZ(S), hipMemcpyHostToDevice);
  if (cuerr != hipSuccess) SUN_CUSP_LASTFLAG(S) = SUNLS_MEM_FAIL;

  cuerr = hipMemcpy(SUN_CUSP_DROWPTR(S), SUNSparseMatrix_IndexPointers(A),
                     sizeof(int) * (SUN_CUSP_SUBSYS_SIZE(S)+1), hipMemcpyHostToDevice);
  if (cuerr != hipSuccess) SUN_CUSP_LASTFLAG(S) = SUNLS_MEM_FAIL;

  cuerr = hipMemcpy(SUN_CUSP_DVALUES(S), SUNSparseMatrix_Data(A),
                     sizeof(realtype) * SUN_CUSP_SUBSYS_NNZ(S) * SUN_CUSP_NUM_SUBSYS(S),
                     hipMemcpyHostToDevice);
  if (cuerr != hipSuccess) SUN_CUSP_LASTFLAG(S) = SUNLS_MEM_FAIL;

  if (SUN_CUSP_LASTFLAG(S) != SUNLS_SUCCESS)
    return(SUN_CUSP_LASTFLAG(S));

  if (SUN_CUSP_FIRSTFACTORIZE(S)) {

    /* Free old workspace and symbloic analysis */
    if (SUN_CUSP_QRWORKSPACE(S)) {
      hipFree(SUN_CUSP_QRWORKSPACE(S));
      cusolverSpDestroyCsrqrInfo(SUN_CUSP_QRINFO(S));
    }

    /* We must create a new csrqrinfo_t context every time we want to
       do a symbolic analysis. Trying to reuse it results in a
       HIPSOLVER_STATUS_INVALID_VALUE error. */
    status = cusolverSpCreateCsrqrInfo(&SUN_CUSP_QRINFO(S));
    if (status != HIPSOLVER_STATUS_SUCCESS) {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return(SUN_CUSP_LASTFLAG(S));
    }

    /* Perform symbolic analysis of sparsity structure */
    status = cusolverSpXcsrqrAnalysisBatched(SUN_CUSP_HANDLE(S),
                                             SUN_CUSP_SUBSYS_SIZE(S),
                                             SUN_CUSP_SUBSYS_SIZE(S),
                                             SUN_CUSP_SUBSYS_NNZ(S),
                                             SUN_CUSP_MATDESC(S),
                                             SUN_CUSP_DROWPTR(S),
                                             SUN_CUSP_DCOLIND(S),
                                             SUN_CUSP_QRINFO(S));

    if (status != HIPSOLVER_STATUS_SUCCESS) {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return(SUN_CUSP_LASTFLAG(S));
    }

    /* Compute the workspace we will need */
    status = _cusolverSpXcsrqrBufferInfoBatched(SUN_CUSP_HANDLE(S),
                                                SUN_CUSP_SUBSYS_SIZE(S),
                                                SUN_CUSP_SUBSYS_SIZE(S),
                                                SUN_CUSP_SUBSYS_NNZ(S),
                                                SUN_CUSP_MATDESC(S),
                                                SUN_CUSP_DVALUES(S),
                                                SUN_CUSP_DROWPTR(S),
                                                SUN_CUSP_DCOLIND(S),
                                                SUN_CUSP_NUM_SUBSYS(S),
                                                SUN_CUSP_QRINFO(S),
                                                &SUN_CUSP_INTERNAL_SIZE(S),
                                                &SUN_CUSP_WORK_SIZE(S));

    if (status != HIPSOLVER_STATUS_SUCCESS) {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return(SUN_CUSP_LASTFLAG(S));
    }

    cuerr = hipMalloc((void**) &SUN_CUSP_QRWORKSPACE(S), SUN_CUSP_WORK_SIZE(S));
    if (cuerr != hipSuccess) {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return(SUN_CUSP_LASTFLAG(S));
    }

    SUN_CUSP_FIRSTFACTORIZE(S) = SUNFALSE;
  }

  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;
  return(SUN_CUSP_LASTFLAG(S));
}

int SUNLinSolSolve_cuSolverSp_batchQR(SUNLinearSolver S, SUNMatrix A,
                                      N_Vector x, N_Vector b, realtype tol)
{
  hipsolverStatus_t status;

  if ((S == NULL) || (A == NULL) || (x == NULL) || (b == NULL))
    return(SUNLS_MEM_NULL);

  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;

  realtype* device_b = N_VGetDeviceArrayPointer_Cuda(b);
  realtype* device_x = N_VGetDeviceArrayPointer_Cuda(x);

  if (SUN_CUSP_LASTFLAG(S) != SUNLS_SUCCESS)
    return(SUN_CUSP_LASTFLAG(S));

  /* solve the system */
  status = _cusolverSpXcsrqrsvBatched(SUN_CUSP_HANDLE(S),
                                      SUN_CUSP_SUBSYS_SIZE(S),
                                      SUN_CUSP_SUBSYS_SIZE(S),
                                      SUN_CUSP_SUBSYS_NNZ(S),
                                      SUN_CUSP_MATDESC(S),
                                      SUN_CUSP_DVALUES(S),
                                      SUN_CUSP_DROWPTR(S),
                                      SUN_CUSP_DCOLIND(S),
                                      device_b,
                                      device_x,
                                      SUN_CUSP_NUM_SUBSYS(S),
                                      SUN_CUSP_QRINFO(S),
                                      SUN_CUSP_QRWORKSPACE(S));

  if (status != HIPSOLVER_STATUS_SUCCESS) {
    SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
    return(SUN_CUSP_LASTFLAG(S));
  }

  return(SUN_CUSP_LASTFLAG(S));
}

sunindextype SUNLinSolLastFlag_cuSolverSp_batchQR(SUNLinearSolver S)
{
  if (S == NULL) return(-1);
  return(SUN_CUSP_LASTFLAG(S));
}

int SUNLinSolFree_cuSolverSp_batchQR(SUNLinearSolver S)
{
  /* return with success if already freed */
  if (S == NULL) return(SUNLS_SUCCESS);

  /* free stuff in the content structure */
  hipsolverSpDestroy(SUN_CUSP_HANDLE(S));
  cusolverSpDestroyCsrqrInfo(SUN_CUSP_QRINFO(S));
  hipFree(SUN_CUSP_DCOLIND(S));
  hipFree(SUN_CUSP_DROWPTR(S));
  hipFree(SUN_CUSP_DVALUES(S));
  hipFree(SUN_CUSP_QRWORKSPACE(S));

  /* free content structure */
  if (S->content) {
    free(S->content);
    S->content = NULL;
  }

  /* free ops structure */
  if (S->ops) {
    free(S->ops);
    S->ops = NULL;
  }

  /* free the actual SUNLinSol */
  free(S);
  S = NULL;

  return(SUNLS_SUCCESS);
}
